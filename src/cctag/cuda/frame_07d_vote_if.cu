#include "hip/hip_runtime.h"
/*
 * Copyright 2016, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include <iostream>
#include <algorithm>
#include <limits>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <stdio.h>
#include "debug_macros.hpp"
#include "debug_is_on_edge.h"

#include "frame.h"
#include "frameparam.h"
#include "assist.h"
#include "onoff.h"

using namespace std;

namespace cctag {

struct NumVotersIsGreaterEqual
{
    DevEdgeList<TriplePoint> _array;
    int                      _compare;

    __host__ __device__
    __forceinline__
    NumVotersIsGreaterEqual( DevEdgeList<TriplePoint> _d_array )
        : _array( _d_array )
    {}

    __device__
    __forceinline__
    bool operator()(const int &a) const {
        return (_array.ptr[a]._winnerSize >= tagParam.minVotesToSelectCandidate );
    }
};

#ifdef USE_SEPARABLE_COMPILATION_FOR_VOTE_IF

__global__
void dp_call_vote_if(
    FrameMetaPtr             meta,
    DevEdgeList<TriplePoint> voters,        // input
    DevEdgeList<int>         inner_points,   // output
    DevEdgeList<int>         interm_inner_points,  // input
    cv::cuda::PtrStepSzb     intermediate ) // buffer
{
    /* Filter all chosen inner points that have fewer
     * voters than required by Parameters.
     */

    hipError_t err;

    if( meta.list_size_interm_inner_points() == 0 ) {
        meta.list_size_inner_points() = 0;
        return;
    }

    NumVotersIsGreaterEqual select_op( voters );

    size_t assist_buffer_sz = 0;
    err = hipcub::DeviceSelect::If( 0,
                                 assist_buffer_sz,
                                 interm_inner_points.ptr,
                                 inner_points.ptr,
                                 &meta.list_size_inner_points(),
                                 meta.list_size_interm_inner_points(),
                                 select_op,
                                 0,     // use stream 0
                                 DEBUG_CUB_FUNCTIONS ); // synchronous for debugging
    if( err != hipSuccess ) {
        return;
    }
    if( assist_buffer_sz > intermediate.step * intermediate.rows ) {
        meta.list_size_inner_points() = 0;
        return;
    }
    void*  assist_buffer = (void*)intermediate.data;

    hipcub::DeviceSelect::If( assist_buffer,
                           assist_buffer_sz,
                           interm_inner_points.ptr,
                           inner_points.ptr,
                           &meta.list_size_inner_points(),
                           meta.list_size_interm_inner_points(),
                           select_op,
                           0,     // use stream 0
                           DEBUG_CUB_FUNCTIONS ); // synchronous for debugging
}

__host__
bool Frame::applyVoteIf( )
{
    dp_call_vote_if
        <<<1,1,0,_stream>>>
        ( _meta,
          _voters.dev,  // input
          _inner_points.dev,        // output
          _interm_inner_points.dev,      // input
          cv::cuda::PtrStepSzb(_d_intermediate) ); // buffer
    POP_CHK_CALL_IFSYNC;

    _inner_points.copySizeFromDevice( _stream, EdgeListCont );

    return true;
}
#else // not USE_SEPARABLE_COMPILATION_FOR_VOTE_IF
__host__
bool Frame::applyVoteIf( )
{
    if( _interm_inner_points.host.size == 0 ) {
        return false;
    }

    hipError_t err;

    void*  assist_buffer = (void*)_d_intermediate.data;
    size_t assist_buffer_sz;

    NumVotersIsGreaterEqual select_op( _voters.dev );
    assist_buffer_sz  = 0;
    err = hipcub::DeviceSelect::If( 0,
                                 assist_buffer_sz,
                                 _interm_inner_points.dev.ptr,
                                 _inner_points.dev.ptr,
                                 _d_interm_int,
                                 _interm_inner_points.host.size,
                                 select_op,
                                 _stream,
                                 DEBUG_CUB_FUNCTIONS );

    POP_CUDA_FATAL_TEST( err, "CUB DeviceSelect::If failed in init test" );

    if( assist_buffer_sz >= _d_intermediate.step * _d_intermediate.rows ) {
        std::cerr << "hipcub::DeviceSelect::If requires too much intermediate memory. Crashing." << std::endl;
        exit( -1 );
    }

    /* Filter all chosen inner points that have fewer
     * voters than required by Parameters.
     */
    err = hipcub::DeviceSelect::If( assist_buffer,
                                 assist_buffer_sz,
                                 _interm_inner_points.dev.ptr,
                                 _inner_points.dev.ptr,
                                 _d_interm_int,
                                 _interm_inner_points.host.size,
                                 select_op,
                                 _stream,
                                 DEBUG_CUB_FUNCTIONS );
    POP_CHK_CALL_IFSYNC;
    POP_CUDA_FATAL_TEST( err, "CUB DeviceSelect::If failed" );

    _meta.toDevice_D2S( List_size_inner_points, _d_interm_int, _stream );
    _inner_points.copySizeFromDevice( _stream, EdgeListCont );
    return true;
}
#endif // not USE_SEPARABLE_COMPILATION_FOR_VOTE_IF

} // namespace cctag


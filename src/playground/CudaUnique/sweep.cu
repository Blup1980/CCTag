#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "assist.h"
#include "device_prop.h"
#include "d_block_prefixsum.h"
#include "d_dev_unique.h"

using namespace std;

const size_t NumItemPerThread = 1;
const size_t WarpSize = 32; // threadIdx.x ; threadDim.x
const size_t NumWarpsPerBlock = 32; // threadIdx.y ; threadDim.y
const size_t WarpOffset  = WarpSize * NumItemPerThread;
const size_t BlockOffset = NumWarpsPerBlock * WarpSize * NumItemPerThread;

namespace popart {

__global__
void SweepEqualityBlock( const int32_t* in_array,
                         int16_t*       out_offset_array,
                         int32_t*       out_block_total,
                         const size_t   num_in )
{
    int baseOffset = blockIdx.x * BlockOffset + threadIdx.y * WarpOffset + threadIdx.x;
    int leftOffset = baseOffset - 1 ;
    int ridx = min( baseOffset, (int)(num_in-1) );
    int lidx = max( 0, min( leftOffset, (int)(num_in-1) ) );
    int32_t l = in_array[lidx];
    int32_t r = in_array[ridx];
    size_t  isUniqueElem = ( ridx == 0 ) ? 1 : ( l == r ) ? 0 : 1;
    size_t  blockTotal;
    int16_t writeOffset = (int16_t)PrefixSumBlockExclusive( isUniqueElem, blockTotal );
    if( baseOffset < num_in ) {
        out_offset_array[baseOffset] = isUniqueElem ? writeOffset : -1;
    }
    if( threadIdx.x == 0 && threadIdx.y == 0 ) {
        out_block_total[ blockIdx.x ] = blockTotal;
    }
}

__global__
void SumEqualityBlock( const int      items_per_thread,
                       const int32_t* in_block_total,
                       const int      in_block_items,
                       int32_t*       out_block_prefixsum,
                       size_t*        out_block_overallsum )
{
    size_t offset = ( threadIdx.y * 32 + threadIdx.x ) * items_per_thread;
    int32_t counter = 0;
    for( int i=0; i<items_per_thread; i++ ) {
        counter += (offset+i < in_block_items ) ? in_block_total[offset+i] : 0;
    }
    size_t total;
    size_t exclusiveSum = PrefixSumBlockExclusive( (size_t)counter, total );
    counter = 0;
    for( int i=0; i<items_per_thread; i++ ) {
        if( offset+i >= in_block_items ) return;
        int32_t counterIncrease = in_block_total[offset+i];
        out_block_prefixsum[offset+i] = exclusiveSum + counter;
        counter += counterIncrease;
    }
    if( threadIdx.x == 0 && threadIdx.y == 0 ) {
        *out_block_overallsum = total;
    }
}

/* must have the same block structure as SweepEqualityBlock */
__global__
void WriteUniqueValues( const int32_t* in_array,
                        const int32_t* in_block_sum,
                        const int16_t* in_offset_array,
                        int32_t*       out_array,
                        const size_t   num_in )
{
    size_t baseOffset = blockIdx.x * BlockOffset + threadIdx.y * WarpOffset + threadIdx.x;
    if( baseOffset >= num_in ) return;
    int16_t writeIndex = in_offset_array[baseOffset];
    if( writeIndex == -1 ) return;
    writeIndex += in_block_sum[blockIdx.x];
    out_array[writeIndex] = in_array[baseOffset];
}


} // namespace popart

using namespace popart;

int main( )
{
    device_prop_t dev;
    dev.print();

    const size_t num = 3000;
    size_t    h_num_out;
    int32_t*  h_ptr;
    int32_t*  d_ptr_in;
    int32_t*  d_ptr_out;
    int32_t*  d_ptr_intermediate_1;
    int32_t*  d_ptr_intermediate_2;
    int32_t*  d_ptr_intermediate_3;
    int32_t*   d_ptr_intermediate_4;
    hipHostMalloc( &h_ptr, num*sizeof(int32_t) );
    hipMalloc( &d_ptr_in,  num*sizeof(int32_t) );
    hipMalloc( &d_ptr_out, num*sizeof(int32_t) );
    hipMalloc( &d_ptr_intermediate_1, num*sizeof(int32_t) );
    hipMalloc( &d_ptr_intermediate_2, num*sizeof(int32_t) );
    hipMalloc( &d_ptr_intermediate_3, num*sizeof(int32_t) );
    hipMalloc( &d_ptr_intermediate_4, sizeof(int32_t) );
    for( int i=0; i<num; i++ ) {
        h_ptr[i] = random() % 10;
    }
    cout << "Input array:" << endl;
    for( int i=0; i<num; i++ ) {
        cout << h_ptr[i] << " ";
        if( i%16==15 ) cout << endl;
    }
    cout << endl;
    UniqueArray( h_ptr,
                 h_num_out,
                 num,
                 d_ptr_in,
                 d_ptr_out,
                 d_ptr_intermediate_1,
                 d_ptr_intermediate_2,
                 d_ptr_intermediate_3,
                 d_ptr_intermediate_4 );
    cout << "Unique output item: " << h_num_out << endl;
    cout << "Output array:" << endl;
    for( int i=0; i<num; i++ ) {
        cout << h_ptr[i] << " ";
        if( i%16==15 ) cout << endl;
    }
    cout << endl;
}


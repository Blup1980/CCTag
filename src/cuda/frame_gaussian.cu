#include "hip/hip_runtime.h"
// #include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
// #include <stdio.h>
#include "debug_macros.hpp"

#include "frame.h"
#include "clamp.h"
#include "cctag/talk.hpp" // for DO_TALK macro

namespace popart
{

using namespace std;

/* These numbers are taken from Lilian's file cctag/fiter/cvRecode.cpp
 * Note that the array looks like because a __constant__ device array
 * with 2 dimensions is conceptually very problematic. The reason is
 * that the compiler pads each dimension separately, but there is no
 * way of asking about this padding (pitch, stepsize, whatever you
 * call it).
 * If the kernels should be multi-use, we need one array with two offsets.
 * Aligning to anything less than 16 floats is a bad idea.
 */

#undef NORMALIZE_GAUSS_VALUES

#ifdef NORMALIZE_GAUSS_VALUES
static const float sum_of_gauss_values = 0.000053390535453f +
                                         0.001768051711852f +
                                         0.021539279301849f +
                                         0.096532352630054f +
                                         0.159154943091895f +
                                         0.096532352630054f +
                                         0.021539279301849f +
                                         0.001768051711852f +
                                         0.000053390535453f;
static const float normalize_derived = 2.0f * ( 1.213061319425269f + 0.541341132946452f + 0.066653979229454f + 0.002683701023220f );
#endif // NORMALIZE_GAUSS_VALUES

static const float h_gauss_filter[32] =
{
    0.000053390535453f,
    0.001768051711852f,
    0.021539279301849f,
    0.096532352630054f,
    0.159154943091895f,
    0.096532352630054f,
    0.021539279301849f,
    0.001768051711852f,
    0.000053390535453f,
    0.0f,
    0.0f,
    0.0f,
    0.0f,
    0.0f,
    0.0f,
    0.0f,
    -0.002683701023220f,
    -0.066653979229454f,
    -0.541341132946452f,
    -1.213061319425269f,
    0.0f,
    1.213061319425269f,
    0.541341132946452f,
    0.066653979229454f,
    0.002683701023220f,
    0.0f,
    0.0f,
    0.0f,
    0.0f,
    0.0f,
    0.0f,
    0.0f
};

__device__ __constant__ float d_gauss_filter[32];
// __device__ __constant__ float d_gauss_filter_by_256[16];

template <class SrcType, class DestType>
__global__
void filter_gauss_horiz( cv::cuda::PtrStepSz<SrcType>  src,
                         cv::cuda::PtrStepSz<DestType> dst,
                         int                           filter,
                         float                         scale )
{
    const int idx     = blockIdx.x * 32 + threadIdx.x;
    const int idy     = blockIdx.y;
    float out = 0;

    for( int offset = 0; offset<9; offset++ ) {
        float g  = d_gauss_filter[filter + offset];

        int lookup = clamp( idx + offset - 4, src.cols );
        float val = src.ptr(idy)[lookup];
        out += ( val * g );
    }

    if( idy >= dst.rows ) return;
    if( idx*sizeof(DestType) >= src.step ) return;

    bool nix = ( idx >= dst.cols ) || ( idy >= dst.rows );
    out /= scale;
    dst.ptr(idy)[idx] = nix ? 0 : (DestType)out;
}

template <class SrcType, class DestType>
__global__
void filter_gauss_vert( cv::cuda::PtrStepSz<SrcType>  src,
                        cv::cuda::PtrStepSz<DestType> dst,
                        int                           filter,
                        float                         scale )
{
    const int idx     = blockIdx.x * 32 + threadIdx.x;
    const int idy     = blockIdx.y;
    float out = 0;

    if( idx*sizeof(SrcType) >= src.step ) return;

    for( int offset = 0; offset<9; offset++ ) {
        float g  = d_gauss_filter[filter + offset];

        int lookup = clamp( idy + offset - 4, src.rows );
        float val = src.ptr(lookup)[idx];
        out += ( val * g );
    }

    if( idy >= dst.rows ) return;

    bool nix = ( idx >= dst.cols ) || ( idy >= dst.rows );
    out /= scale;
    dst.ptr(idy)[idx] = nix ? 0 : (DestType)out;
}

__host__
void Frame::initGaussTable( )
{
    POP_CUDA_MEMCPY_HOST_TO_SYMBOL_SYNC( d_gauss_filter,
                                         h_gauss_filter,
                                         32*sizeof(float) );
}

__host__
void Frame::applyGauss( const cctag::Parameters & params )
{
    // cerr << "Enter " << __FUNCTION__ << endl;

    POP_CHK_CALL_IFSYNC;

    dim3 block;
    dim3 grid;
    block.x = 32;
    grid.x  = ( getWidth() / 32 )  + ( getWidth() % 32 == 0 ? 0 : 1 );
    grid.y  = getHeight();
    assert( grid.x > 0 && grid.y > 0 && grid.z > 0 );
    assert( block.x > 0 && block.y > 0 && block.z > 0 );

#ifdef DEBUG_WRITE_ORIGINAL_AS_PGM
    // optional download for debugging
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_plane, getWidth(),
                              _d_plane.data, _d_plane.step,
                              _d_plane.cols,
                              _d_plane.rows,
                              hipMemcpyDeviceToHost, _stream );
    POP_CHK_CALL_IFSYNC;
#endif // DEBUG_WRITE_ORIGINAL_AS_PGM

//    /*
//     * This is the original approach, following the explanation in cvRecode.
//     * However, the 1D tables that we use have already been convolved with
//     * an initial Gauss step, and give the wrong results. So, the first sweep
//     * must be removed.
//     * If the goal was to smoothe the picture, that would be a mistake,
//     * because multiple sweeps extend the range of the filter and bring the
//     * result closer to a globally applied Gaussian filter. However, for CCTag,
//     * this is just a strengthening of the edge signal of a single pixel in its
//     * surrounding area. The far distant pixels don't matter.
//     */
//
//    filter_gauss_horiz_from_uchar<<<grid,block,0,_stream>>>( _d_plane, _d_intermediate, sum_of_gauss_values );
//    filter_gauss_vert<<<grid,block,0,_stream>>>( _d_intermediate, _d_smooth, GAUSS_TABLE, sum_of_gauss_values );
//    filter_gauss_vert<<<grid,block,0,_stream>>>( _d_smooth, _d_intermediate, GAUSS_TABLE, 1.0f );
//    filter_gauss_horiz<<<grid,block,0,_stream>>>( _d_intermediate, _d_debug_dx, GAUSS_DERIV, 1.0f );
//    filter_gauss_horiz<<<grid,block,0,_stream>>>( _d_smooth, _d_intermediate, GAUSS_TABLE, 1.0f );
//    filter_gauss_vert<<<grid,block,0,_stream>>>( _d_intermediate, _d_dy, GAUSS_DERIV, 1.0f );
//

#ifdef NORMALIZE_GAUSS_VALUES
    const float normalize   = sum_of_gauss_values;
    const float normalize_d = normalize_derived;
#else // NORMALIZE_GAUSS_VALUES
    const float normalize   = 1.0f;
    const float normalize_d = 1.0f;
#endif // NORMALIZE_GAUSS_VALUES
    /*
     * Vertical sweep for DX computation: use Gaussian table
     */
    filter_gauss_vert<<<grid,block,0,_stream>>>( _d_plane, _d_intermediate, GAUSS_TABLE, normalize );
    POP_CHK_CALL_IFSYNC;

    /*
     * Compute DX
     */
    filter_gauss_horiz<<<grid,block,0,_stream>>>( _d_intermediate, _d_dx, GAUSS_DERIV, normalize_d );
    POP_CHK_CALL_IFSYNC;

    /*
     * Compute DY
     */
    filter_gauss_vert <<<grid,block,0,_stream>>>( _d_plane, _d_intermediate, GAUSS_DERIV, normalize_d );

    /*
     * Horizontal sweep for DY computation: use Gaussian table
     */
    filter_gauss_horiz<<<grid,block,0,_stream>>>( _d_intermediate, _d_dy, GAUSS_TABLE, normalize );

    // After these linking operations, dx and dy are created for
    // all edge points and we can copy them to the host

    POP_CUDA_MEMCPY_2D_ASYNC( _h_dx.data, _h_dx.step,
                              _d_dx.data, _d_dx.step,
                              _d_dx.cols * sizeof(int16_t),
                              _d_dx.rows,
                              hipMemcpyDeviceToHost, _stream );

    POP_CUDA_MEMCPY_2D_ASYNC( _h_dy.data, _h_dy.step,
                              _d_dy.data, _d_dy.step,
                              _d_dy.cols * sizeof(int16_t),
                              _d_dy.rows,
                              hipMemcpyDeviceToHost, _stream );

    POP_CHK_CALL_IFSYNC;
#ifndef NDEBUG
    if( params._debugDir == "" ) {
        DO_TALK( cerr << __FUNCTION__ << ":" << __LINE__
            << ": debugDir not set, not writing debug output" << endl; )
    } else {
        DO_TALK( cerr << __FUNCTION__ << ":" << __LINE__ << ": debugDir is ["
            << params._debugDir << "] using that directory" << endl; )

        POP_CUDA_SYNC( _stream );

        ostringstream dx_i_out_n;
        dx_i_out_n << params._debugDir << "gpu-dx-" << _layer << "-i.txt";
        ofstream dx_i_out( dx_i_out_n.str().c_str() );
        for( int y=0; y<_d_dx.rows; y++ ) {
            for( int x=0; x<_d_dx.cols; x++ ) {
                dx_i_out << setw(3) << _h_dx.ptr(y)[x] << " ";
            }
            dx_i_out << endl;
        }

        ostringstream dy_i_out_n;
        dy_i_out_n << params._debugDir << "gpu-dy-" << _layer << "-i.txt";
        ofstream dy_i_out( dy_i_out_n.str().c_str() );
        for( int y=0; y<_d_dx.rows; y++ ) {
            for( int x=0; x<_d_dx.cols; x++ ) {
                dy_i_out << setw(3) << _h_dy.ptr(y)[x] << " ";
            }
            dy_i_out << endl;
        }
    }
#endif // not NDEBUG

    // cerr << "Leave " << __FUNCTION__ << endl;
}
}; // namespace popart


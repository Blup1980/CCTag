#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <assert.h>
#include <fstream>
#include <string.h>
#include <hip/hip_runtime.h>
#include <sys/stat.h>
#include "debug_macros.hpp"

#include "../cctag/cmdline.hpp"

#include "frame.h"
#include "assist.h"

#undef CHATTY_WRITE_DEBUG_PLANE

namespace popart {

using namespace std;

/*************************************************************
 * Frame
 *************************************************************/

void Frame::hostDebugDownload( const cctag::Parameters& params )
{
    delete [] _h_debug_plane;
    delete [] _h_debug_smooth;
    delete [] _h_debug_dx;
    delete [] _h_debug_dy;
    delete [] _h_debug_mag;
    delete [] _h_debug_map;
    delete [] _h_debug_hyst_edges;
    delete [] _h_debug_edges;
    delete [] _h_debug_edgelist;
    delete [] _h_debug_edgelist_2;

    _h_debug_plane      = new unsigned char[ getWidth() * getHeight() ];
    _h_debug_smooth     = new float[ getWidth() * getHeight() ];
    _h_debug_dx         = new int16_t[ getWidth() * getHeight() ];
    _h_debug_dy         = new int16_t[ getWidth() * getHeight() ];
    _h_debug_mag        = new uint32_t[ getWidth() * getHeight() ];
    _h_debug_map        = new unsigned char[ getWidth() * getHeight() ];
    _h_debug_hyst_edges = new unsigned char[ getWidth() * getHeight() ];
    _h_debug_edges      = new unsigned char[ getWidth() * getHeight() ];
    _h_debug_edgelist   = new int2[ min(params._maxEdges,_h_edgelist_sz) ];
    _h_debug_edgelist_2 = new TriplePoint[ min(params._maxEdges,_h_edgelist_2_sz) ];

    POP_SYNC_CHK;

#if 0
    cerr << "Trigger download of debug plane: "
         << "(" << _d_plane.cols << "," << _d_plane.rows << ") pitch " << _d_plane.step
         << " to "
         << "(" << getWidth() << "," << getHeight() << ")" << endl;
#endif
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_plane, getWidth(),
                              _d_plane.data, _d_plane.step,
                              _d_plane.cols,
                              _d_plane.rows,
                              hipMemcpyDeviceToHost, _stream );
#if 0
    cerr << "Trigger download of Gaussian debug plane: "
         << "(" << _d_smooth.cols << "," << _d_smooth.rows << ") pitch " << _d_smooth.step
         << " to "
         << "(" << getWidth() << "," << getHeight() << ")" << endl;
#endif
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_smooth, getWidth() * sizeof(float),
                              _d_smooth.data, _d_smooth.step,
                              _d_smooth.cols * sizeof(float),
                              _d_smooth.rows,
                              hipMemcpyDeviceToHost, _stream );
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_dx, getWidth() * sizeof(int16_t),
                              _d_dx.data, _d_dx.step,
                              _d_dx.cols * sizeof(int16_t),
                              _d_dx.rows,
                              hipMemcpyDeviceToHost, _stream );
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_dy, getWidth() * sizeof(int16_t),
                              _d_dy.data, _d_dy.step,
                              _d_dy.cols * sizeof(int16_t),
                              _d_dy.rows,
                              hipMemcpyDeviceToHost, _stream );
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_mag, getWidth() * sizeof(uint32_t),
                              _d_mag.data, _d_mag.step,
                              _d_mag.cols * sizeof(uint32_t),
                              _d_mag.rows,
                              hipMemcpyDeviceToHost, _stream );
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_map, getWidth() * sizeof(uint8_t),
                              _d_map.data, _d_map.step,
                              _d_map.cols * sizeof(uint8_t),
                              _d_map.rows,
                              hipMemcpyDeviceToHost, _stream );
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_hyst_edges, getWidth() * sizeof(uint8_t),
                              _d_hyst_edges.data, _d_hyst_edges.step,
                              _d_hyst_edges.cols * sizeof(uint8_t),
                              _d_hyst_edges.rows,
                              hipMemcpyDeviceToHost, _stream );
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_edges, getWidth() * sizeof(uint8_t),
                              _d_edges.data, _d_edges.step,
                              _d_edges.cols * sizeof(uint8_t),
                              _d_edges.rows,
                              hipMemcpyDeviceToHost, _stream );
    if( _h_edgelist_sz > 0 ) {
        POP_CUDA_MEMCPY_ASYNC( _h_debug_edgelist,
                               _d_edgelist,
                               min(params._maxEdges,_h_edgelist_sz) * sizeof(int2),
                               hipMemcpyDeviceToHost, _stream );
    }
    if( _h_edgelist_2_sz > 0 ) {
        POP_CUDA_MEMCPY_ASYNC( _h_debug_edgelist_2,
                               _d_edgelist_2,
                               min(params._maxEdges,_h_edgelist_2_sz) * sizeof(TriplePoint),
                               hipMemcpyDeviceToHost, _stream );
    }
}

#if 0
__host__
static void testme( cv::cuda::PtrStepSzf src )
{
    size_t non_null_ct = 0;
    float minval = 1000.0f;
    float maxval = -1000.0f;
    for( size_t i=0; i<src.rows; i++ )
        for( size_t j=0; j<src.cols; j++ ) {
            float f = src.ptr(i)[j];
            if( f != 0.0f )
                non_null_ct++;
            minval = min( minval, f );
            maxval = max( maxval, f );
        }
    printf("testme: There are %lu non-null values in the Gaussian end result (min %f, max %f)\n", (unsigned long)non_null_ct, minval, maxval );
}
#endif

void Frame::writeDebugPlane1( const char* filename, const cv::cuda::PtrStepSzb& plane )
{
#ifdef CHATTY_WRITE_DEBUG_PLANE
    cerr << "Enter " << __FUNCTION__ << endl;
#endif
    assert( plane.data );

    ofstream of( filename );
    of << "P5" << endl
       << plane.cols << " " << plane.rows << endl
       << "255" << endl;
    of.write( (char*)plane.data, plane.cols * plane.rows );
#ifdef CHATTY_WRITE_DEBUG_PLANE
    cerr << "Leave " << __FUNCTION__ << endl;
#endif
}

template<class T>
__host__
void Frame::writeDebugPlane( const char* filename, const cv::cuda::PtrStepSz<T>& plane )
{
#ifdef CHATTY_WRITE_DEBUG_PLANE
    cerr << "Enter " << __FUNCTION__ << endl;
    cerr << "    filename: " << filename << endl;
#endif

    ofstream of( filename );
    of << "P5" << endl
       << plane.cols << " " << plane.rows << endl
       << "255" << endl;

    // T minval = 1000;  // std::numeric_limits<float>::max();
    // T maxval = -1000; // std::numeric_limits<float>::min();
    T minval = std::numeric_limits<T>::max();
    T maxval = std::numeric_limits<T>::min();
    // for( uint32_t i=0; i<plane.rows*plane.cols; i++ ) {
    for( size_t i=0; i<plane.rows; i++ ) {
        for( size_t j=0; j<plane.cols; j++ ) {
            T f = plane.ptr(i)[j];
            // float f = plane.data[i];
            minval = min( minval, f );
            maxval = max( maxval, f );
        }
    }
#ifdef CHATTY_WRITE_DEBUG_PLANE
    cerr << "    step size is " << plane.step << endl;
    cerr << "    found minimum value " << minval << endl;
    cerr << "    found maximum value " << maxval << endl;
#endif

    // testme( plane );

    float fmaxval = 255.0 / ( (float)maxval - (float)minval );
    for( uint32_t i=0; i<plane.rows*plane.cols; i++ ) {
        T f = plane.data[i];
        float outf = ( (float)f - (float)minval ) * fmaxval;
        unsigned char uc = (unsigned char)outf;
        of << uc;
    }

#ifdef CHATTY_WRITE_DEBUG_PLANE
    cerr << "Leave " << __FUNCTION__ << endl;
#endif
}

void Frame::writeInt2Array( const char* filename, const int2* array, uint32_t sz )
{
    ofstream of( filename );

    for( uint32_t i=0; i<sz; i++ ) {
        of << array[i].x << " " << array[i].y << endl;
    }
}

void Frame::writeTriplePointArray( const char* filename, const TriplePoint* array, uint32_t sz )
{
    ofstream of( filename );

    for( uint32_t i=0; i<sz; i++ ) {
        of << array[i].coord.x << " " << array[i].coord.y << " "
           << array[i].befor.x << " " << array[i].befor.y << " "
           << array[i].after.x << " " << array[i].after.y << endl;
    }
}

void Frame::hostDebugCompare( unsigned char* pix )
{
    bool found_mistake = false;
    size_t mistake_ct = 0;

    for( int h=0; h<_d_plane.rows; h++ ) {
        for( int w=0; w<_d_plane.cols; w++ ) {
            if( pix[h*_d_plane.cols+w] != _h_debug_plane[h*_d_plane.cols+w] ) {
                mistake_ct++;
                if( found_mistake == false ) {
                    found_mistake = true;
                    cerr << "Found first error at (" << w << "," << h << "): "
                         << "orig " << pix[h*_d_plane.cols+w]
                         << "copy " << _h_debug_plane[h*_d_plane.cols+w]
                         << endl;
                }
            }
        }
    }
    if( found_mistake ) {
        cerr << "Total errors: " << mistake_ct << endl;
    } else {
        cerr << "Found no difference between original and re-downloaded frame" << endl;
    }
}

void Frame::debugPlotPointsIntoImage( const TriplePoint* array, uint32_t sz, cv::cuda::PtrStepSzb img )
{
    for( uint32_t x=0; x<img.cols; x++ ) {
        for( uint32_t y=0; y<img.rows; y++ ) {
            if( img.ptr(y)[x] != 0 ) img.ptr(y)[x] = 1;
        }
    }
    for( uint32_t i=0; i<sz; i++ ) {
        const int2& coord = array[i].coord;
        const int2& befor = array[i].befor;
        // const int2& after = array[i].after;
        if( outOfBounds( coord.x, coord.y, img ) ) {
            cout << "Coord of point (" << coord.x << "," << coord.y << ") is out of bounds" << endl;
        } else {
            // if( befor.x != 0 && befor.y != 0 && after.x != 0 && after.y != 0 )
            if( befor.x != 0 && befor.y != 0 ) {
                img.ptr(coord.y)[coord.x] = 3;
            }
        }
    }
}

void Frame::writeHostDebugPlane( string filename, const cctag::Parameters& params )
{
    struct stat st = {0};

    string dir = cmdline.debugDir;
    char   dirtail = dir[ dir.size()-1 ];
    if( dirtail != '/' ) {
        filename = dir + "/" + filename;
    } else {
        filename = dir + filename;
    }

    if (stat( dir.c_str(), &st) == -1) {
        mkdir( dir.c_str(), 0700);
    }

    string s = filename + ".pgm";
    cv::cuda::PtrStepSzb b( getHeight(),
                            getWidth(),
                            _h_debug_plane,
                            getWidth() );
    writeDebugPlane1( s.c_str(), b );

    {
        ofstream of( ( filename + "-img-ascii.txt" ).c_str() );
        for( int y=0; y<getHeight(); y++ ) {
            for( int x=0; x<getWidth(); x++ )
            {
                int val = b.ptr(y)[x];
                of << val << " ";
            }
            of << endl;
        }
    }

    s = filename + "-gauss.pgm";
    cv::cuda::PtrStepSzf smooth( getHeight(),
                                 getWidth(),
                                 _h_debug_smooth,
                                 getWidth()*sizeof(float) );
    writeDebugPlane( s.c_str(), smooth );

    {
        ofstream of( ( filename + "-gauss-ascii.txt" ).c_str() );
        for( int y=0; y<getHeight(); y++ ) {
            for( int x=0; x<getWidth(); x++ )
            {
                int val = smooth.ptr(y)[x];
                of << val << " ";
            }
            of << endl;
        }
    }

    s = filename + "-dx.pgm";
    cv::cuda::PtrStepSz16s dx( getHeight(),
                               getWidth(),
                               _h_debug_dx,
                               getWidth()*sizeof(int16_t) );
    writeDebugPlane( s.c_str(), dx );

    {
        ofstream of( ( filename + "-dx-ascii.txt" ).c_str() );
        for( int y=0; y<getHeight(); y++ ) {
            for( int x=0; x<getWidth(); x++ )
            {
                int val = dx.ptr(y)[x];
                of << val << " ";
            }
            of << endl;
        }
    }

    s = filename + "-dy.pgm";
    cv::cuda::PtrStepSz16s dy( getHeight(),
                               getWidth(),
                               _h_debug_dy,
                               getWidth()*sizeof(int16_t) );
    writeDebugPlane( s.c_str(), dy );

    {
        ofstream of( ( filename + "-dy-ascii.txt" ).c_str() );
        for( int y=0; y<getHeight(); y++ ) {
            for( int x=0; x<getWidth(); x++ )
            {
                int val = dy.ptr(y)[x];
                of << val << " ";
            }
            of << endl;
        }
    }

    s = filename + "-mag.pgm";
    cv::cuda::PtrStepSz32u mag( getHeight(),
                                getWidth(),
                                _h_debug_mag,
                                getWidth()*sizeof(uint32_t) );
    writeDebugPlane( s.c_str(), mag );

    s = filename + "-map.pgm";
    cv::cuda::PtrStepSzb   map( getHeight(),
                                getWidth(),
                                _h_debug_map,
                                getWidth()*sizeof(uint8_t) );
    writeDebugPlane( s.c_str(), map );

    {
        ofstream of( ( filename + "-map-ascii.txt" ).c_str() );
        for( int y=0; y<getHeight(); y++ ) {
            for( int x=0; x<getWidth(); x++ )
            {
                int val = map.ptr(y)[x];
                of << val << " ";
            }
            of << endl;
        }
    }

    s = filename + "-hystedges.pgm";
    cv::cuda::PtrStepSzb   hystedges( getHeight(),
                                      getWidth(),
                                      _h_debug_hyst_edges,
                                      getWidth()*sizeof(uint8_t) );
    writeDebugPlane( s.c_str(), hystedges );

    s = filename + "-edges.pgm";
    cv::cuda::PtrStepSzb   edges( getHeight(),
                                  getWidth(),
                                  _h_debug_edges,
                                  getWidth()*sizeof(uint8_t) );
    writeDebugPlane( s.c_str(), edges );

    if( _h_edgelist_sz > 0 ) {
        s = filename + "-edgelist.txt";
        writeInt2Array( s.c_str(), _h_debug_edgelist, min(params._maxEdges,_h_edgelist_sz) );
    }

    if( _h_edgelist_2_sz > 0 ) {
        s = filename + "-edgelist2.txt";
        writeTriplePointArray( s.c_str(), _h_debug_edgelist_2, min(params._maxEdges,_h_edgelist_2_sz) );

        debugPlotPointsIntoImage( _h_debug_edgelist_2, min(params._maxEdges,_h_edgelist_2_sz), edges );

        s = filename + "-edges-dots.pgm";
        writeDebugPlane( s.c_str(), edges );
    }
}

}; // namespace popart


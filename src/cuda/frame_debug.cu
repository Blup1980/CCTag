#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <assert.h>
#include <fstream>
#include <string.h>
#include <hip/hip_runtime.h>
#include "debug_macros.hpp"

#include "frame.h"

namespace popart {

using namespace std;

/*************************************************************
 * Frame
 *************************************************************/

void Frame::hostDebugDownload( )
{
    delete [] _h_debug_plane;
    delete [] _h_debug_smooth;
    delete [] _h_debug_dx;
    delete [] _h_debug_dy;

    _h_debug_plane  = new unsigned char[ getWidth() * getHeight() ];
    _h_debug_smooth = new float[ getWidth() * getHeight() ];
    _h_debug_dx     = new float[ getWidth() * getHeight() ];
    _h_debug_dy     = new float[ getWidth() * getHeight() ];

    POP_SYNC_CHK;

#if 0
    cerr << "Trigger download of debug plane: "
         << "(" << _d_plane.cols << "," << _d_plane.rows << ") pitch " << _d_plane.step
         << " to "
         << "(" << getWidth() << "," << getHeight() << ")" << endl;
#endif
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_plane, getWidth(),
                              _d_plane.data, _d_plane.step,
                              _d_plane.cols,
                              _d_plane.rows,
                              hipMemcpyDeviceToHost, _stream );
#if 0
    cerr << "Trigger download of Gaussian debug plane: "
         << "(" << _d_smooth.cols << "," << _d_smooth.rows << ") pitch " << _d_smooth.step
         << " to "
         << "(" << getWidth() << "," << getHeight() << ")" << endl;
#endif
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_smooth, getWidth() * sizeof(float),
                              _d_smooth.data, _d_smooth.step,
                              _d_smooth.cols * sizeof(float),
                              _d_smooth.rows,
                              hipMemcpyDeviceToHost, _stream );
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_dx, getWidth() * sizeof(float),
                              _d_dx.data, _d_dx.step,
                              _d_dx.cols * sizeof(float),
                              _d_dx.rows,
                              hipMemcpyDeviceToHost, _stream );
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_dy, getWidth() * sizeof(float),
                              _d_dy.data, _d_dy.step,
                              _d_dy.cols * sizeof(float),
                              _d_dy.rows,
                              hipMemcpyDeviceToHost, _stream );
}

void Frame::writeDebugPlane( const char* filename, const cv::cuda::PtrStepSzb& plane )
{
    assert( plane.data );

    ofstream of( filename );
    of << "P5" << endl
       << plane.cols << " " << plane.rows << endl
       << "255" << endl;
    of.write( (char*)plane.data, plane.cols * plane.rows );
}

__host__
static void testme( cv::cuda::PtrStepSzf src )
{
    size_t non_null_ct = 0;
    float minval = 1000.0f;
    float maxval = -1000.0f;
    for( size_t i=0; i<src.rows; i++ )
        for( size_t j=0; j<src.cols; j++ ) {
            float f = src.ptr(i)[j];
            if( f != 0.0f )
                non_null_ct++;
            minval = min( minval, f );
            maxval = max( maxval, f );
        }
    printf("testme: There are %lu non-null values in the Gaussian end result (min %f, max %f)\n", (unsigned long)non_null_ct, minval, maxval );
}

void Frame::writeDebugPlane( const char* filename, const cv::cuda::PtrStepSzf& plane )
{
    cerr << "Enter " << __FUNCTION__ << endl;

    ofstream of( filename );
    of << "P5" << endl
       << plane.cols << " " << plane.rows << endl
       << "255" << endl;

    float minval = 1000.0f;  // std::numeric_limits<float>::max();
    float maxval = -1000.0f; // std::numeric_limits<float>::min();
    // for( uint32_t i=0; i<plane.rows*plane.cols; i++ ) {
    for( size_t i=0; i<plane.rows; i++ ) {
        for( size_t j=0; j<plane.cols; j++ ) {
            float f = plane.ptr(i)[j];
            // float f = plane.data[i];
            minval = min( minval, f );
            maxval = max( maxval, f );
        }
    }
    cerr << "    step size is " << plane.step << endl;
    cerr << "    found minimum value " << minval << endl;
    cerr << "    found maximum value " << maxval << endl;

    testme( plane );

    maxval = 255.0f / ( maxval - minval );
    for( uint32_t i=0; i<plane.rows*plane.cols; i++ ) {
        float f = plane.data[i];
        f = ( f - minval ) * maxval;
        unsigned char uc = (unsigned char)f;
        of << uc;
    }

    cerr << "Leave " << __FUNCTION__ << endl;
}

void Frame::hostDebugCompare( unsigned char* pix )
{
    bool found_mistake = false;
    size_t mistake_ct = 0;

    for( int h=0; h<_d_plane.rows; h++ ) {
        for( int w=0; w<_d_plane.cols; w++ ) {
            if( pix[h*_d_plane.cols+w] != _h_debug_plane[h*_d_plane.cols+w] ) {
                mistake_ct++;
                if( found_mistake == false ) {
                    found_mistake = true;
                    cerr << "Found first error at (" << w << "," << h << "): "
                         << "orig " << pix[h*_d_plane.cols+w]
                         << "copy " << _h_debug_plane[h*_d_plane.cols+w]
                         << endl;
                }
            }
        }
    }
    if( found_mistake ) {
        cerr << "Total errors: " << mistake_ct << endl;
    } else {
        cerr << "Found no difference between original and re-downloaded frame" << endl;
    }
}

void Frame::writeHostDebugPlane( string filename )
{
    string s = filename + ".pgm";
    cv::cuda::PtrStepSzb b( getHeight(),
                            getWidth(),
                            _h_debug_plane,
                            getWidth() );
    writeDebugPlane( s.c_str(), b );

    s = filename + "-gauss.pgm";
    cv::cuda::PtrStepSzf smooth( getHeight(),
                                 getWidth(),
                                 _h_debug_smooth,
                                 getWidth()*sizeof(float) );
    writeDebugPlane( s.c_str(), smooth );

    s = filename + "-dx.pgm";
    cv::cuda::PtrStepSzf dx( getHeight(),
                             getWidth(),
                             _h_debug_dx,
                             getWidth()*sizeof(float) );
    writeDebugPlane( s.c_str(), dx );

    s = filename + "-dy.pgm";
    cv::cuda::PtrStepSzf dy( getHeight(),
                             getWidth(),
                             _h_debug_dy,
                             getWidth()*sizeof(float) );
    writeDebugPlane( s.c_str(), dy );
}

}; // namespace popart


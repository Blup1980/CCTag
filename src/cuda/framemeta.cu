#include "hip/hip_runtime.h"
#include <iostream>

#include "onoff.h"
#include "framemeta.h"
#include "debug_macros.hpp"

// #include <hip/hip_runtime.h>
// #include <assert.h>

namespace popart {

/*************************************************************
 * FrameMeta
 * This structure is required for moving things such as counters
 * between host and device. It is meant to replace a memory
 * mapping function that is unreliable.
 */
struct FrameMeta
{
    int   hysteresis_block_counter;
    int   connect_component_block_counter;
    int   ring_counter;
    int   ring_counter_max;
    float identification_result;
    int   identification_resct;
#ifndef NDEBUG
    int   offset_tester;
#endif
#ifdef CPU_GPU_COST_FUNCTION_COMPARE
    int   num_nearby_points;
#endif
};

__device__
FrameMeta frame_meta[ FRAME_META_MAX_PIPES * FRAME_META_MAX_LEVELS ];

__host__
FrameMetaPtr::FrameMetaPtr( int pipeId, int frameId )
    : _pipeId( pipeId )
    , _frameId( frameId )
{
    if( pipeId >= FRAME_META_MAX_PIPES ) {
	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << "Requesting more than " << FRAME_META_MAX_PIPES << " CUDA pipelines."
		  << std::endl
		  << "This requires a recompile."
		  << std::endl;
        exit( -1 );
    }
    if( frameId >= FRAME_META_MAX_LEVELS ) {
	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << "Requesting more than " << FRAME_META_MAX_LEVELS << " CUDA pipelines."
		  << std::endl
		  << "This requires a recompile."
		  << std::endl;
        exit( -1 );
    }

    hipError_t err;
    err = hipGetSymbolAddress( &_d_symbol_ptr, HIP_SYMBOL(frame_meta ));
    POP_CUDA_FATAL_TEST( err, "Could not recover the symbol address for FrameMetas" );
}

#define HOST_DEVICE_TRANSFER_CASE( cond, val ) \
    case cond: \
        offset = (intptr_t)&frame_meta[my_meta].val - (intptr_t)frame_meta; \
        break;

__host__
void FrameMetaPtr::toDevice( FrameMetaEnum e, int val, hipStream_t stream )
{
    const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId;
    intptr_t offset;
    switch( e ) {
    HOST_DEVICE_TRANSFER_CASE( Hysteresis_block_counter, hysteresis_block_counter )
    HOST_DEVICE_TRANSFER_CASE( Connect_component_block_counter, connect_component_block_counter )
    HOST_DEVICE_TRANSFER_CASE( Ring_counter, ring_counter )
    HOST_DEVICE_TRANSFER_CASE( Ring_counter_max, ring_counter_max )
    HOST_DEVICE_TRANSFER_CASE( Identification_resct, identification_resct )
#ifndef NDEBUG
    HOST_DEVICE_TRANSFER_CASE( Offset_tester, offset_tester )
#endif
#ifdef CPU_GPU_COST_FUNCTION_COMPARE
    HOST_DEVICE_TRANSFER_CASE( Num_nearby_points, num_nearby_points )
#endif
    case Identification_result:
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to copy an int to FrameMeta::<float>" << std::endl
		  << "Type is incorrect." << std::endl;
	exit( -1 );
    default :
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to copy an unknown FrameMeta element." << std::endl;
	exit( -1 );
    }
    hipError_t err;
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL( frame_meta), // _d_symbol_ptr,
		                   &val,
				   sizeof(int),
				   offset,
				   hipMemcpyHostToDevice,
				   stream );
    POP_CUDA_FATAL_TEST( err, "Could not copy int variable to device symbol" );
}

__host__
void FrameMetaPtr::toDevice( FrameMetaEnum e, float val, hipStream_t stream )
{
    const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId;
    intptr_t offset;
    switch( e ) {
    HOST_DEVICE_TRANSFER_CASE( Identification_result, identification_result )
    case Hysteresis_block_counter:
    case Connect_component_block_counter:
    case Ring_counter:
    case Ring_counter_max:
    case Identification_resct:
#ifndef NDEBUG
    case Offset_tester:
#endif
#ifdef CPU_GPU_COST_FUNCTION_COMPARE
    case Num_nearby_points:
#endif
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to copy a float to a FrameMeta::<int>" << std::endl
		  << "Type is incorrect." << std::endl;
	exit( -1 );
    default :
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to copy an unknown FrameMeta element." << std::endl;
	exit( -1 );
    }
    hipError_t err;
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL( frame_meta), // _d_symbol_ptr,
		                   &val,
				   sizeof(int),
				   offset,
				   hipMemcpyHostToDevice,
				   stream );
    POP_CUDA_FATAL_TEST( err, "Could not copy float variable to device symbol" );
}

__host__
void FrameMetaPtr::fromDevice( FrameMetaEnum e, int& val, hipStream_t stream )
{
    const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId;
    intptr_t offset;
    switch( e ) {
    HOST_DEVICE_TRANSFER_CASE( Hysteresis_block_counter, hysteresis_block_counter )
    HOST_DEVICE_TRANSFER_CASE( Connect_component_block_counter, connect_component_block_counter )
    HOST_DEVICE_TRANSFER_CASE( Ring_counter, ring_counter )
    HOST_DEVICE_TRANSFER_CASE( Ring_counter_max, ring_counter_max )
    HOST_DEVICE_TRANSFER_CASE( Identification_resct, identification_resct )
#ifndef NDEBUG
    HOST_DEVICE_TRANSFER_CASE( Offset_tester, offset_tester )
#endif
#ifdef CPU_GPU_COST_FUNCTION_COMPARE
    HOST_DEVICE_TRANSFER_CASE( Num_nearby_points, num_nearby_points )
#endif
    case Identification_result:
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to fetch an int to FrameMeta::<float>" << std::endl
		  << "Type is incorrect." << std::endl;
	exit( -1 );
    default :
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to fetch an unknown FrameMeta element." << std::endl;
	exit( -1 );
    }
    hipError_t err;
    err = hipMemcpyFromSymbolAsync( &val,
		                     HIP_SYMBOL(frame_meta), // _d_symbol_ptr,
				     sizeof(int),
				     offset,
				     hipMemcpyDeviceToHost,
				     stream );
    POP_CUDA_FATAL_TEST( err, "Could not copy int variable from device symbol: " );
}

__host__
void FrameMetaPtr::fromDevice( FrameMetaEnum e, float& val, hipStream_t stream )
{
    const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId;
    intptr_t offset;
    switch( e ) {
    HOST_DEVICE_TRANSFER_CASE( Identification_result, identification_result )
    case Hysteresis_block_counter:
    case Connect_component_block_counter:
    case Ring_counter:
    case Ring_counter_max:
    case Identification_resct:
#ifndef NDEBUG
    case Offset_tester:
#endif
#ifdef CPU_GPU_COST_FUNCTION_COMPARE
    case Num_nearby_points:
#endif
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to fetch a float to a FrameMeta::<int>" << std::endl
		  << "Type is incorrect." << std::endl;
        exit( -1 );
    default :
    	std::cerr << __FILE__ << ":" << __LINE__ << std::endl
		  << __FUNCTION__ << std::endl
		  << "Trying to fetch an unknown FrameMeta element." << std::endl;
        exit( -1 );
    }
    hipError_t err;
    err = hipMemcpyFromSymbolAsync( &val,
		                     HIP_SYMBOL(frame_meta), // _d_symbol_ptr,
				     sizeof(int),
				     offset,
				     hipMemcpyDeviceToHost,
				     stream );
    POP_CUDA_FATAL_TEST( err, "Could not copy float variable from device symbol: " );
}

#define OFFSET_GETTER_FUNCTION( type, name ) \
    __device__ \
    type& FrameMetaPtr::name() { \
        const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId; \
        return frame_meta[my_meta].name; \
    } \
    \
    __device__ \
    const type& FrameMetaPtr::name() const { \
        const size_t my_meta = _pipeId*FRAME_META_MAX_LEVELS+_frameId; \
        return frame_meta[my_meta].name; \
    }

OFFSET_GETTER_FUNCTION( int,   hysteresis_block_counter )
OFFSET_GETTER_FUNCTION( int,   connect_component_block_counter )
OFFSET_GETTER_FUNCTION( int,   ring_counter )
OFFSET_GETTER_FUNCTION( int,   ring_counter_max )
OFFSET_GETTER_FUNCTION( float, identification_result )
OFFSET_GETTER_FUNCTION( int,   identification_resct )
#ifndef NDEBUG
OFFSET_GETTER_FUNCTION( int,   offset_tester )
#endif
#ifdef CPU_GPU_COST_FUNCTION_COMPARE
OFFSET_GETTER_FUNCTION( int,   num_nearby_points )
#endif

#ifndef NDEBUG
__global__
void offset_setter( FrameMetaPtr meta )
{
    const size_t my_meta = meta._pipeId*FRAME_META_MAX_LEVELS+meta._frameId;
    int offset = (intptr_t)&frame_meta[my_meta].offset_tester - (intptr_t)frame_meta;
    frame_meta[my_meta].offset_tester = offset;
}

__host__
void FrameMetaPtr::testOffset( hipStream_t stream )
{
    std::cerr << "Enter " << __FUNCTION__ << std::endl;
    std::cerr << "symbol address is " << std::hex << (intptr_t)_d_symbol_ptr
	      << std::dec  << std::endl;
    offset_setter
        <<<1,1,0,stream>>>
	( *this );
    int offset_value;
    fromDevice( Offset_tester, offset_value, stream );
    hipStreamSynchronize( stream );
    std::cerr << "OFFSET TESTING" << std::endl
	      << std::endl
	      << "Offset: " << offset_value << std::endl
	      << std::endl
	      << "END OFFSET TESTING" << std::endl;
    std::cerr << "Leave " << __FUNCTION__ << std::endl;
}
#endif // NDEBUG

}; // namespace popart


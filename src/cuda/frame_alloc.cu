#include "hip/hip_runtime.h"
// #include <iostream>
// #include <limits>
#include <hip/hip_runtime.h>
// #include <stdio.h>
#include "debug_macros.hpp"

#include "frame.h"

namespace popart
{

using namespace std;

__host__
void Frame::allocDevGaussianPlane( const cctag::Parameters& params )
{
    cerr << "Enter " << __FUNCTION__ << endl;

    void* ptr;
    const size_t w = getWidth();
    const size_t h = getHeight();
    size_t p;

    POP_CUDA_MALLOC_PITCH( &ptr, &p, w*sizeof(float), h );
    assert( p % _d_smooth.elemSize() == 0 );
    _d_smooth.data = (float*)ptr;
    _d_smooth.step = p;
    _d_smooth.cols = w;
    _d_smooth.rows = h;

    POP_CUDA_MALLOC_PITCH( &ptr, &p, w*sizeof(int16_t), h );
    assert( p % _d_dx.elemSize() == 0 );
    _d_dx.data = (int16_t*)ptr;
    _d_dx.step = p;
    _d_dx.cols = w;
    _d_dx.rows = h;

    POP_CUDA_MALLOC_PITCH( &ptr, &p, w*sizeof(int16_t), h );
    assert( p % _d_dy.elemSize() == 0 );
    _d_dy.data = (int16_t*)ptr;
    _d_dy.step = p;
    _d_dy.cols = w;
    _d_dy.rows = h;

    POP_CUDA_MALLOC_PITCH( &ptr, &p, w*sizeof(float), h );
    assert( p % _d_intermediate.elemSize() == 0 );
    _d_intermediate.data = (float*)ptr;
    _d_intermediate.step = p;
    _d_intermediate.cols = w;
    _d_intermediate.rows = h;

    POP_CUDA_MALLOC_PITCH( &ptr, &p, w*sizeof(uint32_t), h );
    assert( p % _d_mag.elemSize() == 0 );
    _d_mag.data = (uint32_t*)ptr;
    _d_mag.step = p;
    _d_mag.cols = w;
    _d_mag.rows = h;

    POP_CUDA_MALLOC_PITCH( &ptr, &p, w*sizeof(unsigned char), h );
    assert( p % _d_map.elemSize() == 0 );
    _d_map.data = (unsigned char*)ptr;
    _d_map.step = p;
    _d_map.cols = w;
    _d_map.rows = h;

    POP_CUDA_MALLOC_PITCH( &ptr, &p, w*sizeof(unsigned char), h );
    assert( p % _d_hyst_edges.elemSize() == 0 );
    _d_hyst_edges.data = (unsigned char*)ptr;
    _d_hyst_edges.step = p;
    _d_hyst_edges.cols = w;
    _d_hyst_edges.rows = h;

    POP_CUDA_MALLOC_PITCH( &ptr, &p, w*sizeof(unsigned char), h );
    assert( p % _d_edges.elemSize() == 0 );
    _d_edges.data = (unsigned char*)ptr;
    _d_edges.step = p;
    _d_edges.cols = w;
    _d_edges.rows = h;

    POP_CUDA_MALLOC( &ptr, sizeof(int) );
    _d_hysteresis_block_counter        = (int*)ptr;

    POP_CUDA_MALLOC( &ptr, sizeof(int) );
    _d_connect_component_block_counter = (int*)ptr;

    _vote.alloc( params, w, h );

    POP_CUDA_MEMSET_ASYNC( _d_smooth.data,
                           0,
                           _d_smooth.step * _d_smooth.rows,
                           _stream );

    POP_CUDA_MEMSET_ASYNC( _d_dx.data,
                           0,
                           _d_dx.step * _d_dx.rows,
                           _stream );

    POP_CUDA_MEMSET_ASYNC( _d_dy.data,
                           0,
                           _d_dy.step * _d_dy.rows,
                           _stream );

    POP_CUDA_MEMSET_ASYNC( _d_intermediate.data,
                           0,
                           _d_intermediate.step * _d_intermediate.rows,
                           _stream );

    POP_CUDA_MEMSET_ASYNC( _d_mag.data,
                           0,
                           _d_mag.step * _d_mag.rows,
                           _stream );

    POP_CUDA_MEMSET_ASYNC( _d_map.data,
                           0,
                           _d_map.step * _d_map.rows,
                           _stream );

    POP_CUDA_MEMSET_ASYNC( _d_edges.data,
                           0,
                           _d_edges.step * _d_edges.rows,
                           _stream );

    _vote.init( params, _stream );

    // POP_CUDA_MEMSET_ASYNC( _d_next_edge_after.data,
    //                        0,
    //                        _d_next_edge_after.step * _d_next_edge_after.rows,
    //                        _stream );

    // POP_CUDA_MEMSET_ASYNC( _d_next_edge_befor.data,
    //                        0,
    //                        _d_next_edge_befor.step * _d_next_edge_befor.rows,
    //                        _stream );

    cerr << "Leave " << __FUNCTION__ << endl;
}

void Voting::alloc( const cctag::Parameters& params, size_t w, size_t h )
{
    void*  ptr;
    size_t p;

    POP_CUDA_MALLOC( &ptr, params._maxEdges*sizeof(int2) );
    _all_edgecoords.dev.ptr = (int2*)ptr;

    POP_CUDA_MALLOC( &ptr, sizeof(int) );
    _all_edgecoords.dev.size = (int*)ptr;

    POP_CUDA_MALLOC( &ptr, params._maxEdges*sizeof(TriplePoint) );
    _chained_edgecoords.dev.ptr = (TriplePoint*)ptr;

    POP_CUDA_MALLOC( &ptr, sizeof(int) );
    _chained_edgecoords.dev.size = (int*)ptr;

    POP_CUDA_MALLOC( &ptr, params._maxEdges*sizeof(int) );
    _edge_indices.dev.ptr = (int*)ptr;

    POP_CUDA_MALLOC( &ptr, sizeof(int) );
    _edge_indices.dev.size = (int*)ptr;

    POP_CUDA_MALLOC( &ptr, params._maxEdges*sizeof(int) );
    _edge_indices_2.dev.ptr = (int*)ptr;

    POP_CUDA_MALLOC( &ptr, sizeof(int) );
    _edge_indices_2.dev.size = (int*)ptr;

    POP_CUDA_MALLOC_PITCH( &ptr, &p, w*sizeof(int32_t), h );
    assert( p % _d_edgepoint_index_table.elemSize() == 0 );
    _d_edgepoint_index_table.data = (int32_t*)ptr;
    _d_edgepoint_index_table.step = p;
    _d_edgepoint_index_table.cols = w;
    _d_edgepoint_index_table.rows = h;
}

void Voting::init( const cctag::Parameters& params, hipStream_t stream )
{
    POP_CUDA_MEMSET_ASYNC( _all_edgecoords.dev.ptr,
                           0,
                           params._maxEdges*sizeof(int2),
                           stream );

    POP_CUDA_MEMSET_ASYNC( _chained_edgecoords.dev.ptr,
                           0,
                           params._maxEdges*sizeof(TriplePoint),
                           stream );

    POP_CUDA_MEMSET_ASYNC( _edge_indices.dev.ptr,
                           0,
                           params._maxEdges*sizeof(int),
                           stream );

    POP_CUDA_MEMSET_ASYNC( _edge_indices_2.dev.ptr,
                           0,
                           params._maxEdges*sizeof(int),
                           stream );

    POP_CUDA_MEMSET_ASYNC( _d_edgepoint_index_table.data,
                           0,
                           _d_edgepoint_index_table.step * _d_edgepoint_index_table.rows,
                           stream );
}

void Voting::release( )
{
    POP_CUDA_FREE( _all_edgecoords.dev.ptr );
    POP_CUDA_FREE( _all_edgecoords.dev.size );
    POP_CUDA_FREE( _chained_edgecoords.dev.ptr );
    POP_CUDA_FREE( _chained_edgecoords.dev.size );
    POP_CUDA_FREE( _edge_indices.dev.ptr );
    POP_CUDA_FREE( _edge_indices.dev.size );
    POP_CUDA_FREE( _edge_indices_2.dev.ptr );
    POP_CUDA_FREE( _edge_indices_2.dev.size );
    POP_CUDA_FREE( _d_edgepoint_index_table.data );
}

}; // namespace popart


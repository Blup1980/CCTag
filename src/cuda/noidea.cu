#include "hip/hip_runtime.h"
__global__
void compute_map( const PtrStepSz16s dx,
                  const PtrStepSz16s dy,
                  const PtrStepSz32u mag,
                  PtrStepSzb         map,
                  const float        low_thresh,
                  const float        high_thresh )
{
    const int CANNY_SHIFT = 15;
    const int TG22 = (int32_t)(0.4142135623730950488016887242097*(1<<CANNY_SHIFT) + 0.5);

    const int block_x = blockIdx.x * V7_WIDTH;
    const int idx     = block_x + threadIdx.x;
    const int idy     = blockIdx.y;

    if( idx >= dx.cols ) return;
    if( idy >= dx.rows ) return;

    int32_t  dxVal  = dx.ptr(idy)[idx];
    int32_t  dyVal  = dy.ptr(idy)[idx];
    uint32_t magVal = mag.ptr(idy)[idx];

    // -1 if only is negative, 1 else
    const int signVal = (dxVal ^ dyVal) < 0 ? -1 : 1;

    dxVal = ::abs(dxVal);
    dyVal = ::abs(dyVal);

    // 0 - the pixel can not belong to an edge
    // 1 - the pixel might belong to an edge
    // 2 - the pixel does belong to an edge
    uint8_t edge_type = 0;

    if( magVal > low_thresh )
    {
        const int32_t tg22x = dxVal * TG22;
        const int32_t tg67x = tg22x + ((dxVal + dxVal) << CANNY_SHIFT);

        dyVal <<= CANNY_SHIFT;

        int2 x = (dyVal < tg22x) ? make_int2( idx - 1, idx + 1 )
                                 : (dyVal > tg67x ) ? make_int2( idx, idx )
                                                    : make_int2( idx - signVal, idx + signVal );
        int2 y = (dyVal < tg22x) ? make_int2( idy, idy )
                                 : make_int2( idy - 1, idy + 1 );

        x.x = clamp( x.x, dx.cols );
        x.y = clamp( x.y, dx.cols );
        y.x = clamp( y.x, dx.rows );
        y.y = clamp( y.y, dx.rows );

        if( magVal > mag.ptr(y.x)[x.x] && mavVal >= mag.ptr(y.y)[x.y] ) {
            edge_type = 1 + (int)(magVal > high_thresh);
        }
    }

    map.ptr(idy)[idx] = edge_type;
}


//////////////////////////////////////////////////////////////////////////////////////////

namespace canny
{
    __device__ int counter = 0;

    __device__ __forceinline__ bool checkIdx(int y, int x, int rows, int cols)
    {
        return (y >= 0) && (y < rows) && (x >= 0) && (x < cols);
    }

    __global__ void edgesHysteresisLocalKernel(PtrStepSzi map, short2* st)
    {
        __shared__ volatile int smem[18][18];

        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        smem[threadIdx.y + 1][threadIdx.x + 1] = checkIdx(y, x, map.rows, map.cols) ? map(y, x) : 0;
        if (threadIdx.y == 0)
            smem[0][threadIdx.x + 1] = checkIdx(y - 1, x, map.rows, map.cols) ? map(y - 1, x) : 0;
        if (threadIdx.y == blockDim.y - 1)
            smem[blockDim.y + 1][threadIdx.x + 1] = checkIdx(y + 1, x, map.rows, map.cols) ? map(y + 1, x) : 0;
        if (threadIdx.x == 0)
            smem[threadIdx.y + 1][0] = checkIdx(y, x - 1, map.rows, map.cols) ? map(y, x - 1) : 0;
        if (threadIdx.x == blockDim.x - 1)
            smem[threadIdx.y + 1][blockDim.x + 1] = checkIdx(y, x + 1, map.rows, map.cols) ? map(y, x + 1) : 0;
        if (threadIdx.x == 0 && threadIdx.y == 0)
            smem[0][0] = checkIdx(y - 1, x - 1, map.rows, map.cols) ? map(y - 1, x - 1) : 0;
        if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0)
            smem[0][blockDim.x + 1] = checkIdx(y - 1, x + 1, map.rows, map.cols) ? map(y - 1, x + 1) : 0;
        if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1)
            smem[blockDim.y + 1][0] = checkIdx(y + 1, x - 1, map.rows, map.cols) ? map(y + 1, x - 1) : 0;
        if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1)
            smem[blockDim.y + 1][blockDim.x + 1] = checkIdx(y + 1, x + 1, map.rows, map.cols) ? map(y + 1, x + 1) : 0;

        __syncthreads();

        if (x >= map.cols || y >= map.rows)
            return;

        int n;

        #pragma unroll
        for (int k = 0; k < 16; ++k)
        {
            n = 0;

            if (smem[threadIdx.y + 1][threadIdx.x + 1] == 1)
            {
                n += smem[threadIdx.y    ][threadIdx.x    ] == 2;
                n += smem[threadIdx.y    ][threadIdx.x + 1] == 2;
                n += smem[threadIdx.y    ][threadIdx.x + 2] == 2;

                n += smem[threadIdx.y + 1][threadIdx.x    ] == 2;
                n += smem[threadIdx.y + 1][threadIdx.x + 2] == 2;

                n += smem[threadIdx.y + 2][threadIdx.x    ] == 2;
                n += smem[threadIdx.y + 2][threadIdx.x + 1] == 2;
                n += smem[threadIdx.y + 2][threadIdx.x + 2] == 2;
            }

            __syncthreads();

            if (n > 0)
                smem[threadIdx.y + 1][threadIdx.x + 1] = 2;

            __syncthreads();
        }

        const int e = smem[threadIdx.y + 1][threadIdx.x + 1];

        map(y, x) = e;

        n = 0;

        if (e == 2)
        {
            n += smem[threadIdx.y    ][threadIdx.x    ] == 1;
            n += smem[threadIdx.y    ][threadIdx.x + 1] == 1;
            n += smem[threadIdx.y    ][threadIdx.x + 2] == 1;

            n += smem[threadIdx.y + 1][threadIdx.x    ] == 1;
            n += smem[threadIdx.y + 1][threadIdx.x + 2] == 1;

            n += smem[threadIdx.y + 2][threadIdx.x    ] == 1;
            n += smem[threadIdx.y + 2][threadIdx.x + 1] == 1;
            n += smem[threadIdx.y + 2][threadIdx.x + 2] == 1;
        }

        if (n > 0)
        {
            const int ind =  ::atomicAdd(&counter, 1);
            st[ind] = make_short2(x, y);
        }
    }

    void edgesHysteresisLocal(PtrStepSzi map, short2* st1, hipStream_t stream)
    {
        void* counter_ptr;
        cudaSafeCall( hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(counter)) );

        cudaSafeCall( hipMemsetAsync(counter_ptr, 0, sizeof(int), stream) );

        const dim3 block(16, 16);
        const dim3 grid(divUp(map.cols, block.x), divUp(map.rows, block.y));

        edgesHysteresisLocalKernel<<<grid, block, 0, stream>>>(map, st1);
        cudaSafeCall( hipGetLastError() );

        if (stream == NULL)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

//////////////////////////////////////////////////////////////////////////////////////////

namespace canny
{
    __constant__ int c_dx[8] = {-1,  0,  1, -1, 1, -1, 0, 1};
    __constant__ int c_dy[8] = {-1, -1, -1,  0, 0,  1, 1, 1};

    __global__ void edgesHysteresisGlobalKernel(PtrStepSzi map, short2* st1, short2* st2, const int count)
    {
        const int stack_size = 512;

        __shared__ int s_counter;
        __shared__ int s_ind;
        __shared__ short2 s_st[stack_size];

        if (threadIdx.x == 0)
            s_counter = 0;

        __syncthreads();

        int ind = blockIdx.y * gridDim.x + blockIdx.x;

        if (ind >= count)
            return;

        short2 pos = st1[ind];

        if (threadIdx.x < 8)
        {
            pos.x += c_dx[threadIdx.x];
            pos.y += c_dy[threadIdx.x];

            if (pos.x > 0 && pos.x < map.cols - 1 && pos.y > 0 && pos.y < map.rows - 1 && map(pos.y, pos.x) == 1)
            {
                map(pos.y, pos.x) = 2;

                ind = Emulation::smem::atomicAdd(&s_counter, 1);

                s_st[ind] = pos;
            }
        }

        __syncthreads();

        while (s_counter > 0 && s_counter <= stack_size - blockDim.x)
        {
            const int subTaskIdx = threadIdx.x >> 3;
            const int portion = ::min(s_counter, blockDim.x >> 3);

            if (subTaskIdx < portion)
                pos = s_st[s_counter - 1 - subTaskIdx];

            __syncthreads();

            if (threadIdx.x == 0)
                s_counter -= portion;

            __syncthreads();

            if (subTaskIdx < portion)
            {
                pos.x += c_dx[threadIdx.x & 7];
                pos.y += c_dy[threadIdx.x & 7];

                if (pos.x > 0 && pos.x < map.cols - 1 && pos.y > 0 && pos.y < map.rows - 1 && map(pos.y, pos.x) == 1)
                {
                    map(pos.y, pos.x) = 2;

                    ind = Emulation::smem::atomicAdd(&s_counter, 1);

                    s_st[ind] = pos;
                }
            }

            __syncthreads();
        }

        if (s_counter > 0)
        {
            if (threadIdx.x == 0)
            {
                s_ind = ::atomicAdd(&counter, s_counter);

                if (s_ind + s_counter > map.cols * map.rows)
                    s_counter = 0;
            }

            __syncthreads();

            ind = s_ind;

            for (int i = threadIdx.x; i < s_counter; i += blockDim.x)
                st2[ind + i] = s_st[i];
        }
    }

    void edgesHysteresisGlobal(PtrStepSzi map, short2* st1, short2* st2, hipStream_t stream)
    {
        void* counter_ptr;
        cudaSafeCall( hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(canny::counter)) );

        int count;
        cudaSafeCall( hipMemcpyAsync(&count, counter_ptr, sizeof(int), hipMemcpyDeviceToHost, stream) );
        cudaSafeCall( hipStreamSynchronize(stream) );

        while (count > 0)
        {
            cudaSafeCall( hipMemsetAsync(counter_ptr, 0, sizeof(int), stream) );

            const dim3 block(128);
            const dim3 grid(::min(count, 65535u), divUp(count, 65535), 1);

            edgesHysteresisGlobalKernel<<<grid, block, 0, stream>>>(map, st1, st2, count);
            cudaSafeCall( hipGetLastError() );

            if (stream == NULL)
                cudaSafeCall( hipDeviceSynchronize() );

            cudaSafeCall( hipMemcpyAsync(&count, counter_ptr, sizeof(int), hipMemcpyDeviceToHost, stream) );
            cudaSafeCall( hipStreamSynchronize(stream) );

            count = min(count, map.cols * map.rows);

            //std::swap(st1, st2);
            short2* tmp = st1;
            st1 = st2;
            st2 = tmp;
        }
    }
}

//////////////////////////////////////////////////////////////////////////////////////////

namespace canny
{
    struct GetEdges : unary_function<int, uchar>
    {
        __device__ __forceinline__ uchar operator ()(int e) const
        {
            return (uchar)(-(e >> 1));
        }

        __host__ __device__ __forceinline__ GetEdges() {}
        __host__ __device__ __forceinline__ GetEdges(const GetEdges&) {}
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits<canny::GetEdges> : DefaultTransformFunctorTraits<canny::GetEdges>
    {
        enum { smart_shift = 4 };
    };
}}}

namespace canny
{
    void getEdges(PtrStepSzi map, PtrStepSzb dst, hipStream_t stream)
    {
        transform(map, dst, GetEdges(), WithOutMask(), stream);
    }
}

#endif /* CUDA_DISABLER */

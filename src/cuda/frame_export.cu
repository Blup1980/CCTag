#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "frame.h"

namespace popart {

using namespace std;

bool Frame::applyExport( cctag::EdgePointsImage&         edgesMap,
                         std::vector<cctag::EdgePoint*>& seeds,
                         cctag::WinnerMap&               winners )
{
    // cerr << "Enter " << __FUNCTION__ << endl;

    int vote_sz = _vote._chained_edgecoords.host.size;
    int all_sz  = _vote._all_edgecoords.host.size;

    if( vote_sz <= 0 ) {
        // no voting happened, no need for edge linking,
        // so no need for copying anything
        // cerr << "Leave " << __FUNCTION__ << endl;
        return false;
    }

    edgesMap.resize( boost::extents[ _d_plane.cols ][ _d_plane.rows ] );
    std::fill( edgesMap.origin(), edgesMap.origin() + edgesMap.size(), (cctag::EdgePoint*)NULL );

    cctag::EdgePoint* array = new cctag::EdgePoint[ all_sz ];
    for( int i=0; i<all_sz; i++ ) {
        const int2&   pt = _vote._all_edgecoords.host.ptr[i];
        const int16_t dx = _h_dx.ptr(pt.y)[pt.x];
        const int16_t dy = _h_dy.ptr(pt.y)[pt.x];

        array[i].init( pt.x, pt.y, dx, dy );

        edgesMap[pt.x][pt.y] = &array[i];
    }
    for( int i=1; i<vote_sz; i++ ) {
        const TriplePoint& pt = _vote._chained_edgecoords.host.ptr[i];
        cctag::EdgePoint* ep = edgesMap[pt.coord.x][pt.coord.y];
        assert( ep != 0 );
        assert( ep->_grad.getX() == (double)pt.d.x );
        assert( ep->_grad.getY() == (double)pt.d.y );

        if( pt.descending.after.x != 0 || pt.descending.after.y != 0 ) {
            cctag::EdgePoint* n = edgesMap[pt.descending.after.x][pt.descending.after.y];
            if( n != 0 )
                ep->_after = n;
        }
        if( pt.descending.befor.x != 0 || pt.descending.befor.y != 0 ) {
            cctag::EdgePoint* n = edgesMap[pt.descending.befor.x][pt.descending.befor.y];
            if( n != 0 )
                ep->_before = n;
        }

        ep->_flowLength = pt._flowLength;
        ep->_isMax      = pt._winnerSize;
    }

    // NVCC handles the std::list<...>() construct. GCC does not. Keeping alternative code.
    // std::list<cctag::EdgePoint*> empty_list;
    int* seed_array = _vote._seed_indices.host.ptr;
    for( int i=0; i<_vote._seed_indices.host.size; i++ ) {
        const TriplePoint& pt = _vote._chained_edgecoords.host.ptr[ seed_array[i] ];
        cctag::EdgePoint* ep = edgesMap[pt.coord.x][pt.coord.y];
        seeds.push_back( ep );

        // winners.insert( std::pair<cctag::EdgePoint*,std::list<cctag::EdgePoint*> >( ep, empty_list ) );
        winners.insert( std::pair<cctag::EdgePoint*,
                                  std::list<cctag::EdgePoint*> >( ep, std::list<cctag::EdgePoint*>() ) );
    }

    for( int i=1; i<vote_sz; i++ ) {
        const TriplePoint& pt = _vote._chained_edgecoords.host.ptr[i];

        if( pt.my_vote != 0 ) {
            const TriplePoint& point = _vote._chained_edgecoords.host.ptr[ pt.my_vote ];
            cctag::EdgePoint* potential_seed = edgesMap[point.coord.x][point.coord.y];
            if( winners.find(potential_seed) != winners.end() ) {
                cctag::EdgePoint* this_voter = edgesMap[pt.coord.x][pt.coord.y];
                winners[potential_seed].push_back( this_voter );
            }
        }
    }
#ifndef NDEBUG
#if 0
    std::sort(seeds.begin(), seeds.end(), cctag::receivedMoreVoteThan);

    std::vector<cctag::EdgePoint*>::const_iterator it  = seeds.begin();
    std::vector<cctag::EdgePoint*>::const_iterator end = seeds.end();
    for( ; it!=end; it++ ) {
        cctag::EdgePoint* ep = *it;
        cout << "  " << *ep << " FL=" << ep->_flowLength
                 << " VT=" << ep->_isMax
                 << " voters=";
        std::list<cctag::EdgePoint*>::const_iterator vit  = winners[ep].begin();
        std::list<cctag::EdgePoint*>::const_iterator vend = winners[ep].end();
        for( ; vit!=vend; vit++ ) {
            cout << "(" << (*vit)->getX() << "," << (*vit)->getY() << ") ";
        }
        cout << endl;
    }
#endif
#endif // NDEBUG
    // cerr << "Leave " << __FUNCTION__ << endl;
    return true;
}

}; // namespace popart


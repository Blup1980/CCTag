#include "onoff.h"

// #include <iostream>
// #include <algorithm>
// #include <limits>
#include <hip/hip_runtime.h>
// #include <hipcub/hipcub.hpp>
// #include <thrust/system/cuda/detail/hipcub/hipcub.hpp>
// #include <stdio.h>
// #include "debug_macros.hpp"
// #include "debug_is_on_edge.h"

#include "frame.h"
// #include "assist.h"

using namespace std;

namespace popart
{

/* After vote_eval_chosen, _voters is no longer changed
 * we can copy it to the host for edge linking
 */

__host__
void Frame::applyVoteDownload( )
{
    /* After vote_eval_chosen, _voters is no longer changed
     * we can copy it to the host for edge linking
     */
    _voters.copySizeFromDevice( _stream, EdgeListWait );
    _voters.copyDataFromDeviceAsync( _download_stream );
    _v_chosen_idx.copyDataFromDeviceAsync( _voters.host.size, _download_stream );
    _inner_points.copyDataFromDeviceAsync( _download_stream );
}

} // namespace popart


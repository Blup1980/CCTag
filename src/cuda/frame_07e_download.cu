#include "onoff.h"

// #include <iostream>
// #include <algorithm>
// #include <limits>
#include <hip/hip_runtime.h>
// #include <hipcub/hipcub.hpp>
// #include <thrust/system/cuda/detail/hipcub/hipcub.hpp>
// #include <stdio.h>
// #include "debug_macros.hpp"
// #include "debug_is_on_edge.h"

#include "frame.h"
// #include "assist.h"

using namespace std;

namespace popart
{

/* After vote_eval_chosen, _chained_edgecoords is no longer changed
 * we can copy it to the host for edge linking
 */

__host__
void Frame::applyVoteDownload( )
{
#ifdef EDGE_LINKING_HOST_SIDE
    /* After vote_eval_chosen, _chained_edgecoords is no longer changed
     * we can copy it to the host for edge linking
     */
    _vote._chained_edgecoords.copySizeFromDevice( _stream, EdgeListWait );
    _vote._chained_edgecoords.copyDataFromDeviceAsync( _download_stream );
    _vote._seed_indices.      copyDataFromDeviceAsync( _download_stream );
#endif // EDGE_LINKING_HOST_SIDE
}

} // namespace popart


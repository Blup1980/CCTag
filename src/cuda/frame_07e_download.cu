#include "onoff.h"

// #include <iostream>
// #include <algorithm>
// #include <limits>
#include <hip/hip_runtime.h>
// #include <hipcub/hipcub.hpp>
// #include <thrust/system/cuda/detail/hipcub/hipcub.hpp>
// #include <stdio.h>
// #include "debug_macros.hpp"
// #include "debug_is_on_edge.h"

#include "frame.h"
// #include "assist.h"

using namespace std;

namespace popart
{

/* After vote_eval_chosen, _voters is no longer changed
 * we can copy it to the host for edge linking
 */

__host__
void Frame::applyVoteDownload( )
{
#ifdef EDGE_LINKING_HOST_SIDE
    /* After vote_eval_chosen, _voters is no longer changed
     * we can copy it to the host for edge linking
     */
    _voters.copySizeFromDevice( _stream, EdgeListWait );
    _voters.copyDataFromDeviceAsync( _download_stream );
    _inner_points.copyDataFromDeviceAsync( _download_stream );

    POP_CUDA_MEMCPY_TO_HOST_ASYNC( _h_voters_my_vote, _voters_my_vote, _voters.host.size*sizeof(int), _download_stream );
#endif // EDGE_LINKING_HOST_SIDE
}

} // namespace popart


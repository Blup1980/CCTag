#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "debug_macros.hpp"

#include "frame.h"
#include "clamp.h"
#include "assist.h"

namespace popart
{

using namespace std;

namespace hysteresis
{
#define HYST_H   32
#define HYST_W   32

#if HYST_W < HYST_H
#error The code requires W<=32 and H<=W
#endif

__shared__ volatile uint8_t array[HYST_H+2][4*(HYST_W+2)];

__device__
inline
uint32_t get( cv::cuda::PtrStepSz32u img, const int idx, const int idy )
{
    if( idx < 0 || idy < 0 || idx >= img.cols || idy >= img.rows ) {
        return 0;
    }
    return img.ptr(idy)[idx];
}

__device__
void load( cv::cuda::PtrStepSz32u img )
{
    const int srcidx = blockIdx.x * HYST_W + threadIdx.x;
    const int srcidy = blockIdx.y * HYST_H + threadIdx.y;
    uint32_t v;

    volatile uint32_t* load_line;
    load_line = reinterpret_cast<volatile uint32_t*>(&array[threadIdx.y][0]);

    v = get( img, srcidx-1, srcidy-1 );
    load_line[threadIdx.x  ] = v;

    if( threadIdx.x >= HYST_W - 2 ) {
        v = get( img, srcidx+1, srcidy-1 );
        load_line[threadIdx.x+2] = v;
    }
    __syncthreads();

    if( threadIdx.y >= HYST_H - 2 ) {
        load_line = reinterpret_cast<volatile uint32_t*>(&array[threadIdx.y+2][0]);

        v = get( img, srcidx-1, srcidy+1 );
        load_line[threadIdx.x  ] = v;

        if( threadIdx.x >= HYST_W - 2 ) {
            v = get( img, srcidx+1, srcidy+1 );
            load_line[threadIdx.x+2] = v;
        }
    }
    __syncthreads();
}

__device__
void store( cv::cuda::PtrStepSz32u img, bool printout )
{
    const int dstidx  = blockIdx.x * HYST_W + threadIdx.x;
    const int dstidy  = blockIdx.y * HYST_H + threadIdx.y;

#if 1
    if( dstidx*sizeof(uint32_t) < img.step && dstidy < img.rows ) {
        // volatile uint32_t* shared_line = reinterpret_cast<volatile uint32_t*>(array[threadIdx.y+1]);
        volatile uint32_t* shared_line = reinterpret_cast<volatile uint32_t*>(&array[threadIdx.y+1][0]);
        uint32_t val = shared_line[threadIdx.x+1];

        img.ptr(dstidy)[dstidx] = val;
    }
#else
    union {
        uint8_t  b[4];
        uint32_t i;
    } val;

    val.b[0] = array[threadIdx.y+1][threadIdx.x*4+4];
    val.b[1] = array[threadIdx.y+1][threadIdx.x*4+5];
    val.b[2] = array[threadIdx.y+1][threadIdx.x*4+6];
    val.b[3] = array[threadIdx.y+1][threadIdx.x*4+7];
    if( printout ) {
        printf("(%d,%d)<-%d  (%d,%d)<-%d  (%d,%d)<-%d  (%d,%d)<-%d\n",
               dstidx*4+0, dstidy, val.b[0],
               dstidx*4+1, dstidy, val.b[1],
               dstidx*4+2, dstidy, val.b[2],
               dstidx*4+3, dstidy, val.b[3]);
    }
    if( dstidx*sizeof(uint32_t) < img.step && dstidy < img.rows ) {
        img.ptr(dstidy)[dstidx] = val.i;
    }
#endif
}

__device__
inline
bool update_edge_pixel( int y, int x )
{
    bool something_changed = false;

    union {
        uint8_t  b[12];
        uint3    i;
    } val[3];

    val[0].i = make_uint3( reinterpret_cast<volatile uint32_t*>( &array[y  ][x] )[0],
                           reinterpret_cast<volatile uint32_t*>( &array[y  ][x] )[1],
                           reinterpret_cast<volatile uint32_t*>( &array[y  ][x] )[2] );
    val[1].i = make_uint3( reinterpret_cast<volatile uint32_t*>( &array[y+1][x] )[0],
                           reinterpret_cast<volatile uint32_t*>( &array[y+1][x] )[1],
                           reinterpret_cast<volatile uint32_t*>( &array[y+1][x] )[2] );
    val[2].i = make_uint3( reinterpret_cast<volatile uint32_t*>( &array[y+1][x] )[0],
                           reinterpret_cast<volatile uint32_t*>( &array[y+2][x] )[1],
                           reinterpret_cast<volatile uint32_t*>( &array[y+2][x] )[2] );

    for( int i=0; i<4; i++ ) {
        bool inc = false;
        bool dec = false;

        if( val[1].b[4+i] == 1 ) {
            inc = ( val[0].b[3+i] == 2 || val[0].b[4+i] == 2 || val[0].b[5+i] == 2 ||
                    val[1].b[3+i] == 2 ||                       val[1].b[5+i] == 2 ||
                    val[2].b[3+i] == 2 || val[2].b[4+i] == 2 || val[2].b[5+i] == 2 );
            dec = ( val[0].b[3+i] == 0 && val[0].b[4+i] == 0 && val[0].b[5+i] == 0 &&
                    val[1].b[3+i] == 0 &&                       val[1].b[5+i] == 0 &&
                    val[2].b[3+i] == 0 && val[2].b[4+i] == 0 && val[2].b[5+i] == 0 );
            val[1].b[4+i] = inc ? 2 : dec ? 0 : 1 ;
        }
        __syncthreads();

        something_changed |= inc;
        something_changed |= dec;
    }
    reinterpret_cast<volatile uint32_t*>( &array[y+1][x] )[1] = val[1].i.y;

    return something_changed;
}

__device__
bool edge_block_loop( int debug_roundcount )
{
    __shared__ volatile bool continuation[HYST_H];
    bool            again = true;
    bool            something_changed = false;
    int debug_inner_loop_count = 0;

    // DEBUG NOTE:
    // updating in the inner loop works correctly
    // but the outer loop repeats exactly changes in the inner loop, in particular on
    // x values of 0, 1 or 2
    // WHY ?

    while( again ) { // && debug_inner_loop_count < 10 ) {
        assert( debug_inner_loop_count <= HYST_W*HYST_H );

        bool mark = update_edge_pixel( threadIdx.y, threadIdx.x*4 );

        /* every row checks whether any pixel has been changed */
        bool line_changed = __any( mark );

#if 0
        /* the first thread of each row write the result to continuation[] */
        if( threadIdx.x == 0 ) continuation[threadIdx.y] = line_changed;

        /* make sure all updated pixel are written back to
         * shared memory before continuation[] is modified.
         * This is supposedly redundant with __syncthreads() */
        __threadfence_block();

        /* wait for all rows to fulfill the operation (and to assure that
         * results in continuation[] are visible to all threads, because
         * threadfence() is implied by syncthreads() */
        __syncthreads();

        /* Each thread in a warp reads __any() results for one of 32 warps.
         * Redundant, but I have no better idea for spreading the result
         * to all warps. */
        // mark = threadIdx.x < HYST_H ? continuation[threadIdx.x] : false;
        mark = continuation[threadIdx.x]; // each warp reads result for all 32 warps

        /* Finally, all 32x32 threads know whether at least one of them
         * has changed a pixel.
         * If there has been any change in this round, try to spread
         * the change further.
         */
        again = __any( mark );
#else
        if( threadIdx.x == 0 ) continuation[threadIdx.y] = line_changed;
        __syncthreads();
        if( threadIdx.y == 0 ) {
            mark = continuation[threadIdx.x];
            again = __any(mark);
            if( threadIdx.x == 0 ) {
                continuation[0] = again;
            }
        }
        __syncthreads();
        again = continuation[0];
#endif

        /* Every threads needs to know whether any pixel was changed in
         * any round of the loop because egde_second() uses this return
         * value to write back to global memory using a different alignment. */
        if( again ) something_changed = true;

        /* this should not be necessary ... */
        debug_inner_loop_count++;
    }

    return something_changed;
}

__device__
bool edge( int* block_counter, int debug_roundcount )
{
    bool something_changed = edge_block_loop( debug_roundcount );
    if( threadIdx.x == 0 && threadIdx.y == 0 ) {
        if( something_changed ) {
            if( debug_roundcount > 25 ) {
                printf("Something changed in block (%d,%d)\n", blockIdx.x, blockIdx.y );
            }
            atomicAdd( block_counter, 1 );
        }
    }
    return something_changed;
}

__global__
void edge_first( cv::cuda::PtrStepSzb img, int* block_counter, cv::cuda::PtrStepSzb src, int debug_roundcount )
{
    // const int idx  = blockIdx.x * HYST_W + threadIdx.x;
    // const int idy  = blockIdx.y * HYST_H + threadIdx.y;
    // if( outOfBounds( idx, idy, img ) ) return;
    // uint8_t val = src.ptr(idy)[idx];
    // img.ptr(idy)[idx] = val;
    cv::cuda::PtrStepSz32u input;
    input.data = reinterpret_cast<uint32_t*>(src.data);
    input.step = src.step;
    input.rows = src.rows;
    input.cols = src.cols / 4;
    load( input );

    edge( block_counter, debug_roundcount );

    __syncthreads();

    cv::cuda::PtrStepSz32u output;
    output.data = reinterpret_cast<uint32_t*>(img.data);
    output.step = img.step;
    output.rows = img.rows;
    output.cols = img.cols / 4;
    store( output, false );
}

__global__
void edge_second( cv::cuda::PtrStepSzb img, int* block_counter, int debug_roundcount )
{
    cv::cuda::PtrStepSz32u input;
    input.data = reinterpret_cast<uint32_t*>(img.data);

    input.step = img.step;
    input.rows = img.rows;
    input.cols = img.cols / 4;
    load( input );

    bool something_changed = edge( block_counter, debug_roundcount );

    if( __any( something_changed ) ) {
        store( input, false );
    }
}

}; // namespace hysteresis

#ifndef NDEBUG
__global__
void verify_map_valid( cv::cuda::PtrStepSzb img, cv::cuda::PtrStepSzb ver, int w, int h )
{
    assert( img.cols == w );
    assert( img.rows == h );
    assert( ver.cols == w );
    assert( ver.rows == h );

    const int idx  = blockIdx.x * HYST_W + threadIdx.x;
    const int idy  = blockIdx.y * HYST_H + threadIdx.y;
    uint32_t x = clamp( idx, img.cols );
    uint32_t y = clamp( idy, img.rows );
    uint8_t  val = img.ptr(y)[x];
    if( val > 2 ) {
        printf("idx=%d -> x=%d, idy=%d -> y=%d, img.cols=%d img.rows=%d val=%d\n",
            idx, x, idy, y, img.cols, img.rows, val );
        assert( val <= 2 );
    }
}
#endif // NDEBUG

#if defined(USE_SEPARABLE_COMPILATION)
__global__
void hyst_outer_loop( int width, int height, int* block_counter, cv::cuda::PtrStepSzb img, cv::cuda::PtrStepSzb src )
{
    printf( "Enter %s\n", __FUNCTION__ );

    dim3 block;
    dim3 grid;
    block.x = HYST_W;
    block.y = HYST_H;
    grid.x  = grid_divide( width,   HYST_W * 4 );
    grid.y  = grid_divide( height,  HYST_H );

    printf( "Starting (%d,%d,%d)-grid of (%d,%d,%d) threads\n",
            grid.x, grid.y, grid.z, block.x, block.y, block.z );

    bool first_time = true;
    int debug_roundcount = 0;
    do
    {
        *block_counter = 0;
        if( first_time ) {
            hysteresis::edge_first
                <<<grid,block>>>
                ( img,
                  block_counter,
                  src,
                  debug_roundcount );
            first_time = false;
        } else {
            hysteresis::edge_second
                <<<grid,block>>>
                ( img,
                  block_counter,
                  debug_roundcount );
        }
        hipDeviceSynchronize( );
        printf( "width=%d height=%d block_counter=%d\n", width, height, *block_counter);
        assert( *block_counter <= grid.x * grid.y );
    }
    while( *block_counter > 0 && debug_roundcount++ < 30 ); // *block_counter > 0 );
    printf( "Leave %s\n", __FUNCTION__ );
}
#endif // USE_SEPARABLE_COMPILATION

__host__
void Frame::applyHyst( const cctag::Parameters & params )
{
    cerr << "Enter " << __FUNCTION__ << endl;
    assert( getWidth()  == _d_map.cols );
    assert( getHeight() == _d_map.rows );
    assert( getWidth()  == _d_hyst_edges.cols );
    assert( getHeight() == _d_hyst_edges.rows );

#ifndef NDEBUG
    dim3 block;
    dim3 grid;
    block.x = HYST_W;
    block.y = HYST_H;
    grid.x  = grid_divide( getWidth(),   HYST_W );
    grid.y  = grid_divide( getHeight(),  HYST_H );

    verify_map_valid
        <<<grid,block,0,_stream>>>
        ( _d_map, _d_hyst_edges, getWidth(), getHeight() );
#endif

#if defined(USE_SEPARABLE_COMPILATION)
    hipEvent_t before_hyst, after_hyst;
    float ms;

    hipEventCreate( &before_hyst );
    hipEventCreate( &after_hyst );
    hipEventRecord( before_hyst, _stream );
    cerr << "0" << endl;
    hyst_outer_loop
        <<<1,1,0,_stream>>>
        ( getWidth(), getHeight(), _d_hysteresis_block_counter, _d_hyst_edges, _d_map );
    hipEventRecord( after_hyst, _stream );
    cerr << "0.1" << endl;
    hipEventSynchronize( after_hyst );
    cerr << "0.2" << endl;
    hipEventElapsedTime( &ms, before_hyst, after_hyst );
    hipEventDestroy( before_hyst );
    hipEventDestroy( after_hyst );
    std::cerr << "Hyst took " << ms << " ms" << std::endl;
#else // USE_SEPARABLE_COMPILATION
    bool first_time = true;
    int block_counter;
    do
    {
        block_counter = grid.x * grid.y;
        POP_CUDA_MEMCPY_TO_DEVICE_ASYNC( _d_hysteresis_block_counter,
                                         &block_counter,
                                         sizeof(int), _stream );
        if( first_time ) {
            cerr << "1" << endl;
            hysteresis::edge_first
                <<<grid,block,0,_stream>>>
                ( _d_hyst_edges,
                  _d_hysteresis_block_counter,
                  _d_map );
            first_time = false;
        } else {
            cerr << "2" << endl;
            hysteresis::edge_second
                <<<grid,block,0,_stream>>>
                ( _d_hyst_edges,
                  _d_hysteresis_block_counter );
        }
        POP_CHK_CALL_IFSYNC;

        POP_CUDA_MEMCPY_TO_HOST_ASYNC( &block_counter,
                                       _d_hysteresis_block_counter,
                                       sizeof(int), _stream );
        POP_CUDA_SYNC( _stream );
        cerr << "block_counter=" << block_counter << endl;
    }
    while( block_counter > 0 );
#endif // USE_SEPARABLE_COMPILATION
    cerr << "Leave " << __FUNCTION__ << endl;
}

}; // namespace popart


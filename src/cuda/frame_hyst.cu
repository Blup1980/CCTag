#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "debug_macros.hpp"

#include "frame.h"
#include "clamp.h"
#include "assist.h"

namespace popart
{

using namespace std;

namespace hysteresis
{
#define HYST_H   32
#define HYST_W   32

#if HYST_W < HYST_H
#error The code requires W<=32 and H<=W
#endif

__shared__ volatile uint8_t array[HYST_H+2][4*(HYST_W+2)];

__device__
inline
uint32_t get( cv::cuda::PtrStepSz32u img, const int idx, const int idy )
{
#if 1
    return img.ptr( clamp( idy, img.rows ) )[ clamp( idx, img.cols ) ];
#else
    int x = clamp( idx, img.cols );
    int y = clamp( idy, img.rows );
    assert( x >= 0 );
    assert( y >= 0 );
    assert( x < img.cols );
    assert( y < img.rows );
    uint8_t  val = img.ptr(y)[x];
    if( val > 2 ) {
        printf("idx=%d -> x=%d, idy=%d -> y=%d, img.cols=%d img.rows=%d val=%d\n",
            idx, x, idy, y, img.cols, img.rows, val );
        assert( val <= 2 );
    }
    return val;
#endif
}

__device__
void load( cv::cuda::PtrStepSz32u img )
{
    const int srcidx = blockIdx.x * HYST_W + threadIdx.x;
    const int srcidy = blockIdx.y * HYST_H + threadIdx.y;

    uint32_t val_0_0;
    val_0_0 = get( img, srcidx-1, srcidy-1 );

    volatile uint32_t* reinterpret_array;
    reinterpret_array = reinterpret_cast<volatile uint32_t*>(&array[threadIdx.y][0]);
    reinterpret_array[threadIdx.x  ] = val_0_0;

    if( threadIdx.y >= HYST_H - 2 ) {
        uint32_t val_2_0;
        val_2_0 = get( img, srcidx-1, srcidy+1 );
        reinterpret_array = reinterpret_cast<volatile uint32_t*>(&array[threadIdx.y+2][0]);
        reinterpret_array[threadIdx.x  ] = val_2_0;
        if( threadIdx.x >= HYST_W - 2 ) {
            uint32_t val_2_2;
            val_2_2 = get( img, srcidx+1, srcidy+1 );
            reinterpret_array[threadIdx.x+2] = val_2_2;
        }
    }
    __syncthreads();
    if( threadIdx.x >= HYST_W - 2 ) {
        uint32_t val_0_2;
        val_0_2 = get( img, srcidx+1, srcidy-1 );
        reinterpret_array = reinterpret_cast<volatile uint32_t*>(&array[threadIdx.y][0]);
        reinterpret_array[threadIdx.x+2] = val_0_2;
    }
    __syncthreads();
}

__device__
void store( cv::cuda::PtrStepSz32u img )
{
    const int dstidx  = blockIdx.x * HYST_W + threadIdx.x;
    const int dstidy  = blockIdx.y * HYST_H + threadIdx.y;

    volatile uint32_t* reinterpret_array;

    volatile uint32_t val;
    reinterpret_array = reinterpret_cast<volatile uint32_t*>(&array[threadIdx.y+1][0]);
    val = reinterpret_array[threadIdx.x+1];

    if( dstidx < img.cols && dstidy < img.rows ) {
        img.ptr(dstidy)[dstidx] =  val;
    }
}

__device__
inline
bool update_edge_pixel( int y, int x )
{
    uint8_t val[3][3];
    volatile uint8_t debug_v;
    val[0][0] = array[y  ][x  ];
    val[0][1] = array[y  ][x+1];
    val[0][2] = array[y  ][x+2];
    val[1][0] = array[y+1][x  ];
    val[1][1] = array[y+1][x+1];
    val[1][2] = array[y+1][x+2];
    val[2][0] = array[y+2][x  ];
    val[2][1] = array[y+2][x+1];
    val[2][2] = array[y+2][x+2];

    assert( val[0][0] <= 2 );
    assert( val[0][1] <= 2 );
    assert( val[0][2] <= 2 );
    assert( val[1][0] <= 2 );
    assert( val[1][1] <= 2 );
    assert( val[1][2] <= 2 );
    assert( val[2][0] <= 2 );
    assert( val[2][1] <= 2 );
    assert( val[2][2] <= 2 );

    bool inc = false;
    bool dec = false;

    debug_v = val[1][1];
    if( val[1][1] == 1 ) {
        inc = ( val[0][0] == 2 || val[0][1] == 2 || val[0][2] == 2 ||
                val[1][0] == 2 ||                   val[1][2] == 2 ||
                val[2][0] == 2 || val[2][1] == 2 || val[2][2] == 2 );
        dec = ( val[0][0] == 0 && val[0][1] == 0 && val[0][2] == 0 &&
                val[1][0] == 0 &&                   val[1][2] == 0 &&
                val[2][0] == 0 && val[2][1] == 0 && val[2][2] == 0 );
        debug_v = val[1][1] = inc ? 2 : dec ? 0 : 1 ;
    }
    __syncthreads();
    assert( debug_v == val[1][1] );
    array[y+1][x+1] = val[1][1];

    return ( inc || dec );
}

__device__
bool edge_block_loop( )
{
    __shared__ volatile bool continuation[HYST_H];
    bool            again = true;
    bool            something_changed = false;
    bool            line_changed = false;
    int ct = 0;

    while( again && ct < 10 ) {
        assert( ct <= HYST_W*HYST_H );
        bool mark = false;
        mark = mark || update_edge_pixel( threadIdx.y, threadIdx.x    );
        mark = mark || update_edge_pixel( threadIdx.y, threadIdx.x+32 );
        mark = mark || update_edge_pixel( threadIdx.y, threadIdx.x+64 );
        mark = mark || update_edge_pixel( threadIdx.y, threadIdx.x+96 );

        /* make sure all updated pixel are written back to
         * shared memory before continuation[] is modified */
        __threadfence_block();

        /* every row checks whether any pixel has been changed */
        line_changed = __any( mark );

        /* the first thread of each row write the result to continuation[] */
        if( threadIdx.x == 0 ) continuation[threadIdx.y] = line_changed;

        /* wait for all rows to fulfill the operation (and to assure that
         * results in continuation[] are visible to all threads, because
         * threadfence() is implied by syncthreads() */
        __syncthreads();

        /* Each thread in a warp reads continuation for one row.
         * Redundant, but I have no better idea for spreading the result
         * to all warps. */
        mark = threadIdx.x < HYST_H ? continuation[threadIdx.x] : false;

        /* Finally, all 32x32 threads know whether at least one of them
         * has changed a pixel.
         * If there has been any change in this round, try to spread
         * the change further.
         */
        again = __any( mark );

        /* Every threads needs to know whether any pixel was changed in
         * any round of the loop because egde_second() uses this return
         * value to write back to global memory using a different alignment. */
        if( again ) something_changed = true;

        /* this should not be necessary ... */
        ct++;
    }

    return something_changed;
}

__device__
bool edge( int* block_counter )
{
    bool something_changed = edge_block_loop( );
    if( threadIdx.x == 0 && threadIdx.y == 0 ) {
        if( something_changed ) {
            atomicAdd( block_counter, 1 );
        }
    }
    return something_changed;
}

__global__
void edge_first( cv::cuda::PtrStepSzb img, int* block_counter, cv::cuda::PtrStepSzb src )
{
    // const int idx  = blockIdx.x * HYST_W + threadIdx.x;
    // const int idy  = blockIdx.y * HYST_H + threadIdx.y;
    // if( outOfBounds( idx, idy, img ) ) return;
    // uint8_t val = src.ptr(idy)[idx];
    // img.ptr(idy)[idx] = val;
    cv::cuda::PtrStepSz32u input;
    input.data = reinterpret_cast<uint32_t*>(src.data);
    input.step = src.step;
    input.rows = src.rows;
    input.cols = src.cols / 4;
    load( input );

    edge( block_counter );

    __syncthreads();

    cv::cuda::PtrStepSz32u output;
    output.data = reinterpret_cast<uint32_t*>(img.data);
    output.step = img.step;
    output.rows = img.rows;
    output.cols = img.cols / 4;
    store( output );
}

__global__
void edge_second( cv::cuda::PtrStepSzb img, int* block_counter )
{
    cv::cuda::PtrStepSz32u input;
    input.data = reinterpret_cast<uint32_t*>(img.data);

    input.step = img.step;
    input.rows = img.rows;
    input.cols = img.cols / 4;
    load( input );

    bool something_changed = edge( block_counter );

    if( __any( something_changed ) ) {
        store( input );
    }
}

}; // namespace hysteresis

#ifndef NDEBUG
__global__
void verify_map_valid( cv::cuda::PtrStepSzb img, cv::cuda::PtrStepSzb ver, int w, int h )
{
    assert( img.cols == w );
    assert( img.rows == h );
    assert( ver.cols == w );
    assert( ver.rows == h );

    const int idx  = blockIdx.x * HYST_W + threadIdx.x;
    const int idy  = blockIdx.y * HYST_H + threadIdx.y;
    uint32_t x = clamp( idx, img.cols );
    uint32_t y = clamp( idy, img.rows );
    uint8_t  val = img.ptr(y)[x];
    if( val > 2 ) {
        printf("idx=%d -> x=%d, idy=%d -> y=%d, img.cols=%d img.rows=%d val=%d\n",
            idx, x, idy, y, img.cols, img.rows, val );
        assert( val <= 2 );
    }
}
#endif // NDEBUG

#if defined(USE_SEPARABLE_COMPILATION)
__global__
void hyst_outer_loop( int width, int height, int* block_counter, cv::cuda::PtrStepSzb img, cv::cuda::PtrStepSzb src )
{
    printf( "Enter %s\n", __FUNCTION__ );

    dim3 block;
    dim3 grid;
    block.x = HYST_W;
    block.y = HYST_H;
    grid.x  = grid_divide( width,   HYST_W * 4 );
    grid.y  = grid_divide( height,  HYST_H );

    printf( "Starting (%d,%d,%d)-grid of (%d,%d,%d) threads\n",
            grid.x, grid.y, grid.z, block.x, block.y, block.z );

    bool first_time = true;
    int loopcount = 0;
    do
    {
        *block_counter = 0;
        if( first_time ) {
            hysteresis::edge_first
                <<<grid,block>>>
                ( img,
                  block_counter,
                  src );
            first_time = false;
        } else {
            hysteresis::edge_second
                <<<grid,block>>>
                ( img,
                  block_counter );
        }
        hipDeviceSynchronize( );
        printf( "block_counter=%d\n", *block_counter);
        assert( *block_counter <= grid.x * grid.y );
    }
    while( loopcount++ < 30 ); // *block_counter > 0 );
    printf( "Leave %s\n", __FUNCTION__ );
}
#endif // USE_SEPARABLE_COMPILATION

__host__
void Frame::applyHyst( const cctag::Parameters & params )
{
    cerr << "Enter " << __FUNCTION__ << endl;
    assert( getWidth()  == _d_map.cols );
    assert( getHeight() == _d_map.rows );
    assert( getWidth()  == _d_hyst_edges.cols );
    assert( getHeight() == _d_hyst_edges.rows );

#ifndef NDEBUG
    dim3 block;
    dim3 grid;
    block.x = HYST_W;
    block.y = HYST_H;
    grid.x  = grid_divide( getWidth(),   HYST_W );
    grid.y  = grid_divide( getHeight(),  HYST_H );

    verify_map_valid
        <<<grid,block,0,_stream>>>
        ( _d_map, _d_hyst_edges, getWidth(), getHeight() );
#endif

#if defined(USE_SEPARABLE_COMPILATION)
    hipEvent_t before_hyst, after_hyst;
    float ms;

    hipEventCreate( &before_hyst );
    hipEventCreate( &after_hyst );
    hipEventRecord( before_hyst, _stream );
    cerr << "0" << endl;
    hyst_outer_loop
        <<<1,1,0,_stream>>>
        ( getWidth(), getHeight(), _d_hysteresis_block_counter, _d_hyst_edges, _d_map );
    hipEventRecord( after_hyst, _stream );
    cerr << "0.1" << endl;
    hipEventSynchronize( after_hyst );
    cerr << "0.2" << endl;
    hipEventElapsedTime( &ms, before_hyst, after_hyst );
    hipEventDestroy( before_hyst );
    hipEventDestroy( after_hyst );
    std::cerr << "Hyst took " << ms << " ms" << std::endl;
#else // USE_SEPARABLE_COMPILATION
    bool first_time = true;
    int block_counter;
    do
    {
        block_counter = grid.x * grid.y;
        POP_CUDA_MEMCPY_TO_DEVICE_ASYNC( _d_hysteresis_block_counter,
                                         &block_counter,
                                         sizeof(int), _stream );
        if( first_time ) {
            cerr << "1" << endl;
            hysteresis::edge_first
                <<<grid,block,0,_stream>>>
                ( _d_hyst_edges,
                  _d_hysteresis_block_counter,
                  _d_map );
            first_time = false;
        } else {
            cerr << "2" << endl;
            hysteresis::edge_second
                <<<grid,block,0,_stream>>>
                ( _d_hyst_edges,
                  _d_hysteresis_block_counter );
        }
        POP_CHK_CALL_IFSYNC;

        POP_CUDA_MEMCPY_TO_HOST_ASYNC( &block_counter,
                                       _d_hysteresis_block_counter,
                                       sizeof(int), _stream );
        POP_CUDA_SYNC( _stream );
        cerr << "block_counter=" << block_counter << endl;
    }
    while( block_counter > 0 );
#endif // USE_SEPARABLE_COMPILATION
    cerr << "Leave " << __FUNCTION__ << endl;
}

}; // namespace popart


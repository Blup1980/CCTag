#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "debug_macros.hpp"
#include "debug_is_on_edge.h"

#include "frame.h"
#include "assist.h"

namespace popart
{

using namespace std;

namespace thinning {

static unsigned char h_lut[256] = {
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 0, 1, 1, 1, 0, 1, 1, 1, 
        1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 
        1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 
        1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 
};

// Note that the transposed h_lut_t is not really necessary
// because flipping the 4 LSBs and 4 HSBs in the unsigned char that
// I use for lookup is fast.
static unsigned char h_lut_t[256] = {
        1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 0, 0, 
        1, 1, 1, 1, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 0, 0, 0, 0, 0, 1, 1, 1, 0, 0, 0, 0, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 0, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 
        1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 0, 1, 0, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
};

__device__ __constant__ unsigned char d_lut[256];

__device__ __constant__ unsigned char d_lut_t[256];

__device__
bool update_pixel( const int idx, const int idy, cv::cuda::PtrStepSzb src, cv::cuda::PtrStepSzb dst, bool first_run )
{
    if( src.ptr(idy)[idx] != 2 ) {
        dst.ptr(idy)[idx] = 0;
        return false;
    }

    if( idx >= 1 && idy >=1 && idx <= src.cols-2 && idy <= src.rows-2 ) {
        uint8_t log = 0;

        log |= ( src.ptr(idy-1)[idx  ] == 2 ) ? 0x01 : 0;
        log |= ( src.ptr(idy-1)[idx+1] == 2 ) ? 0x02 : 0;
        log |= ( src.ptr(idy  )[idx+1] == 2 ) ? 0x04 : 0;
        log |= ( src.ptr(idy+1)[idx+1] == 2 ) ? 0x08 : 0;
        log |= ( src.ptr(idy+1)[idx  ] == 2 ) ? 0x10 : 0;
        log |= ( src.ptr(idy+1)[idx-1] == 2 ) ? 0x20 : 0;
        log |= ( src.ptr(idy  )[idx-1] == 2 ) ? 0x40 : 0;
        log |= ( src.ptr(idy-1)[idx-1] == 2 ) ? 0x80 : 0;

        unsigned char result;
        if( first_run ) {
            result = d_lut[log] ? 2 : 0;
        } else {
            result = d_lut_t[log];
        }
        dst.ptr(idy)[idx] = result;

        return ( result != 0 );
        // return true;
    }
    return false;
}

__global__
void first_round( cv::cuda::PtrStepSzb src, cv::cuda::PtrStepSzb dst )
{
    const int block_x = blockIdx.x * 32;
    const int idx     = block_x + threadIdx.x;
    const int idy     = blockIdx.y;

    update_pixel( idx, idy, src, dst, true );
}

__global__
void second_round( cv::cuda::PtrStepSzb src,          // input
                   cv::cuda::PtrStepSzb dst,          // output
                   DevEdgeList<int2>    edgeCoords,   // output
                   uint32_t             param_edgeMax )     // input
{
    const int block_x = blockIdx.x * 32;
    const int idx     = block_x + threadIdx.x;
    const int idy     = blockIdx.y;

    bool keep = update_pixel( idx, idy, src, dst, false );

    uint32_t mask = __ballot( keep );  // bitfield of warps with results
    uint32_t ct   = __popc( mask );    // horizontal reduce
    uint32_t leader = __ffs(mask) - 1; // the highest thread id with indicator==true
    uint32_t write_index;
    if( threadIdx.x == leader ) {
        // leader gets warp's offset from global value and increases it
        write_index = atomicAdd( edgeCoords.size, int(ct) );
    }
    write_index = __shfl( write_index, leader ); // broadcast warp write index to all
    write_index += __popc( mask & ((1 << threadIdx.x) - 1) ); // find own write index

    if( keep && write_index < param_edgeMax ) {
        edgeCoords.ptr[write_index] = make_int2( idx, idy );
    }
}

__global__
void set_edgemax( DevEdgeList<int2> edgeCoords,
                  uint32_t          param_edgeMax )
{
    if( edgeCoords.Size() > param_edgeMax ) {
        edgeCoords.setSize( param_edgeMax );
    }
}

#ifdef USE_SEPARABLE_COMPILATION
__global__
void dp_caller( const size_t         width,          // input
                const size_t         height,         // input
                cv::cuda::PtrStepSzb hystEdges,      // input
                cv::cuda::PtrStepSzb edges,          // output
                DevEdgeList<int2>    edgeCoords,     // output
                cv::cuda::PtrStepSzb intermediate,   // intermediate
                uint32_t             param_edgeMax ) // input param
{
    edgeCoords.size = 0;

    dim3 block;
    dim3 grid;
    block.x = 32;
    grid.x  = ( width / 32 ) + ( width % 32 == 0 ? 0 : 1 );
    grid.y  = height;

    first_round
        <<<grid,block>>>
        ( hystEdges, intermediate );

    second_round
        <<<grid,block>>>
        ( intermediate,    // input
          edges,           // output
          edgeCoords,      // output
          param_edgeMax ); // input param
}
#endif // USE_SEPARABLE_COMPILATION

}; // namespace thinning

__host__
void Frame::initThinningTable( )
{
    POP_CUDA_MEMCPY_HOST_TO_SYMBOL_SYNC( thinning::d_lut,
                                         thinning::h_lut,
                                         256*sizeof(unsigned char) );
    POP_CUDA_MEMCPY_HOST_TO_SYMBOL_SYNC( thinning::d_lut_t,
                                         thinning::h_lut_t,
                                         256*sizeof(unsigned char) );
}

__host__
void Frame::applyThinning( const cctag::Parameters & params )
{
#ifdef USE_SEPARABLE_COMPILATION
    thinning::dp_caller
        <<<1,1,0,_stream>>>
        ( getWidth(),
          getHeight(),
          _d_hyst_edges,                         // input
          _d_edges,                              // output
          _vote._all_edgecoords.dev,             // output
          cv::cuda::PtrStepSzb(_d_intermediate), // intermediate
          params._maxEdges );                    // input param
    POP_CHK_CALL_IFSYNC;
#else // USE_SEPARABLE_COMPILATION
    dim3 block;
    dim3 grid;
    block.x = 32;
    grid.x  = ( getWidth() / 32 ) + ( getWidth() % 32 == 0 ? 0 : 1 );
    grid.y  = getHeight();

    thinning::first_round
        <<<grid,block,0,_stream>>>
        ( _d_hyst_edges, cv::cuda::PtrStepSzb(_d_intermediate) );
    POP_CHK_CALL_IFSYNC;

    POP_CUDA_SET0_ASYNC( _vote._all_edgecoords.dev.size, _stream );

    thinning::secound_round
        <<<grid,block,0,_stream>>>
        ( cv::cuda::PtrStepSzb(_d_intermediate), // input
          _d_edges,                              // output
          _vote._all_edgecoords.dev,             // output
          params._maxEdges );                    // input
    POP_CHK_CALL_IFSYNC;
#endif // USE_SEPARABLE_COMPILATION

    thinning::set_edgemax
        <<<1,1,0,_stream>>>
        ( _vote._all_edgecoords.dev,
          params._maxEdges );
    POP_CHK_CALL_IFSYNC;

#ifndef NDEBUG
    debugPointIsOnEdge( _d_edges, _vote._all_edgecoords, _stream );
#endif // NDEBUG

#ifdef EDGE_LINKING_HOST_SIDE
    /* After thinning_and_store, _all_edgecoords is no longer changed.
     * Make a non-blocking copy the number of items in the list to the host.
     */
    _vote._all_edgecoords.copySizeFromDevice( _stream );
    hipEventRecord( &_download_ready_event.edgecoords, _stream );
    POP_CHK_CALL_IFSYNC;
#endif // EDGE_LINKING_HOST_SIDE
}

__host__
void Frame::applyThinDownload( const cctag::Parameters& )
{
#ifdef EDGE_LINKING_HOST_SIDE
    /* After thinning_and_store, _all_edgecoords is no longer changed
     * we can copy it to the host for edge linking
     */

    /* CPU must wait for counter _vote._all_edgecoords.host.size */
    hipEventSynchronize( _download_ready_event.edgecoords );
    POP_CHK_CALL_IFSYNC;
    if( _vote._all_edgecoords.host.size > 0 ) {
        _vote._all_edgecoords.copyDataFromDevice( _vote._all_edgecoords.host.size,
                                                  _download_stream );
        POP_CHK_CALL_IFSYNC;
    }
#ifndef NDEBUG
    else
    {
        // initialize the hostside array to 0 for debugging
        _vote._all_edgecoords.initHost( );
    }
#endif // NDEBUG
#endif // EDGE_LINKING_HOST_SIDE
}

}; // namespace popart


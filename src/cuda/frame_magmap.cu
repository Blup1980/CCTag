#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "debug_macros.hpp"

#include "frame.h"
#include "clamp.h"
#include "assist.h"


namespace popart
{

using namespace std;

__global__
void compute_mag_l1( cv::cuda::PtrStepSz16s src_dx,
                     cv::cuda::PtrStepSz16s src_dy,
                     cv::cuda::PtrStepSz32u dst )
{
    int block_x = blockIdx.x * 32;
    int idx     = block_x + threadIdx.x;
    int idy     = blockIdx.y;

    if( outOfBounds( idx, idy, dst ) ) return;

    int16_t dx = src_dx.ptr(idy)[idx];
    int16_t dy = src_dy.ptr(idy)[idx];
    dx = d_abs( dx );
    dy = d_abs( dy );
    dst.ptr(idy)[idx] = dx + dy;
}

__global__
void compute_mag_l2( cv::cuda::PtrStepSz16s src_dx,
                     cv::cuda::PtrStepSz16s src_dy,
                     cv::cuda::PtrStepSz32u dst )
{
    int block_x = blockIdx.x * 32;
    int idx     = block_x + threadIdx.x;
    int idy     = blockIdx.y;

    if( outOfBounds( idx, idy, dst ) ) return;

    int16_t dx = src_dx.ptr(idy)[idx];
    int16_t dy = src_dy.ptr(idy)[idx];
    // --- rintf( hypot ( ) ) --
    dx *= dx;
    dy *= dy;
    dst.ptr(idy)[idx] = __fsqrt_rn( (float)( dx + dy ) );
}

__global__
void compute_map( const cv::cuda::PtrStepSz16s dx,
                  const cv::cuda::PtrStepSz16s dy,
                  const cv::cuda::PtrStepSz32u mag,
                  cv::cuda::PtrStepSzb         map,
                  const float                  low_thresh,
                  const float                  high_thresh )
{
    const int CANNY_SHIFT = 15;
    const int TG22 = (int32_t)(0.4142135623730950488016887242097*(1<<CANNY_SHIFT) + 0.5);

    const int block_x = blockIdx.x * 32;
    const int idx     = block_x + threadIdx.x;
    const int idy     = blockIdx.y;

    if( outOfBounds( idx, idy, dx ) ) return;

    int32_t  dxVal  = dx.ptr(idy)[idx];
    int32_t  dyVal  = dy.ptr(idy)[idx];
    uint32_t magVal = mag.ptr(idy)[idx];

    // -1 if only is negative, 1 else
    // const int32_t signVal = (dxVal ^ dyVal) < 0 ? -1 : 1;
    const int32_t signVal = d_sign( dxVal ^ dyVal );

    dxVal = d_abs( dxVal );
    dyVal = d_abs( dyVal );

    // 0 - the pixel can not belong to an edge
    // 1 - the pixel might belong to an edge
    // 2 - the pixel does belong to an edge
    uint8_t edge_type = 0;

    if( magVal > low_thresh )
    {
        const int32_t tg22x = dxVal * TG22;
        const int32_t tg67x = tg22x + ((dxVal + dxVal) << CANNY_SHIFT);

        dyVal <<= CANNY_SHIFT;

        int2 x = (dyVal < tg22x) ? make_int2( idx - 1, idx + 1 )
                                 : (dyVal > tg67x ) ? make_int2( idx, idx )
                                                    : make_int2( idx - signVal, idx + signVal );
        int2 y = (dyVal < tg22x) ? make_int2( idy, idy )
                                 : make_int2( idy - 1, idy + 1 );

        x.x = clamp( x.x, dx.cols );
        x.y = clamp( x.y, dx.cols );
        y.x = clamp( y.x, dx.rows );
        y.y = clamp( y.y, dx.rows );

        if( magVal > mag.ptr(y.x)[x.x] && magVal >= mag.ptr(y.y)[x.y] ) {
            edge_type = 1 + (uint8_t)(magVal > high_thresh);
        }
    }
    __syncthreads();

    assert( edge_type <= 2 );
    map.ptr(idy)[idx] = edge_type;
}

__host__
void Frame::applyMag( const cctag::Parameters & params )
{
    // cerr << "Enter " << __FUNCTION__ << endl;

    dim3 block;
    dim3 grid;
    block.x = 32;
    grid.x  = ( getWidth() / 32 ) + ( getWidth() % 32 == 0 ? 0 : 1 );
    grid.y  = getHeight();

    dim3 big_block;
    dim3 big_grid;
    big_block.x = 32;
    big_block.y = 32;
    big_grid.x  = ( getWidth()  / 32 ) + ( getWidth()  % 32 == 0 ? 0 : 1 );
    big_grid.y  = ( getHeight() / 32 ) + ( getHeight() % 32 == 0 ? 0 : 1 );

    // necessary to merge into 1 stream
    compute_mag_l2
        <<<grid,block,0,_stream>>>
        ( _d_dx, _d_dy, _d_mag );
    POP_CHK_CALL_IFSYNC;

    /* block download until MAG is ready */
    hipEventRecord( &_download_ready_event.mag, _stream );
    hipStreamWaitEvent( _download_stream, _download_ready_event.mag );

    POP_CUDA_MEMCPY_2D_ASYNC( _h_mag.data, _h_mag.step,
                              _d_mag.data, _d_mag.step,
                              _d_mag.cols * sizeof(uint32_t),
                              _d_mag.rows,
                              hipMemcpyDeviceToHost, _download_stream );

    compute_map
        <<<grid,block,0,_stream>>>
        ( _d_dx, _d_dy, _d_mag, _d_map, 256.0f * params._cannyThrLow, 256.0f * params._cannyThrHigh );
    POP_CHK_CALL_IFSYNC;

    /* block download until MAG is ready */
    hipEventRecord( &_download_ready_event.map, _stream );
    hipStreamWaitEvent( _download_stream, _download_ready_event.map );

#ifdef DEBUG_WRITE_MAP_AS_PGM
    POP_CUDA_MEMCPY_2D_ASYNC( _h_debug_map, getWidth() * sizeof(uint8_t),
                              _d_map.data, _d_map.step,
                              _d_map.cols * sizeof(uint8_t),
                              _d_map.rows,
                              hipMemcpyDeviceToHost, _download_stream );
#endif // DEBUG_WRITE_MAP_AS_PGM

    // cerr << "Leave " << __FUNCTION__ << endl;
}

}; // namespace popart


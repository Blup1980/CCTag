#include "hip/hip_runtime.h"
#include "tag.h"
#include "frame.h"
#include "frameparam.h"
#include "debug_macros.hpp"
#include "keep_time.hpp"
#include "pinned_counters.h"
#include <sstream>
#include <iostream>
#include <fstream>

#include "debug_image.h"
#include "cctag/talk.hpp"
#include "cuda/geom_ellipse.h"
#include "cctag/algebra/matrix/Matrix.hpp"

#include "cctag/logtime.hpp"
#include "cuda/onoff.h"

using namespace std;

namespace popart
{

__host__
void TagPipe::initialize( const uint32_t pix_w,
                          const uint32_t pix_h,
                          const cctag::Parameters& params,
                          cctag::logtime::Mgmt* durations )
{
    pinned_counters.init( );

    static bool tables_initialized = false;
    if( not tables_initialized ) {
        tables_initialized = true;
        Frame::initGaussTable( );
        Frame::initThinningTable( );
    }

    FrameParam::init( params );

    int num_layers = params._numberOfMultiresLayers;
    _frame.reserve( num_layers );

    uint32_t w = pix_w;
    uint32_t h = pix_h;
    popart::Frame* f;
#ifdef USE_ONE_DOWNLOAD_STREAM
    hipStream_t download_stream = 0;
    for( int i=0; i<num_layers; i++ ) {
        _frame.push_back( f = new popart::Frame( w, h, i, download_stream ) ); // sync
        if( i==0 ) { download_stream = f->_download_stream; assert( download_stream != 0 ); }
        w = ( w >> 1 ) + ( w & 1 );
        h = ( h >> 1 ) + ( h & 1 );
    }
#else
    for( int i=0; i<num_layers; i++ ) {
        _frame.push_back( f = new popart::Frame( w, h, i, 0 ) ); // sync
        w = ( w >> 1 ) + ( w & 1 );
        h = ( h >> 1 ) + ( h & 1 );
    }
#endif

    _frame[0]->createTexture( popart::FrameTexture::normalized_uchar_to_float); // sync
    _frame[0]->allocUploadEvent( ); // sync

    for( int i=0; i<num_layers; i++ ) {
        _frame[i]->allocRequiredMem( params ); // sync
    }
}

__host__
uint32_t TagPipe::getWidth(  size_t layer ) const
{
    return _frame[layer]->getWidth();
}

__host__
uint32_t TagPipe::getHeight( size_t layer ) const
{
    return _frame[layer]->getHeight();
}


__host__
void TagPipe::load( unsigned char* pix )
{
#ifndef CCTAG_NO_COUT
    KeepTime t( _frame[0]->_stream );
    t.start();
#endif // CCTAG_NO_COUT

    _frame[0]->upload( pix ); // async

#ifndef CCTAG_NO_COUT
    t.stop();
    t.report( "Time for frame upload " );
#endif // CCTAG_NO_COUT
}

__host__
void TagPipe::tagframe( const cctag::Parameters& params )
{
    int num_layers = _frame.size();

#ifdef SHOW_DETAILED_TIMING
    KeepTime* time_gauss[num_layers];
    KeepTime* time_mag  [num_layers];
    KeepTime* time_hyst [num_layers];
    KeepTime* time_thin [num_layers];
    KeepTime* time_desc [num_layers];
    KeepTime* time_vote [num_layers];
    for( int i=0; i<num_layers; i++ ) {
        time_gauss[i] = new KeepTime( _frame[i]->_stream );
        time_mag  [i] = new KeepTime( _frame[i]->_stream );
        time_hyst [i] = new KeepTime( _frame[i]->_stream );
        time_thin [i] = new KeepTime( _frame[i]->_stream );
        time_desc [i] = new KeepTime( _frame[i]->_stream );
        time_vote [i] = new KeepTime( _frame[i]->_stream );
    }
#endif

#ifndef CCTAG_NO_COUT
    KeepTime t( _frame[0]->_stream );
    t.start();
#endif // CCTAG_NO_COUT

    for( int i=0; i<num_layers; i++ ) {
        _frame[i]->initRequiredMem( ); // async
    }

    hipEvent_t ev = _frame[0]->addUploadEvent( ); // async

    for( int i=1; i<num_layers; i++ ) {
        _frame[i]->streamSync( ev ); // aysnc
        _frame[i]->fillFromTexture( *(_frame[0]) ); // aysnc
        // _frame[i]->fillFromFrame( *(_frame[0]) );
    }

#ifdef SHOW_DETAILED_TIMING
#error SHOW_DETAILED_TIMING needs to be rewritten
    for( int i=0; i<num_layers; i++ ) {
        bool success;
        time_gauss[i]->start();
        time_gauss[i]->stop();
        time_mag[i]->start();
        time_mag[i]->stop();
        time_hyst[i]->start();
        time_hyst[i]->stop();
        time_thin[i]->start();
        time_thin[i]->stop();
        time_desc[i]->start();
        time_desc[i]->stop();
        time_vote[i]->start();
        time_vote[i]->stop();
    }
#else // not SHOW_DETAILED_TIMING
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyPlaneDownload(); // async
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyGauss( params ); // async
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyGaussDownload(); // async

    for( int i=0; i<num_layers; i++ ) _frame[i]->applyMag();  // async

    for( int i=0; i<num_layers; i++ ) _frame[i]->applyHyst();  // async
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyMagDownload();

    for( int i=0; i<num_layers; i++ ) _frame[i]->applyThinning();  // async
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyThinDownload(); // sync

#ifdef USE_SEPARABLE_COMPILATION
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyDesc();  // async
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyVoteConstructLine();  // async
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyVoteSortUniqDP( params );  // async
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyVoteEval();  // async
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyVoteIf();  // async
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyVote( params );  // async
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyVoteDownload();   // sync!
#else // USE_SEPARABLE_COMPILATION
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyDesc();  // async
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyVote( params );  // async
    for( int i=0; i<num_layers; i++ ) _frame[i]->applyVoteDownload();   // sync!
#endif // USE_SEPARABLE_COMPILATION
#endif // not SHOW_DETAILED_TIMING

    for( int i=1; i<num_layers; i++ ) {
        hipEventRecord( _frame[i]->_download_stream_done, _frame[i]->_download_stream );
        hipStreamWaitEvent( _frame[i]->_stream, _frame[i]->_download_stream_done, 0 );
        hipEventRecord( _frame[i]->_stream_done, _frame[i]->_stream );
    }
    hipEventRecord( _frame[0]->_download_stream_done, _frame[0]->_download_stream );
    hipStreamWaitEvent( _frame[0]->_stream, _frame[0]->_download_stream_done, 0 );
    for( int i=1; i<num_layers; i++ ) {
        hipStreamWaitEvent( _frame[0]->_stream, _frame[i]->_stream_done, 0 );
    }
    hipEventRecord( _frame[0]->_stream_done, _frame[0]->_stream );

#ifndef CCTAG_NO_COUT
    t.stop();
    t.report( "Time for all frames " );
#endif // CCTAG_NO_COUT

#ifdef SHOW_DETAILED_TIMING
    for( int i=0; i<num_layers; i++ ) {
        time_gauss[i]->report( "time for Gauss " );
        time_mag  [i]->report( "time for Mag   " );
        time_hyst [i]->report( "time for Hyst  " );
        time_thin [i]->report( "time for Thin  " );
        time_desc [i]->report( "time for Desc  " );
        time_vote [i]->report( "time for Vote  " );
        delete time_gauss[i];
        delete time_mag  [i];
        delete time_hyst [i];
        delete time_thin [i];
        delete time_desc [i];
        delete time_vote [i];
    }
#endif // not NDEBUG
}

__host__
void TagPipe::convertToHost( size_t                          layer,
                             std::vector<cctag::EdgePoint>&  vPoints,
                             cctag::EdgePointsImage&         edgeImage,
                             std::vector<cctag::EdgePoint*>& seeds,
                             cctag::WinnerMap&               winners )
{

    assert( layer < _frame.size() );

    _frame[layer]->applyExport( vPoints, edgeImage, seeds, winners );

}

__host__
cv::Mat* TagPipe::getPlane( size_t layer ) const
{
    assert( layer < getNumOctaves() );
    return _frame[layer]->getPlane();
}

__host__
cv::Mat* TagPipe::getDx( size_t layer ) const
{
    assert( layer < getNumOctaves() );
    return _frame[layer]->getDx();
}

__host__
cv::Mat* TagPipe::getDy( size_t layer ) const
{
    assert( layer < getNumOctaves() );
    return _frame[layer]->getDy();
}

__host__
cv::Mat* TagPipe::getMag( size_t layer ) const
{
    assert( layer < getNumOctaves() );
    return _frame[layer]->getMag();
}

__host__
cv::Mat* TagPipe::getEdges( size_t layer ) const
{
    assert( layer < getNumOctaves() );
    return _frame[layer]->getEdges();
}

__host__
void TagPipe::debug( unsigned char* pix, const cctag::Parameters& params )
{
    DO_TALK( cerr << "Enter " << __FUNCTION__ << endl; )

    if( true ) {
        if( params._debugDir == "" ) {
            DO_TALK( cerr << __FUNCTION__ << ":" << __LINE__
                << ": debugDir not set, not writing debug output" << endl; )
            return;
        } else {
            DO_TALK( cerr << __FUNCTION__ << ":" << __LINE__ << ": debugDir is ["
                 << params._debugDir << "] using that directory" << endl; )
        }

        // This is a debug block

        int num_layers = _frame.size();

        for( int i=0; i<num_layers; i++ ) {
            _frame[i]->hostDebugDownload( params );
        }
        POP_SYNC_CHK;

        _frame[0]->hostDebugCompare( pix );

        for( int i=0; i<num_layers; i++ ) {
            std::ostringstream ostr;
            ostr << "gpu-" << i;
            _frame[i]->writeHostDebugPlane( ostr.str(), params );
        }
        POP_SYNC_CHK;
    }

    DO_TALK( cerr << "terminating in tagframe" << endl; )
    DO_TALK( cerr << "Leave " << __FUNCTION__ << endl; )
    // exit( 0 );
}

void TagPipe::debug_cpu_origin( int                      layer,
                                const cv::Mat&           img,
                                const cctag::Parameters& params )
{
    if( params._debugDir == "" ) {
        DO_TALK( cerr << __FUNCTION__ << ":" << __LINE__
            << ": debugDir not set, not writing debug output" << endl; )
        return;
    } else {
        DO_TALK( cerr << __FUNCTION__ << ":" << __LINE__ << ": debugDir is ["
            << params._debugDir << "] using that directory" << endl; )
    }

    ostringstream ascname;
    ascname << params._debugDir << "cpu-" << layer << "-img-ascii.txt";
    ofstream asc( ascname.str().c_str() );

    int cols = img.size().width;
    int rows = img.size().height;
    for( int y=0; y<rows; y++ ) {
        for( int x=0; x<cols; x++ ) {
            uint8_t pix = img.at<uint8_t>(y,x);
            asc << setw(3) << (int)pix << " ";
        }
        asc << endl;
    }
}

void TagPipe::debug_cpu_edge_out( int                      layer,
                                  const cv::Mat&           edges,
                                  const cctag::Parameters& params )
{
    if( params._debugDir == "" ) {
        DO_TALK( cerr << __FUNCTION__ << ":" << __LINE__
            << ": debugDir not set, not writing debug output" << endl; )
        return;
    } else {
        DO_TALK( cerr << __FUNCTION__ << ":" << __LINE__ << ": debugDir is ["
            << params._debugDir << "] using that directory" << endl; )
    }

    ostringstream filename;
    filename << params._debugDir
             << "cpu-" << layer << "-edges.ppm";

    cv::cuda::PtrStepSzb plane;
    plane.step = edges.size().width;
    plane.cols = edges.size().width;
    plane.rows = edges.size().height;
    if( plane.cols == 0 || plane.rows == 0 ) return;
    plane.data = new uint8_t[ plane.cols * plane.rows ];

    for( int y=0; y<plane.rows; y++ )
        for( int x=0; x<plane.cols; x++ ) {
            plane.ptr(y)[x] = edges.at<uint8_t>(y,x);
        }

    DebugImage::writePGM( filename.str(), plane );

    delete [] plane.data;
}

static void local_debug_cpu_dxdy_out( const char*                  dxdy,
                                      size_t                       level,
                                      const cv::Mat&               cpu,
                                      const cv::cuda::PtrStepSz16s gpu,
                                      const cctag::Parameters&     params )
{
    if( params._debugDir == "" ) {
        DO_TALK( cerr << __FUNCTION__ << ":" << __LINE__
            << ": debugDir not set, not writing debug output" << endl; )
        return;
    } else {
        DO_TALK( cerr << __FUNCTION__ << ":" << __LINE__ << ": debugDir is ["
            << params._debugDir << "] using that directory" << endl; )
    }

    if( cpu.size().width  != gpu.cols ) {
        cerr << __FILE__ << ":" << __LINE__
             << " Error: array width CPU " << cpu.size().width << " vs GPU " << gpu.cols << endl;
    }
    if( cpu.size().height != gpu.rows ) {
        cerr << __FILE__ << ":" << __LINE__
             << " Error: array height CPU " << cpu.size().height << " vs GPU " << gpu.rows << endl;
    }

    int cols = min( cpu.size().width, gpu.cols );
    int rows = min( cpu.size().height, gpu.rows );

    if( cols == 0 || rows == 0 ) return;

    cv::cuda::PtrStepSz16s plane;
    plane.step = cols * sizeof(int16_t);
    plane.cols = cols;
    plane.rows = rows;
    plane.data = new int16_t[ cols * rows ];

    for( int y=0; y<rows; y++ ) {
        for( int x=0; x<cols; x++ ) {
            int16_t cpu_val = cpu.at<int16_t>(y,x);
            int16_t gpu_val = gpu.ptr(y)[x];
            plane.ptr(y)[x] = (int16_t)gpu_val - (int16_t)cpu_val;
#if 0
            if( y < 4 || x < 4 || y >= rows-4 || x >= cols-4 ) {
                diffplane.ptr(y)[x] = 0;
            }
#endif
        }
    }
    ostringstream asc_f_diff;
    ostringstream img_f_diff;
    asc_f_diff << params._debugDir << "diffcpugpu-" << level << "-" << dxdy << "-ascii.txt";
    img_f_diff << params._debugDir << "diffcpugpu-" << level << "-" << dxdy << ".pgm";
    DebugImage::writePGMscaled( img_f_diff.str(), plane );
    DebugImage::writeASCII(     asc_f_diff.str(), plane );

    for( int y=0; y<rows; y++ ) {
        for( int x=0; x<cols; x++ ) {
            int16_t cpu_val   = cpu.at<int16_t>(y,x);
            plane.ptr(y)[x] = min<int16_t>( max<int16_t>( (int16_t)cpu_val, -255 ), 255 );
        }
    }

    ostringstream asc_f_cpu;
    ostringstream img_f_cpu;
    asc_f_cpu  << params._debugDir << "cpu-" << level << "-" << dxdy << "-ascii.txt";
    img_f_cpu  << params._debugDir << "cpu-" << level << "-" << dxdy << ".pgm";
    DebugImage::writePGMscaled( img_f_cpu.str(), plane );
    DebugImage::writeASCII(     asc_f_cpu.str(), plane );

    delete [] plane.data;
}

void TagPipe::debug_cpu_dxdy_out( TagPipe*                     pipe,
                                  int                          layer,
                                  const cv::Mat&               cpu_dx,
                                  const cv::Mat&               cpu_dy,
                                  const cctag::Parameters&     params )
{
    const cv::cuda::PtrStepSz16s gpu_dx = pipe->_frame[layer]->_h_dx;
    const cv::cuda::PtrStepSz16s gpu_dy = pipe->_frame[layer]->_h_dy;
    size_t                       level  = pipe->_frame[layer]->getLayer();

    local_debug_cpu_dxdy_out( "dx", level, cpu_dx, gpu_dx, params );
    local_debug_cpu_dxdy_out( "dy", level, cpu_dy, gpu_dy, params );
}

void TagPipe::debug_cmp_edge_table( int                           layer,
                                    const cctag::EdgePointsImage& cpu,
                                    const cctag::EdgePointsImage& gpu,
                                    const cctag::Parameters&      params )
{
    if( params._debugDir == "" ) {
        DO_TALK( cerr << __FUNCTION__ << ":" << __LINE__
            << ": debugDir not set, not writing debug output" << endl; )
        return;
    } else {
        DO_TALK( cerr << __FUNCTION__ << ":" << __LINE__ << ": debugDir is ["
            << params._debugDir << "] using that directory" << endl; )
    }

    ostringstream filename;
    filename << params._debugDir
             << "diffcpugpu-" << layer << "-edge.ppm";

    cv::cuda::PtrStepSzb plane;
    plane.data = new uint8_t[ cpu.shape()[0] * cpu.shape()[1] ];
    plane.step = cpu.shape()[0];
    plane.cols = cpu.shape()[0];
    plane.rows = cpu.shape()[1];

    if( gpu.size() != 0 && gpu.size() != 0 ) {
        for( int y=0; y<cpu.shape()[1]; y++ ) {
            for( int x=0; x<cpu.shape()[0]; x++ ) {
                if( cpu[x][y] != 0 && gpu[x][y] == 0 )
                    plane.ptr(y)[x] = DebugImage::BLUE;
                else if( cpu[x][y] == 0 && gpu[x][y] != 0 )
                    plane.ptr(y)[x] = DebugImage::GREEN;
                else if( cpu[x][y] != 0 && gpu[x][y] != 0 )
                    plane.ptr(y)[x] = DebugImage::GREY1;
                else
                    plane.ptr(y)[x] = DebugImage::BLACK;
            }
        }

        DebugImage::writePPM( filename.str(), plane );
    }

    delete [] plane.data;
}

double TagPipe::idCostFunction( int                                        level,
                                const cctag::numerical::geometry::Ellipse& ellipse,
                                const cctag::Point2dN<double>&             center,
                                std::vector<cctag::ImageCut>&              vCuts,
                                const size_t                               vCutMaxVecLen,
                                const float                                neighbourSize,
                                const size_t                               gridNSample,
                                cctag::Point2dN<double>&                   bestPointOut,
                                cctag::numerical::BoundedMatrix3x3d&       bestHomographyOut )
{
    /* The first part of cctag::identification::getNearbyPoints() applies
     * to all possible centers for the candidate tag. It is best to
     * compute it on the host side.
     * Computing the nearby centers is gradNSample X gridNSample size
     * operation and best moved to the device side.
     */
/*
    cctag::numerical::BoundedMatrix3x3d mT = cctag::numerical::optimization::conditionerFromEllipse( ellipse );
    cctag::numerical::BoundedMatrix3x3d mInvT;
    cctag::numerical::invert_3x3(mT,mInvT);

    cctag::numerical::geometry::Ellipse transformedEllipse(ellipse);
    cctag::viewGeometry::projectiveTransform( mInvT, transformedEllipse );
    neighbourSize *= std::max(transformedEllipse.a(),transformedEllipse.b());

    cctag::Point2dN<double> condCenter = center;
    cctag::numerical::optimization::condition(condCenter, mT);
*/

    popart::geometry::ellipse e( ellipse.matrix()(0,0),
                                 ellipse.matrix()(0,1),
                                 ellipse.matrix()(0,2),
                                 ellipse.matrix()(1,0),
                                 ellipse.matrix()(1,1),
                                 ellipse.matrix()(1,2),
                                 ellipse.matrix()(2,0),
                                 ellipse.matrix()(2,1),
                                 ellipse.matrix()(2,2),
                                 ellipse.center().x(),
                                 ellipse.center().y(),
                                 ellipse.a(),
                                 ellipse.b(),
                                 ellipse.angle() );
    float2 f = make_float2( center.x(), center.y() );

    float2                      bestPoint;
    popart::geometry::matrix3x3 bestHomography;
    double avg = _frame[level]->idCostFunction( e,
                                                f,
                                                vCuts,
                                                vCutMaxVecLen,
                                                neighbourSize,
                                                gridNSample,
                                                bestPoint,
                                                bestHomography );
    if( avg < FLT_MAX ) {
        bestPointOut.x() = bestPoint.x;
        bestPointOut.y() = bestPoint.y;

    #pragma unroll
    for( int i=0; i<3; i++ ) {
        #pragma unroll
        for( int j=0; j<3; j++ ) {
                bestHomographyOut(i,j) = bestHomography(i,j);
            }
        }
    }
    return avg;
}

}; // namespace popart


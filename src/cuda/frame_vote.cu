#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <limits>
#include <hip/hip_runtime.h>
#include <thrust/system/cuda/detail/hipcub/hipcub.hpp>
#include <stdio.h>
#include "debug_macros.hpp"

#include "frame.h"
#include "assist.h"

using namespace std;

#define COMPRESS_VOTING_AND_SELECT

namespace popart
{

namespace vote
{

#ifndef NDEBUG
__device__
void debug_inner_test_consistency( int                            p_idx,
                                   const TriplePoint*             p,
                                   cv::cuda::PtrStepSz32s         edgepoint_index_table,
                                   const DevEdgeList<TriplePoint> chained_edgecoords )
{
    if( p == 0 ) {
        printf("Impossible bug, initialized from memory address\n");
        assert( 0 );
    }

    if( outOfBounds( p->coord, edgepoint_index_table ) ) {
        printf("Index (%d,%d) does not fit into coord lookup tables\n", p->coord.x, p->coord.y );
        assert( 0 );
    }

    int idx = edgepoint_index_table.ptr(p->coord.y)[p->coord.x];
    if( idx < 0 || idx >= chained_edgecoords.Size() ) {
        printf("Looked up index (coord) is out of bounds\n");
        assert( 0 );
    }

    if( idx != p_idx ) {
        printf("Looked up index %d is not identical to input index %d\n", idx, p_idx);
        assert( 0 );
    }

    if( outOfBounds( p->descending.befor, edgepoint_index_table ) ) {
        printf("Before coordinations (%d,%d) do not fit into lookup tables\n", p->descending.befor.x, p->descending.befor.y );
        assert( 0 );
    }

    if( outOfBounds( p->descending.after, edgepoint_index_table ) ) {
        printf("After coordinations (%d,%d) do not fit into lookup tables\n", p->descending.after.x, p->descending.after.y );
        assert( 0 );
    }
}
#endif // NDEBUG

__device__
inline
TriplePoint* find_neigh( const int2&              neigh,
                         cv::cuda::PtrStepSz32s   edgepoint_index_table,
                         DevEdgeList<TriplePoint> chained_edgecoords )
{
    if( neigh.x != 0 || neigh.y != 0 ) {
        int idx = edgepoint_index_table.ptr(neigh.y)[neigh.x];
        if( idx != 0 ) {
            assert( idx >= 0 && idx < chained_edgecoords.Size() );
            TriplePoint* neighbour = &chained_edgecoords.ptr[idx];
#ifndef NDEBUG
            debug_inner_test_consistency( idx, neighbour, edgepoint_index_table, chained_edgecoords );

            if( neigh.x != neighbour->coord.x || neigh.y != neighbour->coord.y ) {
                printf("Intended coordinate is (%d,%d) at index %d, looked up coord is (%d,%d)\n",
                       neigh.x, neigh.y,
                       idx,
                       neighbour->coord.x, neighbour->coord.y );
            }
#endif // NDEBUG
            return neighbour;
        }
    }
    return 0;
}

__device__
inline
TriplePoint* find_befor( const TriplePoint*       p,
                         cv::cuda::PtrStepSz32s   edgepoint_index_table,
                         DevEdgeList<TriplePoint> chained_edgecoords )
{
    return find_neigh( p->descending.befor, edgepoint_index_table, chained_edgecoords );
}


__device__
inline
TriplePoint* find_after( const TriplePoint*             p,
                               cv::cuda::PtrStepSz32s   edgepoint_index_table,
                               DevEdgeList<TriplePoint> chained_edgecoords )
{
    return find_neigh( p->descending.after, edgepoint_index_table, chained_edgecoords );
}

__device__
float inner_prod( const TriplePoint* l,
                  const TriplePoint* r )
{
    assert( l );
    assert( r );
    const int16_t l_dx = l->d.x;
    const int16_t l_dy = l->d.y;
    const int16_t r_dx = r->d.x;
    const int16_t r_dy = r->d.y;
    assert( l_dx != 0 || l_dy != 0 );
    assert( r_dx != 0 || r_dy != 0 );
    const float ret  = (float)l_dx * (float)r_dx
                     + (float)l_dy * (float)r_dy;
    return ret;

    // Point2dN l ( l_dx, l_dy );
    // Point2dN r ( r_dx, r_dy );
    // float return = -inner_prod(subrange(l, 0, 2), subrange(r, 0, 2));
}

__device__
inline float distance( const TriplePoint* l, const TriplePoint* r )
{
    return hypotf( l->coord.x - r->coord.x, l->coord.y - r->coord.y );
}

__device__
void updateXY(const float & dx, const float & dy, int & x, int & y,  float & e, int & stpX, int & stpY)
{
    float d = dy / dx;
    float a = d_abs( d );
    // stpX = ( dx < 0 ) ? -1 : ( dx == 0 ) ? 0 : 1;
    // stpY = ( dy < 0 ) ? -1 : ( dy == 0 ) ? 0 : 1;
    // stpX = ( dx < 0 ) ? -1 : 1;
    // stpY = ( dy < 0 ) ? -1 : 1;
    stpX = d_sign( dx );
    stpY = d_sign( dy );
    e   += a;
    x   += stpX;
    if( e >= 0.5 ) {
        y += stpY;
        e -= 1.0f;
    }
}

__device__
bool gradient_descent_inner( int4&                  out_edge_info,
                             short2&                out_edge_d,
                             DevEdgeList<int2>      all_edgecoords,
                             cv::cuda::PtrStepSzb   edge_image,
                             uint32_t               nmax,
                             cv::cuda::PtrStepSz16s d_dx,
                             cv::cuda::PtrStepSz16s d_dy,
                             int32_t                thrGradient )
{
    const int offset = blockIdx.x * 32 + threadIdx.x;
    int direction    = threadIdx.y == 0 ? -1 : 1;

    if( offset >= all_edgecoords.Size() ) return false;

    const int idx = all_edgecoords.ptr[offset].x;
    const int idy = all_edgecoords.ptr[offset].y;
#if 0
    /* This was necessary to allow the "after" threads (threadIdx.y==1)
     * to return sensible results even if "before" was 0.
     * Now useless, but kept just in case.  */
    out_edge_info.x = idx;
    out_edge_info.y = idy;
#endif

    if( outOfBounds( idx, idy, edge_image ) ) return false; // should never happen

    if( edge_image.ptr(idy)[idx] == 0 ) {
        assert( edge_image.ptr(idy)[idx] != 0 );
        return false; // should never happen
    }

    float  e     = 0.0f;
    out_edge_d.x = d_dx.ptr(idy)[idx];
    out_edge_d.y = d_dy.ptr(idy)[idx];
    float  dx    = direction * out_edge_d.x;
    float  dy    = direction * out_edge_d.y;

    assert( dx!=0 || dy!=0 );

    const float  adx   = d_abs( dx );
    const float  ady   = d_abs( dy );
    size_t n     = 0;
    int    stpX  = 0;
    int    stpY  = 0;
    int    x     = idx;
    int    y     = idy;
    
    if( ady > adx ) {
        updateXY(dy,dx,y,x,e,stpY,stpX);
    } else {
        updateXY(dx,dy,x,y,e,stpX,stpY);
    }
    n += 1;
    if ( dx*dx+dy*dy > thrGradient ) {
        const float dxRef = dx;
        const float dyRef = dy;
        const float dx2 = out_edge_d.x; // d_dx.ptr(idy)[idx];
        const float dy2 = out_edge_d.y; // d_dy.ptr(idy)[idx];
        const float compdir = dx2*dxRef+dy2*dyRef;
        // dir = ( compdir < 0 ) ? -1 : 1;
        direction = d_sign( compdir );
        dx = direction * dx2;
        dy = direction * dy2;
    }
    if( ady > adx ) {
        updateXY(dy,dx,y,x,e,stpY,stpX);
    } else {
        updateXY(dx,dy,x,y,e,stpX,stpY);
    }
    n += 1;

    if( outOfBounds( x, y, edge_image ) ) return false;

    uint8_t ret = edge_image.ptr(y)[x];
    if( ret ) {
        out_edge_info = make_int4( idx, idy, x, y );
        assert( idx != x || idy != y );
        return true;
    }
    
    while( n <= nmax ) {
        if( ady > adx ) {
            updateXY(dy,dx,y,x,e,stpY,stpX);
        } else {
            updateXY(dx,dy,x,y,e,stpX,stpY);
        }
        n += 1;

        if( outOfBounds( x, y, edge_image ) ) return false;

        ret = edge_image.ptr(y)[x];
        if( ret ) {
            out_edge_info = make_int4( idx, idy, x, y );
            assert( idx != x || idy != y );
            return true;
        }

        if( ady > adx ) {
            if( outOfBounds( x, y - stpY, edge_image ) ) return false;

            ret = edge_image.ptr(y-stpY)[x];
            if( ret ) {
                out_edge_info = make_int4( idx, idy, x, y-stpY );
                assert( idx != x || idy != y-stpY );
                return true;
            }
        } else {
            if( outOfBounds( x - stpX, y, edge_image ) ) return false;

            ret = edge_image.ptr(y)[x-stpX];
            if( ret ) {
                out_edge_info = make_int4( idx, idy, x-stpX, y );
                assert( idx != x-stpX || idy != y );
                return true;
            }
        }
    }
    return false;
}

__global__
void gradient_descent( DevEdgeList<int2>        all_edgecoords,
                       DevEdgeList<TriplePoint> chained_edgecoords,    // output
                       cv::cuda::PtrStepSz32s   edgepoint_index_table, // output
                       uint32_t                 max_num_edges,
                       cv::cuda::PtrStepSzb     edge_image,
                       uint32_t                 nmax,
                       cv::cuda::PtrStepSz16s   d_dx,
                       cv::cuda::PtrStepSz16s   d_dy,
                       int32_t                  thrGradient )
{
    assert( blockDim.x * gridDim.x < all_edgecoords.Size() + 32 );
    assert( chained_edgecoords.Size() <= 2*all_edgecoords.Size() );

    int4   out_edge_info;
    short2 out_edge_d;
    bool   keep;
    // before -1  if threadIdx.y == 0
    // after   1  if threadIdx.y == 1

    keep = gradient_descent_inner( out_edge_info,
                                   out_edge_d,
                                   all_edgecoords,
                                   edge_image,
                                   nmax,
                                   d_dx,
                                   d_dy,
                                   thrGradient );

    __syncthreads();
    __shared__ int2 merge_directions[2][32];
    merge_directions[threadIdx.y][threadIdx.x].x = keep ? out_edge_info.z : 0;
    merge_directions[threadIdx.y][threadIdx.x].y = keep ? out_edge_info.w : 0;

    /* The vote.cpp procedure computes points for before and after, and stores all
     * info in one point. In the voting procedure, after is never processed when
     * before is false.
     * Consequently, we ignore after completely when before is already false.
     * Lots of idling cores; but the _inner has a bad loop, and we may run into it,
     * which would be worse.
     */
    if( threadIdx.y == 1 ) return;

    __syncthreads(); // be on the safe side: __ballot syncs only one warp, we have 2

    TriplePoint out_edge;
    out_edge.coord.x = keep ? out_edge_info.x : 0;
    out_edge.coord.y = keep ? out_edge_info.y : 0;
    out_edge.d.x     = keep ? out_edge_d.x : 0;
    out_edge.d.y     = keep ? out_edge_d.y : 0;
    out_edge.descending.befor.x = keep ? merge_directions[0][threadIdx.x].x : 0;
    out_edge.descending.befor.y = keep ? merge_directions[0][threadIdx.x].y : 0;
    out_edge.descending.after.x = keep ? merge_directions[1][threadIdx.x].x : 0;
    out_edge.descending.after.y = keep ? merge_directions[1][threadIdx.x].y : 0;
    out_edge.my_vote            = 0;
    out_edge.chosen_flow_length = 0.0f;
    out_edge._winnerSize        = 0;
    out_edge._flowLength        = 0.0f;

    uint32_t mask = __ballot( keep );  // bitfield of warps with results
    if( mask == 0 ) return;

    uint32_t ct   = __popc( mask );    // horizontal reduce
    assert( ct <= 32 );

#if 0
    uint32_t leader = __ffs(mask) - 1; // the highest thread id with indicator==true
#else
    uint32_t leader = 0;
#endif
    uint32_t write_index;
    if( threadIdx.x == leader ) {
        // leader gets warp's offset from global value and increases it
        // not that it is initialized with 1 to ensure that 0 represents a NULL pointer
        write_index = atomicAdd( chained_edgecoords.size, (int)ct );

        if( chained_edgecoords.Size() > 2*all_edgecoords.Size() ) {
            printf( "max offset: (%d x %d)=%d\n"
                    "my  offset: (%d*32+%d)=%d\n"
                    "edges in:    %d\n"
                    "edges found: %d (total %d)\n",
                    gridDim.x, blockDim.x, blockDim.x * gridDim.x,
                    blockIdx.x, threadIdx.x, threadIdx.x + blockIdx.x*32,
                    all_edgecoords.Size(),
                    ct, chained_edgecoords.Size() );
            assert( chained_edgecoords.Size() <= 2*all_edgecoords.Size() );
        }
    }
    // assert( *chained_edgecoord_list_sz >= 2*all_edgecoord_list_sz );

    write_index = __shfl( write_index, leader ); // broadcast warp write index to all
    write_index += __popc( mask & ((1 << threadIdx.x) - 1) ); // find own write index

    if( keep && write_index < max_num_edges ) {
        assert( out_edge.coord.x != out_edge.descending.befor.x ||
                out_edge.coord.y != out_edge.descending.befor.y );
        assert( out_edge.coord.x != out_edge.descending.after.x ||
                out_edge.coord.y != out_edge.descending.after.y );
        assert( out_edge.descending.befor.x != out_edge.descending.after.x ||
                out_edge.descending.befor.y != out_edge.descending.after.y );

        /* At this point we know that we will keep the point.
         * Obviously, pointer chains in CUDA are tricky, but we can use index
         * chains based on the element's offset index in chained_edgecoord_list.
         */
        edgepoint_index_table.ptr(out_edge.coord.y)[out_edge.coord.x] = write_index;

        chained_edgecoords.ptr[write_index] = out_edge;
    }
}

/* Brief: Voting procedure. For every edge point, construct the 1st order approximation 
 * of the field line passing through it, which consists in a polygonal line whose
 * extremities are two edge points.
 * Input:
 * points: set of edge points to be processed, i.e. considered as the 1st extremity
 * of the constructed field line passing through it.
 * seeds: edge points having received enough votes to be considered as a seed, i.e.
 * as an edge point belonging on an inner elliptical arc of a cctag.
 * edgesMap: map of all the edge points
 * winners: map associating all seeds to their voters
 * cannyGradX: X derivative of the gray image
 * cannyGradY: Y derivative of the gray image
 */
__device__
const TriplePoint* construct_line_inner(
    DevEdgeList<TriplePoint>       chained_edgecoords,
    const cv::cuda::PtrStepSz32s   edgepoint_index_table,
    const size_t                   numCrowns,
    const float                    ratioVoting )
{
    int offset = threadIdx.x + blockIdx.x * 32;
    if( offset >= chained_edgecoords.Size() ) {
        return 0;
    }
    if( offset == 0 ) {
        /* special case: offset 0 is intentionally empty */
        return 0;
    }

    TriplePoint* const p = &chained_edgecoords.ptr[offset];

    if( p == 0 ) return 0;

#ifndef NDEBUG
    p->debug_init( );
    debug_inner_test_consistency( offset, p, edgepoint_index_table, chained_edgecoords );
    p->debug_add( p->coord );
#endif // NDEBUG

    float dist; // scalar to compute the distance ratio

    TriplePoint* current = vote::find_befor( p, edgepoint_index_table, chained_edgecoords );
    // Here current contains the edge point lying on the 2nd ellipse (from outer to inner)
    if( not current ) {
        return 0;
    }
#ifndef NDEBUG
    p->debug_add( current->coord );
#endif

    // To save all sub-segments length
    int       vDistSize = 0;
#ifndef NDEBUG
    const int vDistMax  = numCrowns * 2 - 1;
#endif // NDEBUG
    float     vDist[RESERVE_MEM_MAX_CROWNS * 2 - 1];
    int flagDist = 1;

    // Length of the reconstructed field line approximation between the two
    // extremities.
    float totalDistance = 0.0;

    // compute difference in subsequent gradients orientation
    float cosDiffTheta = -vote::inner_prod( p, current );
    if( cosDiffTheta < 0.0 ) {
        return 0;
    }

    float lastDist = vote::distance( p, current ); // hypotf is CUDA float intrinsic for sqrt(pow2+pow2)
    vDist[vDistSize++] = lastDist;
    assert( vDistSize <= vDistMax );
        
    // Add the sub-segment length to the total distance.
    totalDistance += lastDist;

    TriplePoint* chosen = 0;

    // Iterate over all crowns
    for( int i=1; i < numCrowns; ++i ) {
        chosen = 0;

        // First in the gradient direction
        TriplePoint* target = vote::find_after( current,
                                                edgepoint_index_table,
                                                chained_edgecoords );
        // No edge point was found in that direction
        if( not target ) {
            return 0;
        }
#ifndef NDEBUG
        p->debug_add( target->coord );
#endif

        // Check the difference of two consecutive angles
        cosDiffTheta = -vote::inner_prod( target, current );
        if( cosDiffTheta < 0.0 ) {
            return 0;
        }

        dist = vote::distance( target, current );
        vDist[vDistSize++] = dist;
        assert( vDistSize <= vDistMax );
        totalDistance += dist;

        // Check the distance ratio
        if( vDistSize > 1 ) {
            for( int iDist = 0; iDist < vDistSize; ++iDist ) {
                for (int jDist = iDist + 1; jDist < vDistSize; ++jDist) {
                    flagDist = (vDist[iDist] <= vDist[jDist] * ratioVoting) && (vDist[jDist] <= vDist[iDist] * ratioVoting) && flagDist;
                }
            }
        }

        if( not flagDist ) {
            return 0;
        }

        // lastDist = dist;
        current = target;
        // Second in the opposite gradient direction
        // target = vote::find_befor( current, d_next_edge_befor, chained_edgecoord_list );
        target = vote::find_befor( current,
                                   edgepoint_index_table,
                                   chained_edgecoords );
        if( not target ) {
            return 0;
        }

        cosDiffTheta = -vote::inner_prod( target, current );
        if( cosDiffTheta < 0.0 ) {
            return 0;
        }

        dist = vote::distance( target, current );
        vDist[vDistSize++] = dist;
        assert( vDistSize <= vDistMax );
        totalDistance += dist;

        for( int iDist = 0; iDist < vDistSize; ++iDist ) {
            for (int jDist = iDist + 1; jDist < vDistSize; ++jDist) {
                flagDist = (vDist[iDist] <= vDist[jDist] * ratioVoting) && (vDist[jDist] <= vDist[iDist] * ratioVoting) && flagDist;
            }
        }

        if( not flagDist ) {
            return 0;
        }

        // lastDist = dist;
        current = target;
        chosen = current;

        if( !current ) {
            return 0;
        }
#ifndef NDEBUG
        p->debug_add( current->coord );
#endif
    }
#ifndef NDEBUG
    p->debug_commit( );
#endif

    /* The overhead of competing updates in the chosen points
     * would be huge.
     * But every point chooses at most one chosen, so we can
     * keep the important data in the choosers for now, and
     * update the chosen in a new kernel.
     */
    p->my_vote            = edgepoint_index_table.ptr(chosen->coord.y)[chosen->coord.x];
    p->chosen_flow_length = totalDistance;

    return chosen;
}

#ifndef NDEBUG
__device__ int count_choices = 0;

__global__
void init_choices( )
{
    count_choices = 0;
}

__global__
void print_choices( )
{
    printf("    The number of points chosen is %d\n", count_choices );
}
#endif // NDEBUG

__global__
void construct_line( DevEdgeList<int>             edge_indices,       // output
                     DevEdgeList<TriplePoint>     chained_edgecoords, // input (modified)
                     const int                    edge_index_max,     // input
                     const cv::cuda::PtrStepSz32s edgepoint_index_table, // input
                     const size_t                 numCrowns,
                     const float                  ratioVoting )
{
    const TriplePoint* chosen =
        construct_line_inner( chained_edgecoords,     // input
                              edgepoint_index_table, // input
                              numCrowns,
                              ratioVoting );
    int idx = 0;
    uint32_t mask   = __ballot( chosen != 0 );
    uint32_t ct     = __popc( mask );
    if( ct == 0 ) return;

    uint32_t write_index;
    if( threadIdx.x == 0 ) {
        write_index = atomicAdd( edge_indices.size, (int)ct );
    }
    write_index = __shfl( write_index, 0 );
    write_index += __popc( mask & ((1 << threadIdx.x) - 1) );

    if( chosen ) {
#ifndef NDEBUG
        atomicAdd( &count_choices, 1 );
#endif

        if( edge_indices.Size() < edge_index_max ) {
            idx = edgepoint_index_table.ptr(chosen->coord.y)[chosen->coord.x];
            edge_indices.ptr[write_index] = idx;
        }
    }
}

__device__ inline
int count_winners( const int                       chosen_edge_index,
                   TriplePoint*                    chosen_edge,
                   const DevEdgeList<TriplePoint>& array )
{
    int   winner_size = 0;
    float flow_length = 0.0f;

    /* This loop looks dangerous, but it is actually faster than
     * a manually partially unrolled loop.
     */
    const int voter_list_size = array.Size();
    for( int i=0; i<voter_list_size; i++ )
    // for( int i=0; i<chained_edgecoords.Size(); i++ )
    {
        if( array.ptr[i].my_vote == chosen_edge_index ) {
            winner_size += 1;
            flow_length += array.ptr[i].chosen_flow_length;
        }
    }
    chosen_edge->_winnerSize = winner_size;
    chosen_edge->_flowLength = flow_length / winner_size;
    return winner_size;
}

} // namespace vote

#ifndef COMPRESS_VOTING_AND_SELECT
/* For all chosen inner points, compute the average flow length and the
 * number of voters, and store in the TriplePoint structure of the chosen
 * inner point.
 *
 * chained_edgecoords is the list of all edges with their chaining info.
 * edge_indices is a list of indices into that list, containing the sorted,
 * unique indices of chosen inner points.
 */
__global__
void vote_eval_chosen( DevEdgeList<TriplePoint> chained_edgecoords, // input
                       DevEdgeList<int>         edge_indices        // input
                     )
{
    uint32_t offset = threadIdx.x + blockIdx.x * 32;
    if( offset >= edge_indices.Size() ) {
        return;
    }

    const int    chosen_edge_index = edge_indices.ptr[offset];
    TriplePoint* chosen_edge = &chained_edgecoords.ptr[chosen_edge_index];
#if 1
    vote::count_winners( chosen_edge_index, chosen_edge, chained_edgecoords );
#else

    int          winner_size = 0;
    float        flow_length = 0.0f;

    /* This loop looks dangerous, but it is actually faster than
     * a manually partially unrolled loop.
     */
    const int voter_list_size = chained_edgecoords.Size();
    for( int i=0; i<voter_list_size; i++ )
    // for( int i=0; i<chained_edgecoords.Size(); i++ )
    {
        if( chained_edgecoords.ptr[i].my_vote == chosen_edge_index ) {
            winner_size += 1;
            flow_length += chained_edgecoords.ptr[i].chosen_flow_length;
        }
    }
    chosen_edge->_winnerSize = winner_size;
    chosen_edge->_flowLength = flow_length / winner_size;
#endif
}
#endif // not COMPRESSED

struct NumVotersIsGreaterEqual
{
    DevEdgeList<TriplePoint> _array;
    int                      _compare;

    HIPCUB_RUNTIME_FUNCTION
    __host__ __device__
    __forceinline__
    NumVotersIsGreaterEqual( int compare, DevEdgeList<TriplePoint> _d_array )
        : _compare(compare)
        , _array( _d_array )
    {}

#ifdef COMPRESS_VOTING_AND_SELECT
    // HIPCUB_RUNTIME_FUNCTION
    __device__
    __forceinline__
    bool operator()(const int &a) const {
        TriplePoint* chosen_edge = &_array.ptr[a];
        int winner_size = vote::count_winners( a, chosen_edge, _array );
        return (winner_size >= _compare);
    }
#else // not COMPRESS_VOTING_AND_SELECT
    // HIPCUB_RUNTIME_FUNCTION
    __device__
    __forceinline__
    bool operator()(const int &a) const {
        return (_array.ptr[a]._winnerSize >= _compare);
    }
#endif // not COMPRESS_VOTING_AND_SELECT
};

__host__
bool Voting::gradientDescent( const cctag::Parameters&     params,
                              const cv::cuda::PtrStepSzb   edge_image,
                              const cv::cuda::PtrStepSz16s d_dx,
                              const cv::cuda::PtrStepSz16s d_dy,
                              hipStream_t                 stream )
{
    cout << "  Enter " << __FUNCTION__ << endl;

    int listsize;

    // Note: right here, Dynamic Parallelism would avoid blocking.
    POP_CUDA_MEMCPY_TO_HOST_ASYNC( &listsize, _all_edgecoords.dev.size, sizeof(int), stream );
    POP_CUDA_SYNC( stream );

    if( listsize == 0 ) {
        cerr << "    I have not found any edges!" << endl;
        cerr << "  Leave " << __FUNCTION__ << endl;
        return false;
    }

    const uint32_t nmax          = params._distSearch;
    const int32_t  threshold     = params._thrGradientMagInVote;
    dim3           block;
    dim3           grid;
    block.x = 32;
    block.y = 2;
    block.z = 1;
    grid.x  = listsize / 32 + ( listsize % 32 != 0 ? 1 : 0 );
    grid.y  = 1;
    grid.z  = 1;

    /* Note: the initial _chained_edgecoords.dev.size is set to 1 because it is used
     * as an index for writing points into an array. Starting the counter
     * at 1 allows to distinguish unchained points (0) from chained
     * points non-0.
     */
    POP_CUDA_SETX_ASYNC( _chained_edgecoords.dev.size, (int)1, stream );

#ifndef NDEBUG
    cout << "    calling gradient descent with " << listsize << " edge points" << endl;
    cout << "    max num edges is " << params._maxEdges << endl;

    cout << "    grid (" << grid.x << "," << grid.y << "," << grid.z << ")"
         << " block (" << block.x << "," << block.y << "," << block.z << ")" << endl;
#endif // NDEBUG

    vote::gradient_descent
        <<<grid,block,0,stream>>>
        ( _all_edgecoords.dev,
          _chained_edgecoords.dev,  // output - TriplePoints with before/after info
          _d_edgepoint_index_table, // output - table, map coord to TriplePoint index
          params._maxEdges,
          edge_image, nmax, d_dx, d_dy, threshold );
    POP_CHK_CALL_IFSYNC;

    cout << "  Leave " << __FUNCTION__ << endl;
    return true;
}

__host__
bool Voting::constructLine( const cctag::Parameters&     params,
                            hipStream_t                 stream )
{
    cout << "  Enter " << __FUNCTION__ << endl;

    // Note: right here, Dynamic Parallelism would avoid blocking.
    POP_CUDA_MEMCPY_TO_HOST_ASYNC( &_chained_edgecoords.host.size,
                                   _chained_edgecoords.dev.size,
                                   sizeof(int), stream );
    POP_CUDA_SYNC( stream );

    int listsize = _chained_edgecoords.host.size;

    cout << "    after gradient descent, edge counter is " << listsize << endl;

    if( listsize == 0 ) {
        cout << "  Leave " << __FUNCTION__ << endl;
        return false;
    }

    dim3 block;
    dim3 grid;

    block.x = 32;
    block.y = 1;
    block.z = 1;
    grid.x  = listsize / 32 + ( listsize % 32 != 0 ? 1 : 0 );
    grid.y  = 1;
    grid.z  = 1;

    POP_CUDA_SET0_ASYNC( _edge_indices.dev.size, stream );

#ifndef NDEBUG
    vote::init_choices<<<1,1,0,stream>>>( );
#endif // NDEBUG

    vote::construct_line
        <<<grid,block,0,stream>>>
        ( _edge_indices.dev,        // output
          _chained_edgecoords.dev,  // input
          params._maxEdges,         // input
          _d_edgepoint_index_table, // input
          params._nCrowns,          // input
          params._ratioVoting );    // input
    POP_CHK_CALL_IFSYNC;

#ifndef NDEBUG
    vote::print_choices<<<1,1,0,stream>>>( );
    POP_CHK_CALL_IFSYNC;
#endif // NDEBUG

    cout << "  Leave " << __FUNCTION__ << endl;
    return true;
}

__host__
void Frame::applyVote( const cctag::Parameters& params )
{
    cout << "Enter " << __FUNCTION__ << endl;

    if( params._nCrowns > RESERVE_MEM_MAX_CROWNS ) {
        cerr << "Error in " << __FILE__ << ":" << __LINE__ << ":" << endl
             << "    static maximum of parameter crowns is "
             << RESERVE_MEM_MAX_CROWNS
             << ", parameter file wants " << params._nCrowns << endl
             << "    edit " << __FILE__ << " and recompile" << endl
             << endl;
        exit( -1 );
    }

    bool success;
    
    success = _vote.gradientDescent( params,
                                     _d_edges,
                                     _d_dx,
                                     _d_dy,
                                     _stream );

    if( not success ) {
        cout << "Leave " << __FUNCTION__ << endl;
        return;
    }

#ifndef NDEBUG
#ifdef  DEBUG_RETURN_AFTER_GRADIENT_DESCENT
    {
        /* All TriplePoints have been created.
         */
        cout << "Leave " << __FUNCTION__ << endl;
        return;
    }
#endif //  DEBUG_RETURN_AFTER_GRADIENT_DESCENT
#endif // NDEBUG

    success = _vote.constructLine( params,
                                   _stream );

    if( not success ) {
        cout << "Leave " << __FUNCTION__ << endl;
        return;
    }

    /* For every chosen, compute the average flow size from all
     * of its voters, and count the number of its voters.
     */
    POP_CUDA_MEMCPY_TO_HOST_ASYNC( &_vote._edge_indices.host.size, _vote._edge_indices.dev.size, sizeof(int), _stream );
    POP_CUDA_SYNC( _stream );

#ifndef NDEBUG
#ifdef  DEBUG_RETURN_AFTER_CONSTRUCT_LINE
    {
        /* _vote._edge_indices contains now the indices of all TriplePoints that
         * have received at least one vote.
         * The array has lots of redundant entries. It is not sorted, and the
         * number of voters has not been counted, and it has not been filtered
         * by length or voters count.
         */
        cout << "Leave " << __FUNCTION__ << endl;
        return;
    }
#endif //  DEBUG_RETURN_AFTER_CONSTRUCT_LINE
#endif // NDEBUG

    if( _vote._edge_indices.host.size > 0 ) {
        /* Note: we use the intermediate picture plane, _d_intermediate, as assist
         *       buffer for CUB algorithms. It is extremely likely that this plane
         *       is large enough in all cases. If there are any problems, call
         *       the function with assist_buffer=0, and the function will return
         *       the required size in assist_buffer_sz (call by reference).
         */
        void*  assist_buffer = (void*)_d_intermediate.data;
        size_t assist_buffer_sz = _d_intermediate.step * _d_intermediate.rows;

        hipcub::DoubleBuffer<int> d_keys( _vote._edge_indices.dev.ptr,
                                       _vote._edge_indices_2.dev.ptr );

        /* After SortKeys, both buffers in d_keys have been altered.
         * The final result is stored in d_keys.d_buffers[d_keys.selector].
         * The other buffer is invalid.
         */
        hipcub::DeviceRadixSort::SortKeys( assist_buffer,
                                        assist_buffer_sz,
                                        d_keys,
                                        _vote._edge_indices.host.size,
                                        0,             // begin_bit
                                        sizeof(int)*8, // end_bit
                                        _stream );
        POP_CHK_CALL_IFSYNC;

        if( d_keys.d_buffers[d_keys.selector] == _vote._edge_indices_2.dev.ptr ) {
            std::swap( _vote._edge_indices.dev.ptr,   _vote._edge_indices_2.dev.ptr );
            std::swap( _vote._edge_indices.dev.size,  _vote._edge_indices_2.dev.size );
            std::swap( _vote._edge_indices.host.size, _vote._edge_indices_2.host.size );
        }

        // safety: SortKeys is allowed to alter assist_buffer_sz
        assist_buffer_sz = _d_intermediate.step * _d_intermediate.rows;

        /* Unique ensure that we check every "chosen" point only once.
         * Output is in _vote._edge_indices_2.dev
         */
        hipcub::DeviceSelect::Unique( assist_buffer,
                                   assist_buffer_sz,
                                   _vote._edge_indices.dev.ptr,     // input
                                   _vote._edge_indices_2.dev.ptr,   // output
                                   _vote._edge_indices_2.dev.size,  // output
                                   _vote._edge_indices_2.host.size, // input (unchanged in sort)
                                   _stream );
        POP_CHK_CALL_IFSYNC;

        /* Without Dynamic Parallelism, we must block here to retrieve the
         * value d_num_selected_out from the device before the voting
         * step.
         */
        POP_CUDA_MEMCPY_TO_HOST_ASYNC( &_vote._edge_indices_2.host.size,
                                       _vote._edge_indices_2.dev.size,
                                       sizeof(int), _stream );
        POP_CUDA_SYNC( _stream );

#ifdef COMPRESS_VOTING_AND_SELECT
        /* The computation of vote_eval_chosen can be
         * included into NumVotersIsGreaterEqual::operator()
         */
#else // COMPRESS_VOTING_AND_SELECT
        /* Add number of voters to chosen inner points, and
         * add average flow length to chosen inner points.
         */
        dim3 block;
        dim3 grid;

        block.x = 32;
        block.y = 1;
        block.z = 1;
        grid.x  = _vote._edge_indices_2.host.size / 32 + ( _vote._edge_indices_2.host.size % 32 != 0 ? 1 : 0 );
        grid.y  = 1;
        grid.z  = 1;

        vote_eval_chosen
            <<<grid,block,0,_stream>>>
            ( _vote._chained_edgecoords.dev,
              _vote._edge_indices_2.dev );
        POP_CHK_CALL_IFSYNC;
#endif // COMPRESS_VOTING_AND_SELECT

        // safety: SortKeys is allowed to alter assist_buffer_sz
        assist_buffer_sz = _d_intermediate.step * _d_intermediate.rows;

        /* Filter all chosen inner points that have fewer
         * voters than required by Parameters.
         */
        NumVotersIsGreaterEqual select_op( params._minVotesToSelectCandidate,
                                           _vote._chained_edgecoords.dev );
        hipcub::DeviceSelect::If( assist_buffer,
                               assist_buffer_sz,
                               _vote._edge_indices_2.dev.ptr,
                               _vote._edge_indices.dev.ptr,
                               _vote._edge_indices.dev.size,
                               _vote._edge_indices_2.host.size,
                               select_op,
                               _stream );
        POP_CHK_CALL_IFSYNC;

        /* Without Dynamic Parallelism, we must block here to retrieve the
         * value d_num_selected_out from the device before the voting
         * step.
         */
        POP_CUDA_MEMCPY_TO_HOST_ASYNC( &_vote._edge_indices.host.size, _vote._edge_indices.dev.size, sizeof(int), _stream );
        POP_CUDA_SYNC( _stream );

        cout << "  Number of viable inner points: " << _vote._edge_indices.host.size << endl;
    }
    cout << "Leave " << __FUNCTION__ << endl;
}

} // namespace popart


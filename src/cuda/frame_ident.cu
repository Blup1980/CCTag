#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "frame.h"
#include "clamp.h"
#include "geom_matrix.h"

using namespace std;

namespace popart {

namespace identification {

struct CutStruct
{
    float2 start;
    float2 stop;
    float  beginSig;
    float  endSig;
    int    sigSize;
};

struct NearbyPoint
{
    float result;
    int   resSize;
    bool  readable;

    /* These homographies are computed once for each NearbyPoint,
     * and used for all of its Cuts. The best one must be returned.
     */
    popart::geometry::matrix3x3 mHomography;
    popart::geometry::matrix3x3 mInvHomography;
};

struct CutSignals
{
    uint32_t outOfBounds;
    float    sig[127];
};

__device__
inline float getPixelBilinear( cv::cuda::PtrStepSzb src, float2 xy )
{
    int px = (int)xy.x; // floor of x
    int py = (int)xy.y; // floor of y
#if 0
    if( px != clamp( px, src.cols-1 ) ) {
        printf("Should clamp px from %d to %d\n", px, clamp( px, src.cols-1 ) );
    }
    if( py != clamp( py, src.rows-1 ) ) {
        printf("Should clamp py from %d to %d\n", py, clamp( py, src.rows-1 ) );
    }
    px = clamp( px, src.cols-1 );
    py = clamp( py, src.rows-1 );
#endif

    // uint8_t p0 = src.ptr(py  )[px  ];
    uint8_t p1 = src.ptr(py  )[px  ];
    uint8_t p2 = src.ptr(py  )[px+1];
    uint8_t p3 = src.ptr(py+1)[px  ];
    uint8_t p4 = src.ptr(py+1)[px+1];

    // Calculate the weights for each pixel
    float fx  = xy.x - (float)px;
    float fy  = xy.y - (float)py;
    float fx1 = 1.0f - fx;
    float fy1 = 1.0f - fy;

    float w1 = fx1 * fy1;
    float w2 = fx  * fy1;
    float w3 = fx1 * fy;
    float w4 = fx  * fy;

    // Calculate the weighted sum of pixels (for each color channel)
    return ( p1 * w1 + p2 * w2 + p3 * w3 + p4 * w4 ) / 2.0f;
}

__device__
void extractSignalUsingHomography( float*                             cut_ptr,
                                   cv::cuda::PtrStepSzb               src,
                                   const popart::geometry::matrix3x3& mHomography,
                                   const popart::geometry::matrix3x3& mInvHomography )
{
    CutStruct* cut = reinterpret_cast<CutStruct*>( cut_ptr );
    float*     cut_signals = &cut_ptr[8];

    if( threadIdx.x == 0 ) {
        cut->outOfBounds = 0;
#if 0
        for( int i=0; i<cut->sigSize; i++ ) {
            cut_signals[i] = 0.0f;
        }
#endif
    }
    __syncthreads();

    float2 backProjStop;

    backProjStop = mInvHomography.applyHomography( cut->stop );
  
    const float xStart = backProjStop.x * cut->beginSig;
    const float yStart = backProjStop.y * cut->beginSig;
    const float xStop  = backProjStop.x * cut->endSig; // xStop and yStop must not be normalised but the 
    const float yStop  = backProjStop.y * cut->endSig; // norm([xStop;yStop]) is supposed to be close to 1.

    // Compute the steps stepX and stepY along x and y.
    const std::size_t nSamples = cut->sigSize;
    const float stepX = ( xStop - xStart ) / ( nSamples - 1.0f );
    const float stepY = ( yStop - yStart ) / ( nSamples - 1.0f );
    const float stepX32 = 32.0f * stepX;
    const float stepY32 = 32.0f * stepY;

    // float x =  xStart; - serial code
    // float y =  yStart; - serial code
    float x =  xStart + threadIdx.x * stepX;
    float y =  yStart + threadIdx.x * stepY;
    for( std::size_t i = threadIdx.x; i < nSamples; i += 32 ) {
        float2 xyRes;

        // [xRes;yRes;1] ~= mHomography*[x;y;1.0]
        xyRes = mHomography.applyHomography( x, y );

        bool breaknow = ( xyRes.x < 1.0f && xyRes.x > src.cols-1 && xyRes.y < 1.0f && xyRes.y > src.rows-1 );

        if( __any( breaknow ) )
        {
            if( threadIdx.x == 0 ) cut->outOfBounds = 1;
            return;
        }

        // Bilinear interpolation
        cut_signals[i] = popart::identification::getPixelBilinear( src, xyRes );

        x += stepX32;
        y += stepY32;
    }
}

__global__
void idMakeHomographies( popart::geometry::ellipse ellipse,
                         const float2 center,
                         popart::geometry::matrix3x3* dev_homographies )
{
    // m1: popart::geometry::matrix3x3 mHomography;
    // m2: popart::geometry::matrix3x3 mInvHomography;

    popart::geometry::matrix3x3& m1 = dev_homographies[0];
    popart::geometry::matrix3x3& m2 = dev_homographies[1];
    ellipse.computeHomographyFromImagedCenter( center, m1 );
    m1.invert( m2 );
}

__global__
void idGetSignals( const popart::geometry::matrix3x3* dev_homographies,
                   cv::cuda::PtrStepSzb src,
                   float* d,
                   const int vCutsSize,
                   const int vCutMaxVecLen )
{
    int myCut = blockIdx.x * 32 + threadIdx.y;

    if( myCut >= vCutsSize ) {
        return; // out of bounds
    }

    const popart::geometry::matrix3x3& mHomography    = dev_homographies[0];
    const popart::geometry::matrix3x3& mInvHomography = dev_homographies[1];

    float* cut = &d[myCut * vCutMaxVecLen];

    extractSignalUsingHomography( cut, src, mHomography, mInvHomography);
}

__global__
void idComputeResult( FrameMetaPtr meta, float* d, const int vCutsSize, const int vCutMaxVecLen )
{
    __syncthreads();

    int myPair = blockIdx.x * 32 + threadIdx.y;
    int j      = __float2int_rd( 1.0f + __fsqrt_rd(1.0f+8.0f*myPair) ) / 2;
    int i      = myPair - j*(j-1)/2;

    int   ct   = 0;
    float val  = 0.0f;
    bool  comp = true;

    comp = ( j < vCutsSize && i < j );

    if( comp ) {
        float* l     = &d[i * vCutMaxVecLen];
        float* r     = &d[j * vCutMaxVecLen];
        comp  = ( threadIdx.x < vCutMaxVecLen ) &&
                not reinterpret_cast<CutStruct*>(l)->outOfBounds &&
                not reinterpret_cast<CutStruct*>(r)->outOfBounds;
        if( comp ) {
            int    limit = reinterpret_cast<CutStruct*>(l)->sigSize;
            for( int offset = threadIdx.x; offset < limit; offset += 32 ) {
                float square = l[8+offset]-r[8+offset];
                // val = __fmaf_rn( square, square, val );
                val += ( square * square );
            }
            ct = 1;
        }
    }

    val += __shfl_down( val, 16 );
    val += __shfl_down( val,  8 );
    val += __shfl_down( val,  4 );
    val += __shfl_down( val,  2 );
    val += __shfl_down( val,  1 );

    __shared__ float signal_sum[32];
    __shared__ int   count_sum[32];

    if( threadIdx.x == 0 ) {
        signal_sum[threadIdx.y] = val;
        count_sum [threadIdx.y] = ct;
    }
    __syncthreads();

    if( threadIdx.y == 0 ) {
        val = signal_sum[threadIdx.x];
        val += __shfl_down( val, 16 );
        val += __shfl_down( val,  8 );
        val += __shfl_down( val,  4 );
        val += __shfl_down( val,  2 );
        val += __shfl_down( val,  1 );
        ct  = count_sum[threadIdx.x];
        ct  += __shfl_down( ct, 16 );
        ct  += __shfl_down( ct,  8 );
        ct  += __shfl_down( ct,  4 );
        ct  += __shfl_down( ct,  2 );
        ct  += __shfl_down( ct,  1 );

        if( threadIdx.x == 0 ) {
            atomicAdd( &meta.identification_result(), val );
            atomicAdd( &meta.identification_resct(),  ct );
        }
    }
    __threadfence();
}

} // namespace identification


__host__
double Frame::idCostFunction( const popart::geometry::ellipse& ellipse,
                              const float2                     center,
                              const int                        vCutsSize,
                              const int                        vCutMaxVecLen,
                              bool&                            readable )
{
    readable  = true;

    dim3 block;
    dim3 grid;
    block.x = 32; // we use this to sum up signals
    block.y = 32; // we can use some shared memory/warp magic for summing
    block.z = 1;
    grid.x  = grid_divide( vCutsSize, 32 );
    grid.y  = 1;
    grid.z  = 1;

#if 0
    cerr << "GPU: #vCuts=" << vCutsSize << " vCutMaxLen=" << vCutMaxVecLen
         << " grid=(" << grid.x << "," << grid.y << "," << grid.z << ")"
         << " block=(" << block.x << "," << block.y << "," << block.z << ")"
         << endl;
#endif

    /* misusing another image-sized plane */
    popart::geometry::matrix3x3* dev_homographies = (popart::geometry::matrix3x3*)_d_map.data;

    identification::idMakeHomographies
        <<<1,1,0,_stream>>>
        ( ellipse,
          center,
          dev_homographies );

    identification::idGetSignals
        <<<grid,block,0,_stream>>>
        ( dev_homographies,
          _d_plane,
          _d_intermediate.data,
          vCutsSize,
          vCutMaxVecLen );

    int numPairs = vCutsSize*(vCutsSize-1)/2;
    block.x = 32; // we use this to sum up signals
    block.y = 32; // we can use some shared memory/warp magic for summing
    block.z = 1;
    grid.x  = grid_divide( numPairs, 32 );
    grid.y  = 1;
    grid.z  = 1;

    // _meta.toDevice( Identification_result, 0.0f, _stream );
    // _meta.toDevice( Identification_resct,  0,    _stream );

    popart::identification::idComputeResult2
        <<<grid,block>>>
        ( nPoint, cut_buffer, signals, vCutsSize, vCutMaxVecLen );
}

__global__
void idBestNearbyPoint( NearbyPoint* point_buffer )
{
}

__host__
double Frame::idCostFunction( const popart::geometry::ellipse&    ellipse,
                              const popart::geometry::ellipse&    mInvT,
                              const float2                        center,
                              const std::vector<cctag::ImageCut>& vCuts,
                              const int                           vCutMaxVecLen,
                              const float                         neighbourSize,
                              const size_t                        gridNSample )
{
    const size_t g = gridNSample * gridNSample;
    if( g*sizeof(NearbyPoint) > getNearbyPointBufferByteSize() ) {
        cerr << __FILE__ << ":" << __LINE__
             << "ERROR: re-interpreted image plane too small to hold point search rsults" << endl;
        exit( -1 );
    }

    if( vCuts.size() * sizeof(CutStruct) > getCutStructBufferByteSize() ) {
        cerr << __FILE__ << ":" << __LINE__
             << "ERROR: re-interpreted image plane too small to hold all intermediate homographies" << endl;
        exit( -1 );
    }

    clearSignalBuffer( );
    uploadCuts( vCuts );

    /* reusing various image-sized plane */
    identification::NearbyPoint* point_buffer;
    identification::CutStruct*   cut_buffer;
    identification::CutSignals*  signal_buffer;

    point_buffer  = getNearbyPointBuffer();
    cut_buffer    = getCutStructBuffer();
    signal_buffer = getSignalBuffer();

    dim3 block( 32, 32, 1 );
    dim3 grid( grid_divide( gridNSample, 32 ),
               grid_divide( gridNSample, 32 ),
               1 );

    popart::identification::idNearbyPointDispatcher
        <<<grid,block,0,_stream>>>
        ( _d_meta,
          ellipse,
          vCuts.size(),
          vCutMaxVecLen,
          mInvT,
          center,
          neighbourSize,
          gridNSample,
          point_buffer,
          cut_buffer,
          signal_buffer );

    block.x = 32;
    block.y = 1:
    block.z = 1;
    grid.x  = grid_divide( gridNSample, 32 );
    grid.y  = gridNSample;
    grid.z  = 1;

    popart::identification::idBestNearbyPoint
        <<<grid,block,0,_stream>>>
        ( point_buffer );
}

__host__
size_t Frame::getCutStructBufferByteSize( ) const
{
    /* these are uint8_t */
    return _d_mag.data * _d_mag.step;
}

__host__
identification::CutStruct* Frame::getCutStructBuffer( ) const
{
    return reinterpret_cast<CutStruct*>( _d_mag.data );
}

__host__
identification::CutStruct* Frame::getCutStructBufferHost( ) const
{
    return reinterpret_cast<CutStruct*>( _h_mag.data );
}

__host__
size_t Frame::getNearbyPointBufferByteSize( ) const
{
    /* these are uint32_t */
    return _d_map.rows * _d_map.step;
}

__host__
identification::NearbyPoint* Frame::getNearbyPointBuffer( ) const
{
    return reinterpret_cast<NearbyPoint*>( _d_map.data );
}

__host__
size_t Frame::getSignalBufferByteSize( ) const
{
    /* these are float */
    return _d_intermediate.rows * _d_intermediate.step * sizeof(float);
}

__host__
identification::CutSignals* Frame::getSignalBuffer( ) const
{
    return reinterpret_cast<CutSignals*>( _d_intermediate.data );
}

__host__
void Frame::clearSignalBuffer( )
{
#ifdef DEBUG_FRAME_UPLOAD_CUTS
    if( _d_intermediate.step != _h_intermediate.step ||
        _d_intermediate.rows != _h_intermediate.rows ) {
        cerr << "intermediate dimensions should be identical on host and dev"
             << endl;
        exit( -1 );
    }
    POP_CUDA_MEMSET_ASYNC( _d_intermediate.data,
                           -1,
                           _h_intermediate.step * _h_intermediate.rows,
                           _stream );
#endif // DEBUG_FRAME_UPLOAD_CUTS

    // cerr << "GPU: uploading " << vCuts.size() << " cuts to GPU" << endl;

    using namespace popart::identification;

    float* d = _h_intermediate.data;
    std::vector<cctag::ImageCut>::const_iterator vit  = vCuts.begin();
    std::vector<cctag::ImageCut>::const_iterator vend = vCuts.end();
    for( ; vit!=vend; vit++ ) {
        CutStruct* csptr = (CutStruct*)d;
        csptr->start.x     = vit->start().getX();
        csptr->start.y     = vit->start().getY();
        csptr->stop.x      = vit->stop().getX();
        csptr->stop.y      = vit->stop().getY();
        csptr->outOfBounds = vit->outOfBounds() ? 1 : 0;
        csptr->beginSig    = vit->beginSig();
        csptr->endSig      = vit->endSig();
        csptr->sigSize     = vit->imgSignal().size();
        int idx = 8;
        boost::numeric::ublas::vector<double>::const_iterator sit  = vit->imgSignal().begin();
        boost::numeric::ublas::vector<double>::const_iterator send = vit->imgSignal().end();
        for( ; sit!=send; sit++ ) {
            d[idx] = *sit;
            idx++;
        }
        assert( idx <= vCutMaxVecLen );

        d += vCutMaxVecLen;
    }

    POP_CUDA_MEMCPY_TO_DEVICE_ASYNC( _d_intermediate.data, _h_intermediate.data, vCuts.size()*vCutMaxVecLen*sizeof(float), _stream );
}

}; // namespace popart


#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <assert.h>
#include <fstream>
#include <string.h>
#include <hip/hip_runtime.h>
#include "debug_macros.hpp"

#include "frame.h"
// #include "clamp.h"
// #include "frame_gaussian.h"

namespace popart {

using namespace std;

/*************************************************************
 * Frame
 *************************************************************/

Frame::Frame( uint32_t width, uint32_t height )
    : _h_debug_plane( 0 )
    , _h_debug_gauss_plane( 0 )
    , _texture( 0 )
    , _stream_inherited( false )
{
    cerr << "Allocating frame: " << width << "x" << height << endl;

    POP_CUDA_STREAM_CREATE( &_stream );

    size_t pitch;
    POP_CUDA_MALLOC_PITCH( (void**)&_d_plane.data, &pitch, width, height );
    _d_plane.step = pitch / _d_plane.elemSize();
    _d_plane.cols = width;
    _d_plane.rows = height;
    assert( pitch % _d_plane.elemSize() == 0 );

    POP_CUDA_MEMSET_ASYNC( _d_plane.data,
                           0,
                           _d_plane.step * _d_plane.elemSize() * _d_plane.rows,
                           _stream );
}

Frame::~Frame( )
{
    delete _h_debug_plane;
    delete _h_debug_gauss_plane;
    delete _texture;

    POP_CUDA_FREE( _d_plane.data );
    POP_CUDA_FREE( _d_gaussian_intermediate.data );
    POP_CUDA_FREE( _d_gaussian.data );
    cerr << "Released frame: " << getWidth() << "x" << getHeight() << endl;
}

void Frame::upload( const unsigned char* image )
{
    cerr << "source w=" << _d_plane.cols
         << " source pitch=" << _d_plane.cols
         << " dest pitch=" << _d_plane.step * _d_plane.elemSize()
         << " height=" << _d_plane.rows
         << endl;
    POP_CUDA_MEMCPY_2D_ASYNC( _d_plane.data,
                              getPitch(),
                              image,
                              getWidth(),
                              getWidth(),
                              getHeight(),
                              hipMemcpyHostToDevice, _stream );
}

void Frame::createTexture( FrameTexture::Kind kind )
{
    if( _texture ) delete _texture;

    _texture = new FrameTexture( _d_plane );
}

__global__
void cu_fill_from_frame( unsigned char* dst, uint32_t pitch, uint32_t width, uint32_t height, unsigned char* src, uint32_t spitch, uint32_t swidth, uint32_t sheight )
{
    uint32_t idy = blockIdx.y;
    uint32_t idx = blockIdx.x * 32 + threadIdx.x;
    if( idy >= height ) return;
    if( idx >= pitch ) return;

    dst[ idy * pitch + idx ] = src[ idy * spitch + idx ];
}

void Frame::fillFromFrame( Frame& src )
{
    cerr << "Entering " << __FUNCTION__ << endl;
    cerr << "    copying from src frame with " << src.getWidth() << "x" << src.getHeight() << endl;
    cerr << "    to dst plane           with " << getWidth() << "x" << getHeight() << endl;
    assert( _d_plane );
    dim3 grid;
    dim3 block;
    block.x = 32;
    grid.x  = getWidth() / 32;
    grid.y  = getHeight();

    cu_fill_from_frame
        <<<grid,block,0,_stream>>>
        ( _d_plane, getPitch(), getWidth(), getHeight(), src._d_plane, src.getPitch(), src.getWidth(), src.getHeight() );
}

__global__
// void cu_fill_from_texture( unsigned char* dst, uint32_t pitch, uint32_t width, uint32_t height, hipTextureObject_t tex )
void cu_fill_from_texture( cv::cuda::PtrStepSzb dst, hipTextureObject_t tex )
{
    uint32_t idy = blockIdx.y;
    uint32_t idx = blockIdx.x * 32 + threadIdx.x;
    if( idy >= dst.rows ) return;
    if( idx >= dst.step ) return;
    bool nix = ( idx < dst.cols );
    float d = tex2D<float>( tex, float(idx)/float(dst.cols), float(idy)/float(dst.rows) );
    dst.ptr(idy)[idx] = nix ? (unsigned char)( d * 255 ) : 0;
    // dst[ idy * dst.step + idx ] = nix ? (unsigned char)( d * 255 ) : 0;
}

void Frame::fillFromTexture( Frame& src )
{
    dim3 grid;
    dim3 block;
    block.x = 32;
    grid.x  = getWidth() / 32;
    grid.y  = getHeight();

    cu_fill_from_texture
        <<<grid,block,0,_stream>>>
        // ( _d_plane, getPitch(), getWidth(), getHeight(), src.getTex() );
        ( _d_plane, src.getTex() );
}

void Frame::deleteTexture( )
{
    delete _texture;
    _texture = 0;
}

void Frame::streamSync( )
{
    hipStreamSynchronize( _stream );
}

/*************************************************************
 * FrameTexture
 *************************************************************/

void FrameTexture::makeTex_Normalized_uchar_to_float( const cv::cuda::PtrStepSzb& plane )
{
    memset( &_texDesc, 0, sizeof(hipTextureDesc) );

    _texDesc.normalizedCoords = 1;                           // address 0..1 instead of 0..width/height
    _texDesc.addressMode[0]   = hipAddressModeClamp;
    _texDesc.addressMode[1]   = hipAddressModeClamp;
    _texDesc.addressMode[2]   = hipAddressModeClamp;
    _texDesc.readMode         = hipReadModeNormalizedFloat; // automatic conversion from uchar to float
    _texDesc.filterMode       = hipFilterModeLinear;        // bilinear interpolation

    memset( &_resDesc, 0, sizeof(hipResourceDesc) );
    _resDesc.resType                  = hipResourceTypePitch2D;
    _resDesc.res.pitch2D.devPtr       = plane.data;
    _resDesc.res.pitch2D.desc.f       = hipChannelFormatKindUnsigned;
    _resDesc.res.pitch2D.desc.x       = 8;
    _resDesc.res.pitch2D.desc.y       = 0;
    _resDesc.res.pitch2D.desc.z       = 0;
    _resDesc.res.pitch2D.desc.w       = 0;
    assert( plane.elemSize() == 1 );
    _resDesc.res.pitch2D.pitchInBytes = plane.step; // * plane.elemSize()
    _resDesc.res.pitch2D.width        = plane.cols;
    _resDesc.res.pitch2D.height       = plane.rows;

    hipError_t err;
    err = hipCreateTextureObject( &_texture, &_resDesc, &_texDesc, 0 );
    POP_CUDA_FATAL_TEST( err, "Could not create texture object: " );
}

FrameTexture::FrameTexture( const cv::cuda::PtrStepSzb& plane )
    : _kind( normalized_uchar_to_float )
{
    makeTex_Normalized_uchar_to_float( plane );
}

FrameTexture::~FrameTexture( )
{
    hipError_t err;
    err = hipDestroyTextureObject( _texture );
    POP_CUDA_FATAL_TEST( err, "Could not destroy texture object: " );
}

}; // namespace popart


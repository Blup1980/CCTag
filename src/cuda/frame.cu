#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <assert.h>
#include <fstream>
#include <string.h>
#include <hip/hip_runtime.h>
#include "debug_macros.hpp"

#include "frame.h"
#include "cctag/talk.hpp"
// #include "clamp.h"
// #include "frame_gaussian.h"

namespace popart {

using namespace std;

/*************************************************************
 * Frame
 *************************************************************/

Frame::Frame( uint32_t width, uint32_t height, int my_layer )
    : _layer( my_layer )
    , _h_debug_hyst_edges( 0 )
    , _h_debug_edges( 0 )
    , _texture( 0 )
    , _wait_for_upload( 0 )
    , _wait_done( 0 )
{
#warning This should be unique
    DO_TALK( cerr << "Allocating frame: " << width << "x" << height << endl; )
    _h_ring_output.data = 0;

    POP_CUDA_STREAM_CREATE( &_stream );

    size_t pitch;
    POP_CUDA_MALLOC_PITCH( (void**)&_d_plane.data, &pitch, width, height );
    _d_plane.step = pitch;
    _d_plane.cols = width;
    _d_plane.rows = height;
    assert( pitch % _d_plane.elemSize() == 0 );

    POP_CUDA_MEMSET_ASYNC( _d_plane.data,
                           0,
                           _d_plane.step * _d_plane.rows,
                           _stream );
}

Frame::~Frame( )
{
    deleteUploadEvent( );

    releaseRequiredMem( );

    // host-side plane for debugging
    delete [] _h_debug_hyst_edges;
    delete [] _h_debug_edges;

    // required host-side planes
    delete _texture;
}

void Frame::upload( const unsigned char* image )
{
    DO_TALK(
      cerr << "source w=" << _d_plane.cols
           << " source pitch=" << _d_plane.cols
           << " dest pitch=" << _d_plane.step
           << " height=" << _d_plane.rows
           << endl;)
    POP_CUDA_MEMCPY_2D_ASYNC( _d_plane.data,
                              getPitch(),
                              image,
                              getWidth(),
                              getWidth(),
                              getHeight(),
                              hipMemcpyHostToDevice, _stream );
}

void Frame::createTexture( FrameTexture::Kind kind )
{
    if( _texture ) delete _texture;

    _texture = new FrameTexture( _d_plane );
}

__global__
void cu_fill_from_frame( unsigned char* dst, uint32_t pitch, uint32_t width, uint32_t height, unsigned char* src, uint32_t spitch, uint32_t swidth, uint32_t sheight )
{
    uint32_t idy = blockIdx.y;
    uint32_t idx = blockIdx.x * 32 + threadIdx.x;
    if( idy >= height ) return;
    if( idx >= pitch ) return;

    dst[ idy * pitch + idx ] = src[ idy * spitch + idx ];
}

void Frame::fillFromFrame( Frame& src )
{
    DO_TALK(
      cerr << "Entering " << __FUNCTION__ << endl;
      cerr << "    copying from src frame with " << src.getWidth() << "x" << src.getHeight() << endl;
      cerr << "    to dst plane           with " << getWidth() << "x" << getHeight() << endl;
    )
    assert( _d_plane );
    dim3 grid;
    dim3 block;
    block.x = 32;
    grid.x  = getWidth() / 32;
    grid.y  = getHeight();

    cu_fill_from_frame
        <<<grid,block,0,_stream>>>
        ( _d_plane, getPitch(), getWidth(), getHeight(), src._d_plane, src.getPitch(), src.getWidth(), src.getHeight() );
    POP_CHK_CALL_IFSYNC;
}

__global__
// void cu_fill_from_texture( unsigned char* dst, uint32_t pitch, uint32_t width, uint32_t height, hipTextureObject_t tex )
void cu_fill_from_texture( cv::cuda::PtrStepSzb dst, hipTextureObject_t tex )
{
    uint32_t idy = blockIdx.y;
    uint32_t idx = blockIdx.x * 32 + threadIdx.x;
    if( idy >= dst.rows ) return;
    if( idx >= dst.step ) return;
    bool nix = ( idx < dst.cols );
    float d = tex2D<float>( tex, float(idx)/float(dst.cols), float(idy)/float(dst.rows) );
    dst.ptr(idy)[idx] = nix ? (unsigned char)( d * 255 ) : 0;
}

void Frame::fillFromTexture( Frame& src )
{
    dim3 grid;
    dim3 block;
    block.x = 32;
    grid.x  = ( getWidth() / 32 ) + ( getWidth() % 32 == 0 ? 0 : 1 );
    grid.y  = getHeight();

    cu_fill_from_texture
        <<<grid,block,0,_stream>>>
        // ( _d_plane, getPitch(), getWidth(), getHeight(), src.getTex() );
        ( _d_plane, src.getTex() );
    POP_CHK_CALL_IFSYNC;
}

void Frame::deleteTexture( )
{
    delete _texture;
    _texture = 0;
}

void Frame::allocUploadEvent( )
{
    _wait_for_upload = new FrameEvent;

    hipError_t err;
    err = hipEventCreateWithFlags( _wait_for_upload, hipEventDisableTiming );
    POP_CUDA_FATAL_TEST( err, "Could not create a non-timing event: " );
}

void Frame::deleteUploadEvent( )
{
    if( not _wait_for_upload ) return;
    hipEventDestroy( *_wait_for_upload );
    delete _wait_for_upload;
}

FrameEvent Frame::addUploadEvent( )
{
    hipError_t err;
    err = hipEventRecord( *_wait_for_upload, _stream );
    POP_CUDA_FATAL_TEST( err, "Could not insert an event into a stream: " );
    return *_wait_for_upload;
}

void Frame::allocDoneEvent( )
{
    _wait_done = new FrameEvent;

    hipError_t err;
    err = hipEventCreateWithFlags( _wait_done, hipEventDisableTiming );
    POP_CUDA_FATAL_TEST( err, "Could not create a non-timing event: " );
}

void Frame::deleteDoneEvent( )
{
    if( not _wait_done ) return;
    hipEventDestroy( *_wait_done );
    delete _wait_done;
}

FrameEvent Frame::addDoneEvent( )
{
    hipError_t err;
    err = hipEventRecord( *_wait_done, _stream );
    POP_CUDA_FATAL_TEST( err, "Could not insert an event into a stream: " );
    return *_wait_done;
}

void Frame::streamSync( )
{
    hipStreamSynchronize( _stream );
}

void Frame::streamSync( FrameEvent ev )
{
    hipStreamWaitEvent( _stream, ev, 0 );
}

/*************************************************************
 * FrameTexture
 *************************************************************/

void FrameTexture::makeTex_Normalized_uchar_to_float( const cv::cuda::PtrStepSzb& plane )
{
    memset( &_texDesc, 0, sizeof(hipTextureDesc) );

    _texDesc.normalizedCoords = 1;                           // address 0..1 instead of 0..width/height
    _texDesc.addressMode[0]   = hipAddressModeClamp;
    _texDesc.addressMode[1]   = hipAddressModeClamp;
    _texDesc.addressMode[2]   = hipAddressModeClamp;
    _texDesc.readMode         = hipReadModeNormalizedFloat; // automatic conversion from uchar to float
    _texDesc.filterMode       = hipFilterModeLinear;        // bilinear interpolation

    memset( &_resDesc, 0, sizeof(hipResourceDesc) );
    _resDesc.resType                  = hipResourceTypePitch2D;
    _resDesc.res.pitch2D.devPtr       = plane.data;
    _resDesc.res.pitch2D.desc.f       = hipChannelFormatKindUnsigned;
    _resDesc.res.pitch2D.desc.x       = 8;
    _resDesc.res.pitch2D.desc.y       = 0;
    _resDesc.res.pitch2D.desc.z       = 0;
    _resDesc.res.pitch2D.desc.w       = 0;
    assert( plane.elemSize() == 1 );
    _resDesc.res.pitch2D.pitchInBytes = plane.step;
    _resDesc.res.pitch2D.width        = plane.cols;
    _resDesc.res.pitch2D.height       = plane.rows;

    hipError_t err;
    err = hipCreateTextureObject( &_texture, &_resDesc, &_texDesc, 0 );
    POP_CUDA_FATAL_TEST( err, "Could not create texture object: " );
}

FrameTexture::FrameTexture( const cv::cuda::PtrStepSzb& plane )
    : _kind( normalized_uchar_to_float )
{
    makeTex_Normalized_uchar_to_float( plane );
}

FrameTexture::~FrameTexture( )
{
    hipError_t err;
    err = hipDestroyTextureObject( _texture );
    POP_CUDA_FATAL_TEST( err, "Could not destroy texture object: " );
}

}; // namespace popart


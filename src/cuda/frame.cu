#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <assert.h>
#include <fstream>
#include <string.h>
#include <hip/hip_runtime.h>
#include "debug_macros.hpp"

#include "frame.h"
#include "cctag/talk.hpp"
// #include "clamp.h"
// #include "frame_gaussian.h"

namespace popart {

using namespace std;

/*************************************************************
 * Frame
 *************************************************************/

Frame::Frame( uint32_t width, uint32_t height, int my_layer, hipStream_t download_stream, int my_pipe )
    : _layer( my_layer )
    , _h_debug_hyst_edges( 0 )
    , _texture( 0 )
    , _wait_for_upload( 0 )
    , _meta( my_pipe, my_layer )
{
    DO_TALK( cerr << "Allocating frame: " << width << "x" << height << endl; )
#ifndef EDGE_LINKING_HOST_SIDE
    _h_ring_output.data = 0;
#endif

    if( download_stream != 0 ) {
        _private_download_stream = false;
        _download_stream = download_stream;
    } else {
        _private_download_stream = true;
        hipStreamCreateWithFlags( &_download_stream, hipStreamNonBlocking );
        // POP_CUDA_STREAM_CREATE( &_download_stream );
    }
    POP_CUDA_STREAM_CREATE( &_stream );

    // POP_CUDA_EVENT_CREATE( &_download_ready_event );
    // at least in older CUDA versions, events blocked parallelism
    hipEventCreateWithFlags( &_stream_done,                      hipEventDisableTiming);
    hipEventCreateWithFlags( &_download_stream_done,             hipEventDisableTiming);
    hipEventCreateWithFlags( &_download_ready_event.plane,       hipEventDisableTiming);
    hipEventCreateWithFlags( &_download_ready_event.dxdy,        hipEventDisableTiming);
    hipEventCreateWithFlags( &_download_ready_event.magmap,      hipEventDisableTiming);
    hipEventCreateWithFlags( &_download_ready_event.edgecoords1, hipEventDisableTiming);
    hipEventCreateWithFlags( &_download_ready_event.edgecoords2, hipEventDisableTiming);
    hipEventCreateWithFlags( &_download_ready_event.descent1,    hipEventDisableTiming);
    hipEventCreateWithFlags( &_download_ready_event.descent2,    hipEventDisableTiming);

    size_t pitch;
    POP_CUDA_MALLOC_PITCH( (void**)&_d_plane.data, &pitch, width, height );
    _d_plane.step = pitch;
    _d_plane.cols = width;
    _d_plane.rows = height;
    assert( pitch % _d_plane.elemSize() == 0 );

    POP_CUDA_MEMSET_ASYNC( _d_plane.data,
                           0,
                           _d_plane.step * _d_plane.rows,
                           _stream );
}

Frame::~Frame( )
{
    deleteUploadEvent( );

    releaseRequiredMem( );

    // host-side plane for debugging
    delete [] _h_debug_hyst_edges;

    // required host-side planes
    delete _texture;

    hipEventDestroy( _stream_done );
    hipEventDestroy( _download_stream_done );
    hipEventDestroy( _download_ready_event.plane );
    hipEventDestroy( _download_ready_event.dxdy );
    hipEventDestroy( _download_ready_event.magmap );
    hipEventDestroy( _download_ready_event.edgecoords1 );
    hipEventDestroy( _download_ready_event.edgecoords2 );
    hipEventDestroy( _download_ready_event.descent1 );
    hipEventDestroy( _download_ready_event.descent2 );

    if( _private_download_stream ) {
        POP_CUDA_STREAM_DESTROY( _download_stream );
    }
    POP_CUDA_STREAM_DESTROY( _stream );
}

void Frame::upload( const unsigned char* image )
{
    DO_TALK(
      cerr << "source w=" << _d_plane.cols
           << " source pitch=" << _d_plane.cols
           << " dest pitch=" << _d_plane.step
           << " height=" << _d_plane.rows
           << endl;)
    POP_CUDA_MEMCPY_2D_ASYNC( _d_plane.data,
                              getPitch(),
                              image,
                              getWidth(),
                              getWidth(),
                              getHeight(),
                              hipMemcpyHostToDevice, _stream );
}

void Frame::createTexture( FrameTexture::Kind kind )
{
    if( _texture ) delete _texture;

    _texture = new FrameTexture( _d_plane );
}

#if 0
__global__
void cu_fill_from_frame( unsigned char* dst, uint32_t pitch, uint32_t width, uint32_t height, unsigned char* src, uint32_t spitch, uint32_t swidth, uint32_t sheight )
{
    uint32_t idy = blockIdx.y;
    uint32_t idx = blockIdx.x * 32 + threadIdx.x;
    if( idy >= height ) return;
    if( idx >= pitch ) return;

    dst[ idy * pitch + idx ] = src[ idy * spitch + idx ];
}

void Frame::fillFromFrame( Frame& src )
{
    DO_TALK(
      cerr << "Entering " << __FUNCTION__ << endl;
      cerr << "    copying from src frame with " << src.getWidth() << "x" << src.getHeight() << endl;
      cerr << "    to dst plane           with " << getWidth() << "x" << getHeight() << endl;
    )
    assert( _d_plane );
    dim3 grid;
    dim3 block;
    block.x = 32;
    grid.x  = getWidth() / 32;
    grid.y  = getHeight();

    cu_fill_from_frame
        <<<grid,block,0,_stream>>>
        ( _d_plane, getPitch(), getWidth(), getHeight(), src._d_plane, src.getPitch(), src.getWidth(), src.getHeight() );
    POP_CHK_CALL_IFSYNC;
}
#endif

void Frame::deleteTexture( )
{
    delete _texture;
    _texture = 0;
}

void Frame::allocUploadEvent( )
{
    _wait_for_upload = new hipEvent_t;

    hipError_t err;
    err = hipEventCreateWithFlags( _wait_for_upload, hipEventDisableTiming );
    POP_CUDA_FATAL_TEST( err, "Could not create a non-timing event: " );
}

void Frame::deleteUploadEvent( )
{
    if( not _wait_for_upload ) return;
    hipEventDestroy( *_wait_for_upload );
    delete _wait_for_upload;
}

hipEvent_t Frame::addUploadEvent( )
{
    hipError_t err;
    err = hipEventRecord( *_wait_for_upload, _stream );
    POP_CUDA_FATAL_TEST( err, "Could not insert an event into a stream: " );
    return *_wait_for_upload;
}

void Frame::streamSync( )
{
    hipStreamSynchronize( _stream );
}

void Frame::streamSync( hipEvent_t ev )
{
    hipStreamWaitEvent( _stream, ev, 0 );
}

/*************************************************************
 * FrameTexture
 *************************************************************/

void FrameTexture::makeTex_Normalized_uchar_to_float( const cv::cuda::PtrStepSzb& plane )
{
    memset( &_texDesc, 0, sizeof(hipTextureDesc) );

    _texDesc.normalizedCoords = 1;                           // address 0..1 instead of 0..width/height
    _texDesc.addressMode[0]   = hipAddressModeClamp;
    _texDesc.addressMode[1]   = hipAddressModeClamp;
    _texDesc.addressMode[2]   = hipAddressModeClamp;
    _texDesc.readMode         = hipReadModeNormalizedFloat; // automatic conversion from uchar to float
    _texDesc.filterMode       = hipFilterModeLinear;        // bilinear interpolation

    memset( &_resDesc, 0, sizeof(hipResourceDesc) );
    _resDesc.resType                  = hipResourceTypePitch2D;
    _resDesc.res.pitch2D.devPtr       = plane.data;
    _resDesc.res.pitch2D.desc.f       = hipChannelFormatKindUnsigned;
    _resDesc.res.pitch2D.desc.x       = 8;
    _resDesc.res.pitch2D.desc.y       = 0;
    _resDesc.res.pitch2D.desc.z       = 0;
    _resDesc.res.pitch2D.desc.w       = 0;
    assert( plane.elemSize() == 1 );
    _resDesc.res.pitch2D.pitchInBytes = plane.step;
    _resDesc.res.pitch2D.width        = plane.cols;
    _resDesc.res.pitch2D.height       = plane.rows;

    hipError_t err;
    err = hipCreateTextureObject( &_texture, &_resDesc, &_texDesc, 0 );
    POP_CUDA_FATAL_TEST( err, "Could not create texture object: " );
}

FrameTexture::FrameTexture( const cv::cuda::PtrStepSzb& plane )
    : _kind( normalized_uchar_to_float )
{
    makeTex_Normalized_uchar_to_float( plane );
}

FrameTexture::~FrameTexture( )
{
    hipError_t err;
    err = hipDestroyTextureObject( _texture );
    POP_CUDA_FATAL_TEST( err, "Could not destroy texture object: " );
}

}; // namespace popart


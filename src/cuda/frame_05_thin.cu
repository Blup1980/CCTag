#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "debug_macros.hpp"
#include "debug_is_on_edge.h"

#include "frame.h"
#include "assist.h"

namespace popart
{

using namespace std;

namespace thinning {

static unsigned char h_lut[256] = {
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 0, 1, 1, 1, 0, 1, 1, 1, 
        1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 
        1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 
        1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 
};

// Note that the transposed h_lut_t is not really necessary
// because flipping the 4 LSBs and 4 HSBs in the unsigned char that
// I use for lookup is fast.
static unsigned char h_lut_t[256] = {
        1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 0, 0, 
        1, 1, 1, 1, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 0, 0, 0, 0, 0, 1, 1, 1, 0, 0, 0, 0, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 0, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 
        1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 0, 1, 0, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
};

__device__ __constant__ unsigned char d_lut[256];

__device__ __constant__ unsigned char d_lut_t[256];

__device__
bool update_pixel( const int idx, const int idy, cv::cuda::PtrStepSzb src, cv::cuda::PtrStepSzb dst, bool first_run )
{
    unsigned char result = 0;
    if( src.ptr(idy)[idx] == 2 &&
        idx >= 1 && idy >=1 && idx <= src.cols-2 && idy <= src.rows-2 ) {
        uint8_t log = 0;

        log |= ( src.ptr(idy-1)[idx  ] == 2 ) ? 0x01 : 0;
        log |= ( src.ptr(idy-1)[idx+1] == 2 ) ? 0x02 : 0;
        log |= ( src.ptr(idy  )[idx+1] == 2 ) ? 0x04 : 0;
        log |= ( src.ptr(idy+1)[idx+1] == 2 ) ? 0x08 : 0;
        log |= ( src.ptr(idy+1)[idx  ] == 2 ) ? 0x10 : 0;
        log |= ( src.ptr(idy+1)[idx-1] == 2 ) ? 0x20 : 0;
        log |= ( src.ptr(idy  )[idx-1] == 2 ) ? 0x40 : 0;
        log |= ( src.ptr(idy-1)[idx-1] == 2 ) ? 0x80 : 0;

        if( first_run ) {
            result = d_lut[log] ? 2 : 0;
        } else {
            result = d_lut_t[log];
        }
    }
    __syncthreads();
    dst.ptr(idy)[idx] = result;
    return ( result != 0 );
}

__global__
void first_round( cv::cuda::PtrStepSzb src, cv::cuda::PtrStepSzb dst )
{
    const int block_x = blockIdx.x * 32;
    const int idx     = block_x + threadIdx.x;
    const int idy     = blockIdx.y;

    update_pixel( idx, idy, src, dst, true );
}

__global__
void second_round( cv::cuda::PtrStepSzb src,          // input
                   cv::cuda::PtrStepSzb dst,          // output
#ifndef NDEBUG
                   DevEdgeList<int2>    edgeCoords,   // output
                   FrameMetaPtr         meta )
#else
                   DevEdgeList<int2>    edgeCoords )  // output
#endif
{
    const int block_x = blockIdx.x * 32;
    const int idx     = block_x + threadIdx.x;
    const int idy     = blockIdx.y;

    bool keep = update_pixel( idx, idy, src, dst, false );

    if( keep ) {
        atomicAdd( &meta.num_edges_thinned(), 1 );
    }
#if 0
    uint32_t write_index;
    if( keep ) {
        write_index = atomicAdd( edgeCoords.getSizePtr(), 1 );
    }
#else
    uint32_t mask = __ballot( keep );  // bitfield of warps with results
    uint32_t ct   = __popc( mask );    // horizontal reduce
    uint32_t leader = __ffs(mask) - 1; // the highest thread id with indicator==true
    uint32_t write_index;
    if( threadIdx.x == leader ) {
        // leader gets warp's offset from global value and increases it
        write_index = atomicAdd( edgeCoords.getSizePtr(), int(ct) );
    }
    write_index = __shfl( write_index, leader ); // broadcast warp write index to all
    write_index += __popc( mask & ((1 << threadIdx.x) - 1) ); // find own write index
#endif

    if( keep ) {
        if( write_index < EDGE_POINT_MAX ) {
            edgeCoords.ptr[write_index] = make_int2( idx, idy );
        }
    }
}

__global__
void set_null( DevEdgeList<int2> edgeCoords )
{
    edgeCoords.setSize( 0 );
}

__global__
void set_edgemax( DevEdgeList<int2> edgeCoords )
{
    if( edgeCoords.Size() > EDGE_POINT_MAX ) {
        edgeCoords.setSize( EDGE_POINT_MAX );
    }
}

}; // namespace thinning

__host__
void Frame::initThinningTable( )
{
    POP_CUDA_MEMCPY_HOST_TO_SYMBOL_SYNC( thinning::d_lut,
                                         thinning::h_lut,
                                         256*sizeof(unsigned char) );
    POP_CUDA_MEMCPY_HOST_TO_SYMBOL_SYNC( thinning::d_lut_t,
                                         thinning::h_lut_t,
                                         256*sizeof(unsigned char) );
}

__host__
void Frame::applyThinning( )
{
    dim3 block( 32, 1, 1 );
    dim3 grid( grid_divide( getWidth(), 32 ),
               getHeight(),
               1 );

    thinning::first_round
        <<<grid,block,0,_stream>>>
        ( _d_hyst_edges, cv::cuda::PtrStepSzb(_d_intermediate) );
    POP_CHK_CALL_IFSYNC;

    thinning::set_null
        <<<1,1,0,_stream>>>
        ( _vote._all_edgecoords.dev );

    // POP_CUDA_SET0_ASYNC( _vote._all_edgecoords.dev.getSizePtr(), _stream );

#ifndef NDEBUG
    _meta.toDevice( Num_edges_thinned, 0, _stream );

    thinning::second_round
        <<<grid,block,0,_stream>>>
        ( cv::cuda::PtrStepSzb(_d_intermediate), // input
          _d_edges,                              // output
          _vote._all_edgecoords.dev,             // output
          _meta );

    int val;
    _meta.fromDevice( Num_edges_thinned, val, _stream );
    _vote._all_edgecoords.copySizeFromDevice( _stream, EdgeListWait );
    std::cerr << __FILE__ << ":" << __LINE__ << std::endl
              << "num of edge points after thinning: " << val << std::endl
              << "num of edge points added to list:  " << _vote._all_edgecoords.host.size << std::endl
              << "edgemax: " << EDGE_POINT_MAX << std::endl;
    _vote._all_edgecoords.copyDataFromDeviceSync( );

#else // NDEBUG
    thinning::second_round
        <<<grid,block,0,_stream>>>
        ( cv::cuda::PtrStepSzb(_d_intermediate), // input
          _d_edges,                              // output
          _vote._all_edgecoords.dev );           // output
#endif // NDEBUG

    thinning::set_edgemax
        <<<1,1,0,_stream>>>
        ( _vote._all_edgecoords.dev );

    _vote._all_edgecoords.copySizeFromDevice( _stream, EdgeListCont );
#if 0
    debugPointIsOnEdge( _d_edges, _vote._all_edgecoords, _stream );
#endif // NDEBUG

#ifdef EDGE_LINKING_HOST_SIDE
    /* After thinning_and_store, _all_edgecoords is no longer changed.
     * Make a non-blocking copy the number of items in the list to the host.
     */
    hipEventRecord( _download_ready_event.edgecoords1, _stream );
#endif // EDGE_LINKING_HOST_SIDE
}

__host__
void Frame::applyThinDownload( )
{
#ifdef EDGE_LINKING_HOST_SIDE
    /* After thinning_and_store, _all_edgecoords is no longer changed
     * we can copy it to the host for edge linking
     */
    // hipStreamWaitEvent( _download_stream, _download_ready_event.edgecoords1, 0 );

    /* CPU must wait for counter _vote._all_edgecoords.host.size */
    hipEventSynchronize( _download_ready_event.edgecoords1 );
    POP_CHK_CALL_IFSYNC;

    // hipEventSynchronize( _download_ready_event.edgecoords2 );
    _vote._all_edgecoords.copyDataFromDeviceAsync( _download_stream );
    POP_CHK_CALL_IFSYNC;
#ifndef NDEBUG
    if( _vote._all_edgecoords.host.size <= 0 ) {
        // initialize the hostside array to 0 for debugging
        _vote._all_edgecoords.initHost( );
    }
#endif // NDEBUG
#endif // EDGE_LINKING_HOST_SIDE
}

}; // namespace popart


#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "debug_macros.hpp"

#include "frame.h"
#include "clamp.h"
#include "assist.h"

namespace popart
{

using namespace std;

static unsigned char h_thinning_lut[256] = {
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 0, 1, 1, 1, 0, 1, 1, 1, 
        1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 
        1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 
        1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 1, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 1, 1, 1, 
};

// Note that the transposed h_thinning_lut_t is not really necessary
// because flipping the 4 LSBs and 4 HSBs in the unsigned char that
// I use for lookup is really quick. Therefore: remove soon.
static unsigned char h_thinning_lut_t[256] = {
        1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 0, 0, 
        1, 1, 1, 1, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 0, 0, 0, 0, 0, 1, 1, 1, 0, 0, 0, 0, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 0, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 
        1, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 0, 1, 0, 1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
};

__device__ __constant__ unsigned char d_thinning_lut[256];

__device__ __constant__ unsigned char d_thinning_lut_t[256];

__global__
void compute_mag_l1( cv::cuda::PtrStepSz16s src_dx,
                     cv::cuda::PtrStepSz16s src_dy,
                     cv::cuda::PtrStepSz32u dst )
{
    int block_x = blockIdx.x * 32;
    int idx     = block_x + threadIdx.x;
    int idy     = blockIdx.y;

    if( outOfBounds( idx, idy, dst ) ) return;

    int16_t dx = src_dx.ptr(idy)[idx];
    int16_t dy = src_dy.ptr(idy)[idx];
    dx = d_abs( dx );
    dy = d_abs( dy );
    dst.ptr(idy)[idx] = dx + dy;
}

__global__
void compute_mag_l2( cv::cuda::PtrStepSz16s src_dx,
                     cv::cuda::PtrStepSz16s src_dy,
                     cv::cuda::PtrStepSz32u dst )
{
    int block_x = blockIdx.x * 32;
    int idx     = block_x + threadIdx.x;
    int idy     = blockIdx.y;

    if( outOfBounds( idx, idy, dst ) ) return;

    int16_t dx = src_dx.ptr(idy)[idx];
    int16_t dy = src_dy.ptr(idy)[idx];
    // --- hypot --
    dx *= dx;
    dy *= dy;
    dst.ptr(idy)[idx] = __fsqrt_rz( (float)( dx + dy ) );
}

__global__
void compute_map( const cv::cuda::PtrStepSz16s dx,
                  const cv::cuda::PtrStepSz16s dy,
                  const cv::cuda::PtrStepSz32u mag,
                  cv::cuda::PtrStepSzb         map,
                  const float                  low_thresh,
                  const float                  high_thresh )
{
    const int CANNY_SHIFT = 15;
    const int TG22 = (int32_t)(0.4142135623730950488016887242097*(1<<CANNY_SHIFT) + 0.5);

    const int block_x = blockIdx.x * 32;
    const int idx     = block_x + threadIdx.x;
    const int idy     = blockIdx.y;

    if( outOfBounds( idx, idy, dx ) ) return;

    int32_t  dxVal  = dx.ptr(idy)[idx];
    int32_t  dyVal  = dy.ptr(idy)[idx];
    uint32_t magVal = mag.ptr(idy)[idx];

    // -1 if only is negative, 1 else
    // const int32_t signVal = (dxVal ^ dyVal) < 0 ? -1 : 1;
    const int32_t signVal = d_sign( dxVal ^ dyVal );

    dxVal = d_abs( dxVal );
    dyVal = d_abs( dyVal );

    // 0 - the pixel can not belong to an edge
    // 1 - the pixel might belong to an edge
    // 2 - the pixel does belong to an edge
    uint8_t edge_type = 0;

    if( magVal > low_thresh )
    {
        const int32_t tg22x = dxVal * TG22;
        const int32_t tg67x = tg22x + ((dxVal + dxVal) << CANNY_SHIFT);

        dyVal <<= CANNY_SHIFT;

        int2 x = (dyVal < tg22x) ? make_int2( idx - 1, idx + 1 )
                                 : (dyVal > tg67x ) ? make_int2( idx, idx )
                                                    : make_int2( idx - signVal, idx + signVal );
        int2 y = (dyVal < tg22x) ? make_int2( idy, idy )
                                 : make_int2( idy - 1, idy + 1 );

        x.x = clamp( x.x, dx.cols );
        x.y = clamp( x.y, dx.cols );
        y.x = clamp( y.x, dx.rows );
        y.y = clamp( y.y, dx.rows );

        if( magVal > mag.ptr(y.x)[x.x] && magVal >= mag.ptr(y.y)[x.y] ) {
            edge_type = 1 + (uint8_t)(magVal > high_thresh);
        }
    }
    __syncthreads();

    map.ptr(idy)[idx] = edge_type;
}

__device__
bool thinning_inner( const int idx, const int idy, cv::cuda::PtrStepSzb src, cv::cuda::PtrStepSzb dst, bool first_run )
{
    if( src.ptr(idy)[idx] == 0 ) {
        dst.ptr(idy)[idx] = 0;
        return false;
    }

    if( idx >= 1 && idy >=1 && idx <= src.cols-2 && idy <= src.rows-2 ) {
        uint8_t log = 0;

        log |= ( src.ptr(idy-1)[idx  ] != 0 ) ? 0x01 : 0;
        log |= ( src.ptr(idy-1)[idx+1] != 0 ) ? 0x02 : 0;
        log |= ( src.ptr(idy  )[idx+1] != 0 ) ? 0x04 : 0;
        log |= ( src.ptr(idy+1)[idx+1] != 0 ) ? 0x08 : 0;
        log |= ( src.ptr(idy+1)[idx  ] != 0 ) ? 0x10 : 0;
        log |= ( src.ptr(idy+1)[idx-1] != 0 ) ? 0x20 : 0;
        log |= ( src.ptr(idy  )[idx-1] != 0 ) ? 0x40 : 0;
        log |= ( src.ptr(idy-1)[idx-1] != 0 ) ? 0x80 : 0;

#if 1
        if( first_run )
            dst.ptr(idy)[idx] = d_thinning_lut[log];
        else
            dst.ptr(idy)[idx] = d_thinning_lut_t[log];
#else
        if( first_run == false ) {
            uint8_t b = log;
            b   = ( b   << 4 ) & 0xf0;
            log = ( ( log >> 4 ) & 0x0f ) | b;
        }

        dst.ptr(idy)[idx] = d_thinning_lut[log];
#endif
        return true;
    }
    return false;
}

__global__
void thinning( cv::cuda::PtrStepSzb src, cv::cuda::PtrStepSzb dst )
{
    const int block_x = blockIdx.x * 32;
    const int idx     = block_x + threadIdx.x;
    const int idy     = blockIdx.y;

    thinning_inner( idx, idy, src, dst, true );
}

__global__
void thinning_and_store( cv::cuda::PtrStepSzb src,          // input
                         cv::cuda::PtrStepSzb dst,          // output
                         DevEdgeList<int2>    edgeCoords,   // output
                         uint32_t             edgeMax )     // input
{
    const int block_x = blockIdx.x * 32;
    const int idx     = block_x + threadIdx.x;
    const int idy     = blockIdx.y;

    bool keep = thinning_inner( idx, idy, src, dst, false );

    uint32_t mask = __ballot( keep );  // bitfield of warps with results
    uint32_t ct   = __popc( mask );    // horizontal reduce
    uint32_t leader = __ffs(mask) - 1; // the highest thread id with indicator==true
    uint32_t write_index;
    if( threadIdx.x == leader ) {
        // leader gets warp's offset from global value and increases it
        write_index = atomicAdd( edgeCoords.size, int(ct) );
    }
    write_index = __shfl( write_index, leader ); // broadcast warp write index to all
    write_index += __popc( mask & ((1 << threadIdx.x) - 1) ); // find own write index

    if( keep && write_index < edgeMax ) {
        edgeCoords.ptr[write_index] = make_int2( idx, idy );
    }
}

__host__
void Frame::initThinningTable( )
{
    POP_CUDA_MEMCPY_HOST_TO_SYMBOL_SYNC( d_thinning_lut,
                                         h_thinning_lut,
                                         256*sizeof(unsigned char) );
    POP_CUDA_MEMCPY_HOST_TO_SYMBOL_SYNC( d_thinning_lut_t,
                                         h_thinning_lut_t,
                                         256*sizeof(unsigned char) );
}

__host__
void Frame::applyMag( const cctag::Parameters & params )
{
    cerr << "Enter " << __FUNCTION__ << endl;

    dim3 block;
    dim3 grid;
    block.x = 32;
    grid.x  = ( getWidth() / 32 ) + ( getWidth() % 32 == 0 ? 0 : 1 );
    grid.y  = getHeight();

    dim3 big_block;
    dim3 big_grid;
    big_block.x = 32;
    big_block.y = 32;
    big_grid.x  = ( getWidth()  / 32 ) + ( getWidth()  % 32 == 0 ? 0 : 1 );
    big_grid.y  = ( getHeight() / 32 ) + ( getHeight() % 32 == 0 ? 0 : 1 );

    // necessary to merge into 1 stream
    compute_mag_l2
        <<<grid,block,0,_stream>>>
        ( _d_dx, _d_dy, _d_mag );
    POP_CHK_CALL_IFSYNC;

    compute_map
        <<<grid,block,0,_stream>>>
        ( _d_dx, _d_dy, _d_mag, _d_map, 256.0f * params._cannyThrLow, 256.0f * params._cannyThrHigh );
    POP_CHK_CALL_IFSYNC;

    cerr << "Leave " << __FUNCTION__ << endl;
}

__host__
void Frame::applyThinning( const cctag::Parameters & params )
{
    cerr << "Enter " << __FUNCTION__ << endl;

    dim3 block;
    dim3 grid;
    block.x = 32;
    grid.x  = ( getWidth() / 32 ) + ( getWidth() % 32 == 0 ? 0 : 1 );
    grid.y  = getHeight();

    thinning
        <<<grid,block,0,_stream>>>
        ( _d_hyst_edges, cv::cuda::PtrStepSzb(_d_intermediate) );
    POP_CHK_CALL_IFSYNC;

    POP_CUDA_SET0_ASYNC( _vote._all_edgecoords.dev.size, _stream );

    thinning_and_store
        <<<grid,block,0,_stream>>>
        ( cv::cuda::PtrStepSzb(_d_intermediate), // input
          _d_edges,                              // output
          _vote._all_edgecoords.dev,           // output
          params._maxEdges );                    // input
    POP_CHK_CALL_IFSYNC;

    cerr << "Leave " << __FUNCTION__ << endl;
}

}; // namespace popart


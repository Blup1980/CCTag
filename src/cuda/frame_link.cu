#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_math_constants.h>

#include "frame.h"
#include "assist.h"
#include "recursive_sweep.h"

#undef  ONE_THREAD_ONLY // doesn't work?
#undef  KERNEL_PRINT_ERROR_CAUSE
#define KERNEL_PRINT_SUCCESS_CAUSE

using namespace std;

namespace popart
{

namespace linking
{
#if 0
__constant__
static int xoff_select[8]    =   { 1,  1,  0, -1, -1, -1,  0,  1};

__constant__
static int yoff_select[2][8] = { { 0, -1, -1, -1,  0,  1,  1,  1},
                                 { 0,  1,  1,  1,  0, -1, -1, -1} };
#else
__device__
static const int xoff_select[8]    =   { 1,  1,  0, -1, -1, -1,  0,  1};

__device__
static const int yoff_select[2][8] = { { 0, -1, -1, -1,  0,  1,  1,  1},
                                       { 0,  1,  1,  1,  0, -1, -1, -1} };
#endif

enum Direction
{
    Left = 0,
    Right = 1
};

enum StopCondition
{
    LOW_FLOW            = -4,
    VOTE_LOW            = -3,
    CONVEXITY_LOST      = -2,
    EDGE_NOT_FOUND      = -1,
    NONE                =  0,
    STILL_SEARCHING     =  1,
    FOUND_NEXT          =  2,
    FULL_CIRCLE         =  3
};

struct RingBuffer
{
    Direction direction;
    float     ring_buffer[EDGE_LINKING_MAX_RING_BUFFER_SIZE];
    size_t    max_size;
    size_t    front_index;
    size_t    back_index;
    size_t    ct;
    float     angle_diff;

    __device__ RingBuffer( Direction dir, const size_t size ) {
        assert( size < EDGE_LINKING_MAX_RING_BUFFER_SIZE );
        direction   = dir;
        max_size    = size;
        front_index = 0;
        back_index  = 0;
        ct          = 0;
        angle_diff  = 0.0f;
    }

    __device__ inline void set_direction( Direction dir ) {
        direction = dir;
    }

    __device__ void push_back( float angle )
    {
        if( threadIdx.x == 0 ) {
            if( direction == Left ) {
                ring_buffer[back_index] = angle;
                inc( back_index );
                if( front_index == back_index )
                    inc( front_index );
                else
                    ct++;
            } else {
                dec( front_index );
                if( front_index == back_index )
                    dec( back_index );
                else
                    ct++;
                ring_buffer[front_index] = angle;
            }
            angle_diff = back() - front();
        }
        angle_diff = __shfl( angle_diff, 0 );
        ct         = __shfl( ct, 0 );
    }

    __device__ inline float diff( )
    {
        // replace in lilian's original code back()-front()
        return angle_diff;
    }

    __device__ int  inline size() const {
        return ct;
    }
private:
    __device__ inline float front( )
    {
        assert( threadIdx.x == 0 );
        assert( front_index != back_index );
        return ring_buffer[front_index];
    }

    __device__ inline float back( )
    {
        assert( threadIdx.x == 0 );
        assert( front_index != back_index );
        size_t lookup = back_index;
        dec( lookup );
        return ring_buffer[lookup];
    }

    __device__ inline void inc( size_t& idx )
    {
        assert( threadIdx.x == 0 );
        idx = ( idx >= max_size-1 ) ? 0 : idx + 1;
    }

    __device__ inline void dec( size_t& idx )
    {
        assert( threadIdx.x == 0 );
        idx = ( idx == 0 ) ? max_size-1 : idx - 1;
    }
};

#ifdef ONE_THREAD_ONLY
// include data in class
#else // ONE_THREAD_ONLY
#ifdef DEBUG_LINKED_USE_INT4_BUFFER
__shared__ int4  edge_buffer[EDGE_LINKING_MAX_EDGE_LENGTH]; // convexEdgeSegment
#else // DEBUG_LINKED_USE_INT4_BUFFER
__shared__ int2  edge_buffer[EDGE_LINKING_MAX_EDGE_LENGTH]; // convexEdgeSegment
#endif // DEBUG_LINKED_USE_INT4_BUFFER
__shared__ int   edge_index[2];
#endif // ONE_THREAD_ONLY

struct EdgeBuffer
{
#ifdef ONE_THREAD_ONLY
    int2  edge_buffer[EDGE_LINKING_MAX_EDGE_LENGTH]; // convexEdgeSegment
    int   edge_index[2];
#endif // ONE_THREAD_ONLY
    __device__ inline
    void init( int2 start )
    {
#ifdef ONE_THREAD_ONLY
#ifdef DEBUG_LINKED_USE_INT4_BUFFER
        edge_buffer[0].x  = start.x;
        edge_buffer[0].y  = start.y;
        edge_buffer[0].z  = Left;
        edge_buffer[0].w  = 0;
#else // DEBUG_LINKED_USE_INT4_BUFFER
        edge_buffer[0]    = start;
#endif // DEBUG_LINKED_USE_INT4_BUFFER
        edge_index[Left]  = 1;
        edge_index[Right] = 0;
#else // ONE_THREAD_ONLY
        if( threadIdx.x == 0 ) {
#ifdef DEBUG_LINKED_USE_INT4_BUFFER
            edge_buffer[0].x  = start.x;
            edge_buffer[0].y  = start.y;
            edge_buffer[0].z  = Left;
            edge_buffer[0].w  = 0;
#else // DEBUG_LINKED_USE_INT4_BUFFER
            edge_buffer[0]    = start;
#endif // DEBUG_LINKED_USE_INT4_BUFFER
            edge_index[Left]  = 1;
            edge_index[Right] = 0;
        }
        __syncthreads();
#endif // ONE_THREAD_ONLY
    }

#ifdef DEBUG_LINKED_USE_INT4_BUFFER
    __device__ inline
    int2 get( Direction d )
    {
        int2 retval;
        int  idx;
        if( d == Left ) {
            idx = edge_index[Left];
            dec( idx );
        } else {
            idx = edge_index[Right];
        }
        retval.x = edge_buffer[idx].x;
        retval.y = edge_buffer[idx].y;
        return retval;
    }
#else // DEBUG_LINKED_USE_INT4_BUFFER
    __device__ inline
    const int2& get( Direction d )
    {
        int idx;
        if( d == Left ) {
            idx = edge_index[Left];
            dec( idx );
        } else {
            idx = edge_index[Right];
        }
        return edge_buffer[idx];
    }
#endif // DEBUG_LINKED_USE_INT4_BUFFER

#ifdef DEBUG_LINKED_USE_INT4_BUFFER
    __device__ inline
    void append( Direction d, int2 val, int j )
    {
        if( d == Left ) {
            const int idx = edge_index[Left];
            edge_buffer[idx].x = val.x;
            edge_buffer[idx].y = val.y;
            edge_buffer[idx].z = j;
            edge_buffer[idx].w = idx;
            assert( idx != edge_index[Right] );
            inc( edge_index[Left] );
        } else {
            assert( edge_index[Left] != edge_index[Right] );
            dec( edge_index[Right] );
            const int idx = edge_index[Right];
            edge_buffer[idx].x = val.x;
            edge_buffer[idx].y = val.y;
            edge_buffer[idx].z = 100+j;
            edge_buffer[idx].w = idx;
        }
    }
#else // DEBUG_LINKED_USE_INT4_BUFFER
    __device__ inline
    void append( Direction d, int2 val )
    {
        if( d == Left ) {
            const int idx = edge_index[Left];
            edge_buffer[idx] = val;
            assert( idx != edge_index[Right] );
            inc( edge_index[Left] );
        } else {
            assert( edge_index[Left] != edge_index[Right] );
            dec( edge_index[Right] );
            const int idx = edge_index[Right];
            edge_buffer[idx] = val;
        }
    }
#endif // DEBUG_LINKED_USE_INT4_BUFFER

    __device__ inline
    int size() const
    {
        if( edge_index[Left] > edge_index[Right] ) {
            return edge_index[Left] - edge_index[Right];
        } else {
            return edge_index[Left] + ( EDGE_LINKING_MAX_EDGE_LENGTH - edge_index[Right] );
        }
    }

    __device__
    void copy( cv::cuda::PtrStepSzInt2 output, int idx )
    {
        int sz = size();
        assert( idx < output.rows );
        if( sz > output.cols ) {
            printf("error copying link output, columns %d entries %d\n", output.cols, size() );
            assert( sz <= output.cols );
        }
        cv::cuda::PtrStepInt2_base_t* ptr = output.ptr(idx);
        int pos=edge_index[Right];
        for( int loop=0; loop<sz; loop++ ) {
            ptr[loop] = edge_buffer[pos];
            inc(pos);
        }
    }

private:
    __device__ inline void inc( int& idx )
    {
        idx = ( idx >= EDGE_LINKING_MAX_EDGE_LENGTH-1 ) ? 0 : idx + 1;
    }

    __device__ inline void dec( int& idx )
    {
        idx = ( idx == 0 ) ? EDGE_LINKING_MAX_EDGE_LENGTH-1 : idx - 1;
    }
};

/**
 * @param p Starting seed
 * @param triplepoints          List of all edge points that were potential voters
 * @param edgepoint_index_table Map: map coordinate to triplepoints entry
 * @param edges                 Map: non-0 if edge
 * @param d_dx, d_dy            Map: gradients
 * @param param_windowSizeOnInnerEllipticSegment
 * @param param_averageVoteMin
 */
__device__
void edge_linking_seed( const TriplePoint*           p,
                        DevEdgeList<TriplePoint>     triplepoints,
                        cv::cuda::PtrStepSz32s       edgepoint_index_table, // coord->triplepoint
                        cv::cuda::PtrStepSzb         edges,
                        cv::cuda::PtrStepSz16s       d_dx,
                        cv::cuda::PtrStepSz16s       d_dy,
                        int*                         d_ring_counter,
                        int                          d_ring_counter_max,
                        cv::cuda::PtrStepSzInt2      d_ring_output,
                        const size_t param_windowSizeOnInnerEllipticSegment,
                        const float  param_averageVoteMin )
{
    Direction direction       = Left;
    Direction other_direction = Right;

    float averageVote        = p->_winnerSize;
    float averageVoteCollect = averageVote;

    EdgeBuffer buf;

    buf.init( p->coord );

    RingBuffer    phi( Left, param_windowSizeOnInnerEllipticSegment );
    size_t        i     = 1;
    StopCondition found = STILL_SEARCHING;

    // const int* xoff = xoff_select;
    // const int* yoff = yoff_select[direction];

    // int2   this_cycle_coord = p->coord;
    // short2 this_cycle_gradients;

    while( (i < EDGE_LINKING_MAX_EDGE_LENGTH) && (found==STILL_SEARCHING) && (averageVote >= param_averageVoteMin) )
    {
        // this cycle coordinates
        int2  tcc = buf.get( direction );

        if( buf.size() > 1 ) {
            // other direction coordinates
            int2 odc = buf.get( other_direction );

            if( odc.x == tcc.x && odc.y == tcc.y ) {
                // We have gone a full circle.
                // End processing.
                found = FULL_CIRCLE;
                continue;
            }
        }

        assert( not outOfBounds( tcc.x, tcc.y, d_dx ) );

        // Angle refers to the gradient direction angle (might be optimized):
        short2 tcg; // this cycle gradients
        tcg.x = d_dx.ptr(tcc.y)[tcc.y];
        tcg.y = d_dy.ptr(tcc.y)[tcc.y];
        float atanval = atan2f( tcg.x, tcg.y );

        float angle = fmodf( atanval + 2.0f * HIP_PI_F, 2.0f * HIP_PI_F );

        phi.push_back( angle ); // thread 0 stores and all get the angle diff

        int shifting = rintf( ( (angle + HIP_PI_F / 4.0f)
                              / (2.0f * HIP_PI_F) ) * 8.0f ) - 1;

        // int j = threadIdx.x; // 0..7
#ifdef ONE_THREAD_ONLY
        int j = 0;
        while( j<8 ) {
            // winner is always the lowest j that finds a point
            int  off_index = ( direction == Right ) ?  ( ( 8 - shifting + j ) % 8 )
                                                    :  (     ( shifting + j ) % 8 );
            assert( off_index >= 0 );
            assert( off_index <  8 );
            int xoffset = xoff_select[off_index];
            int yoffset = yoff_select[direction][off_index];
            int2 new_point = make_int2( tcc.x + xoffset, tcc.y + yoffset );

            if( ( new_point.x >= 0 && new_point.x < edges.cols ) &&
                ( new_point.y >= 0 && new_point.y < edges.rows ) &&
                ( edges.ptr(new_point.y)[new_point.x] > 0 ) )
            {
                // This j has found a point.

                float s;
                float c;
                __sincosf( phi.diff(), &s, &c );
                s = ( direction == Left ) ? s : -s;

                //
                // three conditions to conclude CONVEXITY_LOST
                //
                bool stop;
                stop = ( ( ( phi.size() == param_windowSizeOnInnerEllipticSegment ) &&
                         ( s <  0.0f   ) ) ||
                         ( ( s < -0.707f ) && ( c > 0.0f ) ) ||
                         ( ( s <  0.0f   ) && ( c < 0.0f ) ) );
                if( stop ) {
                    if( direction == Right ) {
                        found = CONVEXITY_LOST;
                        break;
                    } else {
                        found           = STILL_SEARCHING;
                        direction       = Right;
                        other_direction = Left;
                        phi.set_direction( Right );
                    }
                } else {
                    found = STILL_SEARCHING;
                    buf.append( direction, new_point );
                    int idx = edgepoint_index_table.ptr(new_point.y)[new_point.x];
                    if( idx > 0 ) {
                        assert( idx < triplepoints.Size() );
                        TriplePoint* ptr = &triplepoints.ptr[idx];
                        averageVoteCollect += ptr->_winnerSize;
                    }
                }

                break;
            }
            j++;
        }
        if( found == STILL_SEARCHING ) {
            // checked all 8 directions, but no point found
            if( direction == Right ) {
                found = EDGE_NOT_FOUND;
                continue;
            }
            found           = STILL_SEARCHING;
            direction       = Right;
            other_direction = Left;
            phi.set_direction( Right );
            continue;
        }
#else // not ONE_THREAD_ONLY
#if 0
        int j = 7 - threadIdx.x; // counting backwards, so that the winner in __ffs
                                 // is identical to winner in loop code that starts
                                 // at 0
#else
        int j = threadIdx.x;
#endif
        int  off_index = ( direction == Right ) ?  ( ( 8 - shifting + j ) % 8 )
                                                :  (     ( shifting + j ) % 8 );
        assert( off_index >= 0 );
        assert( off_index <  8 );
        int xoffset = xoff_select[off_index];
        int yoffset = yoff_select[direction][off_index];
        int2 new_point = make_int2( tcc.x + xoffset, tcc.y + yoffset );
        // int2 new_point = make_int2( tcc.x + xoff[off_index],
        //                             tcc.y + yoff[off_index] );

        bool point_found = false;
        if( ( new_point.x >= 0 && new_point.x < edges.cols ) &&
            ( new_point.y >= 0 && new_point.y < edges.rows ) &&
            ( edges.ptr(new_point.y)[new_point.x] > 0 ) )
        {
            // at least one of 8 threads has found an edge
            // point, and has its coordinates in new_point
            point_found = true;
        }
        uint32_t any_point_found = __ballot( point_found );

        if( not any_point_found ) {
            if( direction == Right ) {
                found = EDGE_NOT_FOUND;
                continue;
            }
            found           = STILL_SEARCHING;
            direction       = Right;
            other_direction = Left;
            phi.set_direction( Right );
            continue;
        }

#if 0
        // This direction still has points.
        // We can identify the highest threadId / lowest rotation value j
        uint32_t computer = __ffs( any_point_found ) - 1;
#else
        if( point_found == false ) j = 8;
        j = min( __shfl_xor( j, 4 );
        j = min( __shfl_xor( j, 2 );
        j = min( __shfl_xor( j, 1 );
        assert( j < 8 );
#endif

        found = LOW_FLOW;

        float winnerSize = 0.0f;
#if 0
        if( threadIdx.x == computer ) {
#else
        if( threadIdx.x == j ) {
#endif
            //
            // The whole if/else block is identical for all j.
            // No reason to do it more than once. Astonishingly,
            // the decision to use the next point f is entirely
            // independent of its position ???
            //
            float s;
            float c;
            __sincosf( phi.diff(), &s, &c );
            s = ( direction == Left ) ? s : -s;

            //
            // three conditions to conclude CONVEXITY_LOST
            //
            bool stop;

            stop = ( ( ( phi.size() == param_windowSizeOnInnerEllipticSegment ) &&
                       ( s <  0.0f   ) ) ||
                     ( ( s < -0.707f ) && ( c > 0.0f ) ) ||
                     ( ( s <  0.0f   ) && ( c < 0.0f ) ) );
            
            if( not stop ) {
#ifdef DEBUG_LINKED_USE_INT4_BUFFER
                buf.append( direction, new_point, j );
#else // DEBUG_LINKED_USE_INT4_BUFFER
                buf.append( direction, new_point );
#endif // DEBUG_LINKED_USE_INT4_BUFFER
                int idx = edgepoint_index_table.ptr(new_point.y)[new_point.x];
                if( idx > 0 ) {
                    // ptr can be any seed or voter candidate, and its _winnerSize
                    // may be 0
                    assert( idx < triplepoints.Size() );
                    TriplePoint* ptr = &triplepoints.ptr[idx];
                    winnerSize = ptr->_winnerSize;
                }

                // we collect votes after the IF-block, using a reduce
                found = FOUND_NEXT;
            } else {
                found = CONVEXITY_LOST;
            }
        } // end of asynchronous block
        assert( found == LOW_FLOW || found == FOUND_NEXT || found == CONVEXITY_LOST );

        // both FOUND_NEXT and CONVEXITY_LOST are > LOW_FLOW
        found = (StopCondition)max( (int)found, __shfl_xor( (int)found, 4 ) );
        found = (StopCondition)max( (int)found, __shfl_xor( (int)found, 2 ) );
        found = (StopCondition)max( (int)found, __shfl_xor( (int)found, 1 ) );

        assert( found == FOUND_NEXT || found == CONVEXITY_LOST );

        if( found == FOUND_NEXT ) {
            found      = STILL_SEARCHING;
            // only the thread going into the if() is not null
            winnerSize = winnerSize + __shfl_xor( winnerSize, 1 );
            winnerSize = winnerSize + __shfl_xor( winnerSize, 2 );
            winnerSize = winnerSize + __shfl_xor( winnerSize, 4 );
            averageVoteCollect += winnerSize;
        } else {
            assert( found == CONVEXITY_LOST );
            if( direction == Right ) {
                found = CONVEXITY_LOST;
                continue;
            }
            found           = STILL_SEARCHING;
            direction       = Right;
            other_direction = Left;
            phi.set_direction( Right );
        }
#endif // ONE_THREAD_ONLY

        ++i;

        averageVote = averageVoteCollect / buf.size();
    } // while

    if( found == STILL_SEARCHING && averageVote < param_averageVoteMin ) {
        found = VOTE_LOW;
    }

#ifdef ONE_THREAD_ONLY
    if( true )
#else // ONE_THREAD_ONLY
    if( threadIdx.x == 0 )
#endif // ONE_THREAD_ONLY
    {
        if( (i == EDGE_LINKING_MAX_EDGE_LENGTH) || (found == CONVEXITY_LOST) || (found == FULL_CIRCLE) ) {
            int convexEdgeSegmentSize = buf.size();
            if (convexEdgeSegmentSize > param_windowSizeOnInnerEllipticSegment) {
                int write_index = atomicAdd( d_ring_counter, 1 );
                if( write_index <= d_ring_counter_max ) {
#ifdef KERNEL_PRINT_SUCCESS_CAUSE
                    const char* c;
                    if( i == EDGE_LINKING_MAX_EDGE_LENGTH ) {
                        c = "max length";
                    } if( found == CONVEXITY_LOST ) {
                        c = "conv lost";
                    } else {
                        c = "full circle";
                    }
                    printf("From (%d,%d): %d (average vote %f) - accept (%s), edge segment size %d, write pos %d\n",
                           p->coord.x, p->coord.y,
                           i, averageVote,
                           c, convexEdgeSegmentSize,
                           write_index );
#endif // KERNEL_PRINT_SUCCESS_CAUSE
                    buf.copy( d_ring_output, write_index );
                }
#ifdef KERNEL_PRINT_ERROR_CAUSE
                else {
                    printf("From (%d,%d): %d (average vote %f) - skip, max number of arcs reached (%d)\n", p->coord.x, p->coord.y, i, averageVote, d_ring_counter_max );
                }
#endif // KERNEL_PRINT_ERROR_CAUSE
            }
#ifdef KERNEL_PRINT_ERROR_CAUSE
            else {
                int d = param_windowSizeOnInnerEllipticSegment;
                printf("From (%d,%d): %d (average vote %f) - skip, edge segment size %d <= %d\n", p->coord.x, p->coord.y, i, averageVote, convexEdgeSegmentSize, d );
            }
#endif // KERNEL_PRINT_ERROR_CAUSE
        }
#ifdef KERNEL_PRINT_ERROR_CAUSE
        else {
            const char* c;
            switch(found) {
                case LOW_FLOW : c = "LOW_FLOW"; break;
                case VOTE_LOW : c = "VOTE_LOW"; break;
                case CONVEXITY_LOST : c = "CONVEXITY_LOST"; break;
                case EDGE_NOT_FOUND : c = "EDGE_NOT_FOUND"; break;
                case NONE : c = "NONE"; break;
                case STILL_SEARCHING : c = "STILL_SEARCHING"; break;
                case FOUND_NEXT : c = "FOUND_NEXT"; break;
                case FULL_CIRCLE : c = "FULL_CIRCLE"; break;
                default: c = "UNKNOWN code"; break;
            }
            printf("From (%d,%d): %d (average vote %f) - skip, not max length, not convexity lost, but %s\n", p->coord.x, p->coord.y, i, averageVote, c );
        }
#endif // KERNEL_PRINT_ERROR_CAUSE
    }
}

/**
 * @param edges         The 0/1 map of edge points
 * @param d_dx
 * @param d_dy
 * @param triplepoints  The array of points including voters and seeds
 * @param seed_indices  The array of indices of seeds in triplepoints
 * @param d_ring_counter A frame-global counter of edge segments
 * @param d_ring_ouput   A huge buffer to hold all edge segments multiple times
 * @param param_windowSizeOnInnerEllipticSegment
 * @param param_averageVoteMin
 */
__global__
void edge_linking( DevEdgeList<TriplePoint>     triplepoints,
                   DevEdgeList<int>             seed_indices,
                   cv::cuda::PtrStepSz32s       edgepoint_index_table,
                   cv::cuda::PtrStepSzb         edges,
                   cv::cuda::PtrStepSz16s       d_dx,
                   cv::cuda::PtrStepSz16s       d_dy,
                   int*                         d_ring_counter,
                   int                          d_ring_counter_max,
                   cv::cuda::PtrStepSzInt2      d_ring_output,
                   size_t param_windowSizeOnInnerEllipticSegment,
                   float  param_averageVoteMin )
{
    const int       offset    = blockIdx.x;

    // The first seed index is always invalid
    if( offset == 0 ) return;

    int idx = seed_indices.ptr[offset];
    if( idx >= triplepoints.Size() ) return;

    TriplePoint* p = &triplepoints.ptr[idx];

    edge_linking_seed( p,
                       triplepoints,
                       edgepoint_index_table,
                       edges,
                       d_dx,
                       d_dy,
                       d_ring_counter,
                       d_ring_counter_max,
                       d_ring_output,
                       param_windowSizeOnInnerEllipticSegment,
                       param_averageVoteMin );
}

}; // namespace linking

__host__
void Frame::applyLink( const cctag::Parameters& params )
{
    cout << "Enter " << __FUNCTION__ << endl;

    if( params._windowSizeOnInnerEllipticSegment > EDGE_LINKING_MAX_RING_BUFFER_SIZE ) {
        cerr << "Error in " << __FILE__ << ":" << __LINE__ << ":" << endl
             << "    static maximum of parameter ring buffer size is "
             << EDGE_LINKING_MAX_RING_BUFFER_SIZE << "," << endl
             << "    parameter file wants " << params._windowSizeOnInnerEllipticSegment << endl
             << "    edit " << __FILE__ << " and recompile" << endl
             << endl;
        exit( -1 );
    }

    if( _vote._seed_indices.host.size <= 0 ) {
        cout << "Leave " << __FUNCTION__ << endl;
        // We have note found any seed, return
        return;
    }

#ifndef NDEBUG
    POP_CUDA_SYNC( _stream );
    cout << "  Searching arcs from " << _vote._seed_indices.host.size << " seeds" << endl;
    cout << "  Parameters: _windowSizeOnInnerEllipticSegment="
         << params._windowSizeOnInnerEllipticSegment << endl
         << "              _averageVoteMin=" << params._averageVoteMin << endl;
#endif // NDEBUG

    /* Both init steps should be done in another stream, earlier. No reason to do
     * this synchronously.
     */
    POP_CUDA_SET0_ASYNC( _d_ring_counter, _stream );

    POP_CUDA_MEMSET_ASYNC( _d_ring_output.data, 0, _d_ring_output.step*_d_ring_output.rows, _stream );

    dim3 block;
    dim3 grid;

    /* Seeds have an index in the _seed_indices list.
     * For each of those seeds, mark their coordinate with a label.
     * This label is their index in the _seed_indices list, because
     * it is a unique int strictly > 0
     */
#ifdef ONE_THREAD_ONLY
    block.x = 1;
#else // ONE_THREAD_ONLY
    block.x = 8;
#endif // ONE_THREAD_ONLY
    block.y = 1;
    block.z = 1;
    grid.x  = _vote._seed_indices.host.size;
    grid.y  = 1;
    grid.z  = 1;

    linking::edge_linking
        <<<grid,block,0,_stream>>>
        ( _vote._chained_edgecoords.dev,
          _vote._seed_indices.dev,
          _vote._d_edgepoint_index_table,
          _d_edges,
          _d_dx,
          _d_dy,
          _d_ring_counter,
          _d_ring_counter_max,
          _d_ring_output,
          params._windowSizeOnInnerEllipticSegment,
          params._averageVoteMin );

    POP_CHK_CALL_IFSYNC;

    POP_CUDA_MEMCPY_2D_ASYNC( _h_ring_output.data, _h_ring_output.step,
                              _d_ring_output.data, _d_ring_output.step,
                              _d_ring_output.cols*sizeof(cv::cuda::PtrStepInt2_base_t),
                              _d_ring_output.rows,
                              hipMemcpyDeviceToHost,
                              _stream );

    POP_CHK_CALL_IFSYNC;

#ifndef NDEBUG
    int h_ring_counter;
    POP_CUDA_MEMCPY_TO_HOST_ASYNC( &h_ring_counter, _d_ring_counter, sizeof(int), _stream );
    POP_CHK_CALL_IFSYNC;
    POP_CUDA_SYNC( _stream );
    cout << "  Found arcs from " << h_ring_counter << " seeds" << endl;
#endif // NDEBUG

    cout << "Leave " << __FUNCTION__ << endl;
}
}; // namespace popart



#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_math_constants.h>

#include "frame.h"

#define EDGE_NOT_FOUND -1
#define CONVEXITY_LOST -2
#define LOW_FLOW -3

using namespace std;

namespace popart
{

__constant__
static int xoff_select[8]    =   { 1,  1,  0, -1, -1, -1,  0,  1};

__constant__
static int yoff_select[2][8] = { { 0, -1, -1, -1,  0,  1,  1,  1},
                                 { 0,  1,  1,  1,  0, -1, -1, -1} };

#define EDGE_LINKING_MAX_EDGE_LENGTH    100

template<typename T>
struct ListReplacement
{
    T oops;

    __device__ ListReplacement() { }
    __device__ void push_back( T& val ) { oops = val; }
    __device__ void pop_front( ) { }
    __device__ T&   back()  { return oops; }
    __device__ T&   front() { return oops; }
    __device__ int  size() const { return 1; }
};

/**
 * @param edges         The 0/1 map of edge points
 * @param d_dx
 * @param d_dy
 * @param triplepoints  The array of points including voters and seeds
 * @param edge_indices  The array of indices of seeds in triplepoints
 * @param param_windowSizeOnInnerEllipticSegment
 * @param param_averageVoteMin
 */
__device__
void edge_linking( cv::cuda::PtrStepSzb     edges,
                   DevEdgeList<TriplePoint> triplepoints,
                   DevEdgeList<int>         edge_indices,
                   size_t param_windowSizeOnInnerEllipticSegment,
                   float  param_averageVoteMin )
{
    // pmax              = one seed

    const int offset = blockIdx.x * 32 + threadIdx.z;
    int direction = ( threadIdx.y == 0 ) ?  1   // link left
                                         : -1;  // link right

    if( offset == 0 ) return;

    int idx = edge_indices.ptr[offset];
    if( idx >= triplepoints.Size() ) return;

    TriplePoint* p = &triplepoints.ptr[idx];

    __shared__ TriplePoint* convexEdgeSegment[2 * EDGE_LINKING_MAX_EDGE_LENGTH+1];
    __shared__ int          ces_idx[2]; // 0 offset right, 1 negative offset left
    __shared__ float        averageVoteCollect[2];
    float                   averageVote =  p->_winnerSize;
    if( threadIdx.x == 0 ) {
        ces_idx[threadIdx.y] = EDGE_LINKING_MAX_EDGE_LENGTH + 1 + direction;
        averageVoteCollect[threadIdx.y] = ( threadIdx.y==0 ) ? averageVote : 0;
        if( threadIdx.y == 0 ) {
            convexEdgeSegment[EDGE_LINKING_MAX_EDGE_LENGTH+1] = p;
        }
    }
    __syncthreads();

    ListReplacement<float> phi;
    std::size_t i = 0;
    bool found = true;

    int stop = 0;


    const int* xoff = xoff_select;
    const int* yoff = yoff_select[threadIdx.y];

    while( (i < EDGE_LINKING_MAX_EDGE_LENGTH) && (found) && (averageVote >= param_averageVoteMin) )
    {
        bool skip;
        if( threadIdx.x == 0 ) {
            skip = atomicExch( &p->edgeLinking.processed, true );
        }
        skip = __shfl( skip, 0 );
        if( skip ) {
            // Another warp has processed this point or is processing this
            // point.
            // End processing.
            break;
        }

        // Angle refers to the gradient direction angle (might be optimized):
        float angle = fmodf( atan2f(p->d.x,p->d.y) + 2.0f * HIP_PI_F, 2.0f * HIP_PI_F );

        phi.push_back(angle);

        if (phi.size() > param_windowSizeOnInnerEllipticSegment) // TODO , 4 est un paramètre de l'algorithme, + les motifs à détecter sont importants, + la taille de la fenêtre doit être grande
        {
            phi.pop_front();
        }

        int shifting = rintf( ( (angle + HIP_PI_F / 4.0f)
                              / (2.0f * HIP_PI_F) ) * 8.0f ) - 1;

        // int j = threadIdx.x; // 0..7
        int j = 7 - threadIdx.x; // counting backwards, so that the winner in __ffs
                                 // is identical to winner in loop code that starts
                                 // at 0
        int sx, sy;

        if (direction == 1) {
            int off_index = ( 8 - shifting + j ) % 8;
            sx = p->coord.x + xoff[off_index];
            sy = p->coord.y + yoff[off_index];
        } else {
            int off_index = ( shifting + j ) % 8;
            sx = p->coord.x + xoff[off_index];
            sy = p->coord.y + yoff[off_index];
        }

        TriplePoint* f;
        int new_edgepoint_index;
        bool point_found = false;
        if( ( sx >= 0 && sx < edges.cols ) &&
            ( sy >= 0 && sy < edges.rows ) &&
            ( new_edgepoint_index = edges.ptr(sy)[sx] ) )
        {
            f = &triplepoints.ptr[new_edgepoint_index];
            point_found = true;
        }

        uint32_t any_point_found = __ballot( point_found );
        uint32_t computer        = __ffs( any_point_found );

        if( not any_point_found ) {
            stop  = EDGE_NOT_FOUND;
            found = false;
            break;
        }

        if( threadIdx.x == computer ) {
            if( f->edgeLinking.processed ) {
                stop  = 0;
                found = false;
                break;
            }
            //
            // The whole if/else block is identical for all j.
            // No reason to do it more than once. Astonishingly,
            // the decision to use the next point f is entirely
            // independent of its position ???
            //
            float s;
            float c;
            __sincosf( phi.back() - phi.front(), &s, &c );
            s *= direction;

            //
            // three conditions to conclude CONVEXITY_LOST
            //
            stop = ( ( ( phi.size() == param_windowSizeOnInnerEllipticSegment ) &&
                       ( s <  0.0f   ) ) ||
                     ( ( s < -0.707f ) && ( c > 0.0f ) ) ||
                     ( ( s <  0.0f   ) && ( c < 0.0f ) ) );
            
            if( not stop ) {
                // this outcome of this test does not depend on
                // threadIdx.x / j

                p->edgeLinking.processed = true;

                convexEdgeSegment[ces_idx[threadIdx.y]] = f;
                ces_idx[threadIdx.y]     += direction;
                averageVoteCollect[threadIdx.y] += f->_winnerSize;
                stop = 1;

                if( f->edgeLinking.processed ) {
                    found = false;
                }
            } else {
                stop  = CONVEXITY_LOST;
                found = false;
            }
        } // end of asynchronous block

        p     = (TriplePoint*)__shfl( (size_t)f,     computer );
        found = __shfl( found, computer );
        ++i;

        __syncthreads();

        averageVote = averageVoteCollect[0] + averageVoteCollect[1];
        int convexEdgeSegmentSize = ces_idx[0] - ces_idx[1];
        averageVote /= convexEdgeSegmentSize;
    } // while


    if( threadIdx.x == 0 && threadIdx.y == 0 )
    {
        int n = 0;
        if ((i == EDGE_LINKING_MAX_EDGE_LENGTH) || (stop == CONVEXITY_LOST)) {
            int convexEdgeSegmentSize = ces_idx[0] - ces_idx[1];
            if (convexEdgeSegmentSize > param_windowSizeOnInnerEllipticSegment) {
                for( int i=ces_idx[1]; i<ces_idx[0]; i++ ) {
                    TriplePoint* collectedP = convexEdgeSegment[i];
                    if (n == convexEdgeSegmentSize - param_windowSizeOnInnerEllipticSegment) {
                        break;
                    } else {
                        // collectedP->_processedIn = true;
                        ++n;
                    }
                }
            }
        } else if (stop == EDGE_NOT_FOUND) {
            for( int i=ces_idx[1]; i<ces_idx[0]; i++ ) {
                TriplePoint* collectedP = convexEdgeSegment[i];
                // collectedP->_processedIn = true;
            }
        }
    }
}

}; // namespace popart



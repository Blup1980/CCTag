#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_math_constants.h>

#include "frame.h"
#include "assist.h"
#include "recursive_sweep.h"

using namespace std;

namespace popart
{

namespace linking
{
__constant__
static int xoff_select[8]    =   { 1,  1,  0, -1, -1, -1,  0,  1};

__constant__
static int yoff_select[2][8] = { { 0, -1, -1, -1,  0,  1,  1,  1},
                                 { 0,  1,  1,  1,  0, -1, -1, -1} };

#define EDGE_LINKING_MAX_EDGE_LENGTH    100
#define MAX_RING_BUFFER_SIZE            40

enum Direction
{
    Left = 0,
    Right = 1
};

enum StopCondition
{
    LOW_FLOW        =     -3,
    CONVEXITY_LOST      = -2,
    EDGE_NOT_FOUND      = -1,
    NONE                =  0,
    STILL_SEARCHING     =  1,
    FOUND_NEXT          =  2,
    FULL_CIRCLE         =  3
};

__shared__ float ring_buffer[2][MAX_RING_BUFFER_SIZE];

struct RingBuffer
{
    Direction direction;
    size_t    max_size;
    size_t    front_index;
    size_t    back_index;
    size_t    ct;

    __device__ RingBuffer( Direction dir, size_t size ) {
        direction   = dir;
        max_size    = size;
        front_index = 0;
        back_index  = 0;
        ct          = 0;
    }

    __device__ inline void set_direction( Direction dir ) {
        direction = dir;
    }

    __device__ inline size_t inc( size_t& idx )
    {
        return ( idx == max_size-1 ) ? 0 : idx + 1;
    }

    __device__ inline size_t dec( size_t& idx )
    {
        return ( idx == 0 ) ? max_size-1 : idx - 1;
    }

    __device__ void push_back( float angle )
    {
        ring_buffer[direction][front_index].angle = angle;
        back_index = inc( back_index );
        if( front_index == back_index )
            front_index = inc( front_index );
        else
            ct++;
    }

    __device__ inline float front( )
    {
        assert( front_index != back_index );
        return ring_buffer[direction][front_index].angle;
    }

    __device__ inline float back( )
    {
        assert( front_index != back_index );
        size_t lookup = dec( back_index );
        return ring_buffer[direction][lookup].angle;
    }

    __device__ int  inline size() const {
        return ct;
    }
};

__shared__ int2  edge_buffer[2][EDGE_LINKING_MAX_EDGE_LENGTH]; // convexEdgeSegment

struct EdgeBuffer
{
    int head[2];

    __device__ inline
    void init( int2 start )
    {
        edge_buffer[Left ][0] = start;
        edge_buffer[Right][0] = start;
        head[Left ] = 0;
        head[Right] = 0;
    }

    __device__ inline
    const int2& get( Direction d )
    {
        return edge_buffer[d][head[d]];
    }

    __device__ inline
    void append( Direction d, int2 val )
    {
        head[d]++;
        edge_buffer[d][head[d]] = val;
    }

    __device__ inline
    int size() const
    {
        return head[Left] + head[Right] + 1;
    }

    __device__
    void copy( cv::cuda::PtrStepSzInt2 output, int idx )
    {
        assert( idx < output.rows );
        assert( size() < output.cols );
        int j = 0;
        int2* ptr = output.ptr(idx);
        for( int i=head[Right]; i>=0; i-- ) {
            ptr[j] = edge_buffer[Right][i];
            j++;
        }
        for( int i=1; i<=head[Left]; i++ ) {
            ptr[j] = edge_buffer[Left][i];
            j++;
        }
    }
};

/**
 * @param p Starting seed
 * @param triplepoints          List of all edge points that were potential voters
 * @param edgepoint_index_table Map: map coordinate to triplepoints entry
 * @param edges                 Map: non-0 if edge
 * @param param_windowSizeOnInnerEllipticSegment
 * @param param_averageVoteMin
 */
__device__
void edge_linking_seed( const TriplePoint*           p,
                        DevEdgeList<TriplePoint>     triplepoints,
                        cv::cuda::PtrStepSz32s       edgepoint_index_table, // coord->triplepoint
                        cv::cuda::PtrStepSzb         edges,
                        int*                         d_ring_counter,
                        cv::cuda::PtrStepSzInt2      d_ring_output,
                        size_t param_windowSizeOnInnerEllipticSegment,
                        float  param_averageVoteMin )
{
    const Direction direction       = Left;
    const Direction other_direction = Right;

    float averageVote        = p->_winnerSize;
    float averageVoteCollect = averageVote;

    EdgeBuffer buf;

    if( threadIdx.x == 0 ) {
        buf.init( p->coord );
    }
    __syncthreads();

    RingBuffer<float> phi( Left, param_windowSizeOnInnerEllipticSegment );
    size_t        i     = 0;
    StopCondition found = STILL_SEARCHING;
    // int           stop  = 0;

    const int* xoff = xoff_select;
    const int* yoff = yoff_select[direction];

    // int2   this_cycle_coord = p->coord;
    // short2 this_cycle_gradients;

    while( (i < EDGE_LINKING_MAX_EDGE_LENGTH) && (found==STILL_SEARCHING) && (averageVote >= param_averageVoteMin) )
    {
        // this cycle coordinates
        const int2&  tcc = buf.get( direction );

        if( i > 0 ) {
            // other direction coordinates
            const int2& odc = buf.get( other_direction );

            if( odc.x == tcc.x && odc.y == tcc.y ) {
                // We have gone a full circle.
                // End processing.
                found = FULL_CIRCLE;
                continue;
            }
        }

        float angle;

        if( threadIdx.x == 0 ) {
            // this cycle gradients
            short2 tcg;

            // Angle refers to the gradient direction angle (might be optimized):
            tcg.x = _d_dx.ptr(tcc.y)[tcc.y];
            tcg.y = _d_dy.ptr(tcc.y)[tcc.y];
            float atanval = atan2f( tcg.x, tcg.y );

            angle = fmodf( atanval + 2.0f * HIP_PI_F, 2.0f * HIP_PI_F );

            phi.push_back( angle );
        }
        angle = __shfl( angle, 0 );

        int shifting = rintf( ( (angle + HIP_PI_F / 4.0f)
                              / (2.0f * HIP_PI_F) ) * 8.0f ) - 1;

        // int j = threadIdx.x; // 0..7
        int j = 7 - threadIdx.x; // counting backwards, so that the winner in __ffs
                                 // is identical to winner in loop code that starts
                                 // at 0
        int  off_index = ( direction == Right ) ?  ( ( 8 - shifting + j ) % 8 )
                                                :  (     ( shifting + j ) % 8 );
        int2 new_point = make_int2( tcc.x + xoff[off_index],
                                    tcc.y + yoff[off_index] );

        bool point_found = false;
        if( ( new_point.x >= 0 && new_point.x < edges.cols ) &&
            ( new_point.y >= 0 && new_point.y < edges.rows ) &&
            ( edges.ptr(new_point.y)[new_point.x] > 0 ) )
        {
            // at least one of 8 threads has found an edge
            // point, and has its coordinates in new_point
            point_found = true;
        }

        uint32_t any_point_found = __ballot( point_found );

        if( not any_point_found ) {
            if( direction == Right ) {
                found = EDGE_NOT_FOUND;
                continue;
            }
            found           = STILL_SEARCHING;
            direction       = Right;
            other_direction = Left;
            ring_buffer.set_direction( Right );
            continue;
        }

        // This direction still has points.
        // We can identify the highest threadId / lowest rotation value j
        uint32_t computer = __ffs( any_point_found );

        found = LOW_FLOW;

        float winnerSize = 0.0f;
        if( threadIdx.x == computer ) {
            //
            // The whole if/else block is identical for all j.
            // No reason to do it more than once. Astonishingly,
            // the decision to use the next point f is entirely
            // independent of its position ???
            //
            float s;
            float c;
            __sincosf( phi.back() - phi.front(), &s, &c );
            s = ( direction == Left ) ? s : -s;

            //
            // three conditions to conclude CONVEXITY_LOST
            //
            stop = ( ( ( phi.size() == param_windowSizeOnInnerEllipticSegment ) &&
                       ( s <  0.0f   ) ) ||
                     ( ( s < -0.707f ) && ( c > 0.0f ) ) ||
                     ( ( s <  0.0f   ) && ( c < 0.0f ) ) );
            
            if( not stop ) {
                buffer.append( new_point );
                int idx = edgepoint_index_table.ptr(new_point.y)[new_point.x];
                float winnerSize = 0.0f;
                if( idx > 0 ) {
                    // ptr can be any seed or voter candidate, and its _winnerSize
                    // may be 0
                    TriplePoint* ptr = &triplepoints[idx];
                    winnerSize = ptr->_winnerSize;
                }

                // we collect votes after the IF-block, using a reduce
                found = FOUND_NEXT;
            } else {
                found = CONVEXITY_LOST;
            }
        } // end of asynchronous block

        // both FOUND_NEXT and CONVEXITY_LOST are > LOW_FLOW
        found = max( found, __shfl_xor( found, 1 ) );
        found = max( found, __shfl_xor( found, 2 ) );
        found = max( found, __shfl_xor( found, 4 ) );

        if( found == FOUND_NEXT ) {
            found      = STILL_SEARCHING;
            // only the thread going into the if() is not null
            winnerSize = winnerSize + __shfl_xor( winnerSize, 1 );
            winnerSize = winnerSize + __shfl_xor( winnerSize, 2 );
            winnerSize = winnerSize + __shfl_xor( winnerSize, 4 );
            averageVoteCollect += winnerSize;
        } else {
            assert( found == CONVEXITY_LOST );
            if( direction == Right ) {
                found = CONVEXITY_LOST;
                continue;
            }
            found           = STILL_SEARCHING;
            direction       = Right;
            other_direction = Left;
            ring_buffer.set_direction( Right );
        }

        ++i;

        averageVote = averageVoteCollect / buffer.size();
    } // while

    if( threadIdx.x == 0 )
    {
        if( (i == EDGE_LINKING_MAX_EDGE_LENGTH) || (found == CONVEXITY_LOST) ) {
            int convexEdgeSegmentSize = buffer.size();
            if (convexEdgeSegmentSize > param_windowSizeOnInnerEllipticSegment) {
                int write_index = atomicAdd( &d_ring_counter, 1 );
                if( write_index <= d_ring_counter_max ) {
                    buffer.copy( d_ring_output, write_index );
                }
            }
        }
    }
}

/**
 * @param edges         The 0/1 map of edge points
 * @param d_dx
 * @param d_dy
 * @param triplepoints  The array of points including voters and seeds
 * @param seed_indices  The array of indices of seeds in triplepoints
 * @param d_ring_counter A frame-global counter of edge segments
 * @param d_ring_ouput   A huge buffer to hold all edge segments multiple times
 * @param param_windowSizeOnInnerEllipticSegment
 * @param param_averageVoteMin
 */
__global__
void edge_linking( DevEdgeList<TriplePoint>     triplepoints,
                   DevEdgeList<int>             seed_indices,
                   cv::cuda::PtrStepSz32s       edgepoint_index_table,
                   cv::cuda::PtrStepSzb         edges,
                   int*                         d_ring_counter,
                   cv::cuda::PtrStepSzInt2      d_ring_output,
                   size_t param_windowSizeOnInnerEllipticSegment,
                   float  param_averageVoteMin )
{
    const int       offset    = blockIdx.x;

    // The first seed index is always invalid
    if( offset == 0 ) return;

    int idx = seed_indices.ptr[offset];
    if( idx >= triplepoints.Size() ) return;

    TriplePoint* p = &triplepoints.ptr[idx];

    edge_linking_seed( p,
                       triplepoints,
                       edgepoint_index_table,
                       edges,
                       d_ring_counter,
                       d_ring_output,
                       param_windowSizeOnInnerEllipticSegment,
                       param_averageVoteMin );
}

}; // namespace linking

__host__
void Frame::applyLink( const cctag::Parameters& params )
{
    cout << "Enter " << __FUNCTION__ << endl;

    if( param.windowSizeOnInnerEllipticSegment > MAX_RING_BUFFER_SIZE ) {
        cerr << "Error in " << __FILE__ << ":" << __LINE__ << ":" << endl
             << "    static maximum of parameter ring buffer size is "
             << MAX_RING_BUFFER_SIZE << "," << endl;
             << "    parameter file wants " << param.windowSizeOnInnerEllipticSegment << endl
             << "    edit " << __FILE__ << " and recompile" << endl
             << endl;
        exit( -1 );
    }

    if( _vote._seed_indices.host.size <= 0 ) {
        cout << "Leave " << __FUNCTION__ << endl;
        // We have note found any seed, return
        return;
    }

    /* We could re-use edges, but it has only 1-byte cells.
     * Seeds may have int-valued indices, so we need 4-byte cells.
     * _d_intermediate is a float image, and can be used.
     */
    // cv::cuda::PtrStepSz32s edge_cast( _d_intermediate );

    dim3 block;
    dim3 grid;

    /* Seeds have an index in the _seed_indices list.
     * For each of those seeds, mark their coordinate with a label.
     * This label is their index in the _seed_indices list, because
     * it is a unique int strictly > 0
     */
    block.x = 8;
    block.y = 0;
    block.z = 1;
    grid.x  = _vote._seed_indices.host.size;
    grid.y  = 1;
    grid.z  = 1;

    linking::edge_linking
        <<<grid,block,0,_stream>>>
        ( _vote._chained_edgecoords.dev,
          _vote._seed_indices.dev,
          _vote._d_edgepoint_index_table,
          _d_edges,
          _d_ring_counter,
          _d_ring_output,
          param.windowSizeOnInnerEllipticSegment,
          param.averageVoteMin );

    cout << "Leave " << __FUNCTION__ << endl;
}
}; // namespace popart


